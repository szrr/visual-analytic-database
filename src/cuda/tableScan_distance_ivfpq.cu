#include "hip/hip_runtime.h"
/*
   Copyright (c) 2012-2013 The Ohio State University.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <string.h>
#include <unistd.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <random>
#include <algorithm>
#include <unordered_set>
#include "scanImpl.cu"
#include "../include/common.h"
#include "../include/gpuCudaLib.h"
#include "../include/Mempool.h"
#include "../include/utils.h"

#include <thrust/execution_policy.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/binary_search.h>

#include <faiss/gpu/GpuAutoTune.h>
#include <faiss/gpu/GpuCloner.h>
#include <faiss/gpu/GpuIndexIVFPQ.h>
#include <faiss/gpu/GpuIndexIVFFlat.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/index_io.h>
#include <faiss/IndexFlat.h>
#include <faiss/IndexIVFPQ.h>

#define CHECK_POINTER(p)   do {                     \
    if(p == NULL){                                  \
        perror("Failed to allocate host memory");   \
        exit(-1);                                   \
    }} while(0)

/*
 * stringCmp: Compare two strings on GPU using one single GPU thread.
 * @buf1: the first input buffer
 * @buf2: the second input buffer
 * @size: the length of data to be compared
 *
 * Return
 *  1 if buf1 is larger
 *  0 if they are equal
 *  -1 if buf2 is larger
 */

__device__ static inline int stringCmp(char* buf1, char *buf2, int size){
    int i;
    int res = 0;
    for(i=0;i<size;i++){
        if(buf1[i] > buf2[i]){
            res = 1;
            break;
        }else if (buf1[i] < buf2[i]){
            res = -1;
            break;
        }
        if(buf1[i] == 0 && buf2[i] == 0)
            break;
    }
    return res;
}

/*
 * stringFind: Find a substring in a string on GPU using one single GPU thread.
 * @buf1: the string it searches in
 * @buf2: the substring
 * @len1: the length of the string in buf1
 * @len2: the length of the string in buf2
 * @pos:  the position of the string in buf1 where the search starts
 *
 * Return
 *  -1 Failed to find the substring
 *  >0 the position of the substring
 */
__device__ static inline int stringFind(const char *buf1, const char *buf2, int len1, int len2, int pos)
{
    int res = -1;
    const char *str1 = buf1 + pos;
    const char *end1 = buf1 + len1;
    const char *str2 = buf2;
    int matched = 0;

    while(str1 != end1){
        if(*str1 == *str2){
            if(matched == 0)
                res = str1 - buf1;
            matched++;
            if(matched == len2)
                break;
            str1++;
            str2++;
            continue;
        }

        if(matched)
            return -1;

        str1++;
    }
    if(matched != len2) res = -1;
    return res;
}

__device__ static inline int stringLen(const char* buf)
{
    int res = 0;
    while(*buf++ != '\0')
        res++;
    return res;
}


/*
 * testCon: evaluate one selection predicate using one GPU thread
 * @buf1: input data to be tested
 * @buf2: the test criterion, usually a number of a string.
 * @size: the size of the input data buf1
 * @type: the type of the input data buf1
 * @rel: >,<, >=, <= or ==.
 *
 * Return:
 *  0 if the input data meets the criteria
 *  1 otherwise
 */

__device__ static inline int testCon(char *buf1, char* buf2, int size, int type, int rel){
    int res = 1;
    if (type == INT){
        if(rel == EQ){
            res = ( *((int*)buf1) == *(((int*)buf2)) );
        }else if (rel == NOT_EQ){
            res = ( *((int*)buf1) != *(((int*)buf2)) );
        }else if (rel == GTH){
            res = ( *((int*)buf1) > *(((int*)buf2)) );
        }else if (rel == LTH){
            res = ( *((int*)buf1) < *(((int*)buf2)) );
        }else if (rel == GEQ){
            res = ( *((int*)buf1) >= *(((int*)buf2)) );
        }else if (rel == LEQ){
            res = ( *((int*)buf1) <= *(((int*)buf2)) );
        }

    }else if (type == FLOAT){
        if(rel == EQ){
            res = ( *((float*)buf1) == *(((float*)buf2)) );
        }else if (rel == NOT_EQ){
            res = ( *((float*)buf1) != *(((float*)buf2)) );
        }else if (rel == GTH){
            res = ( *((float*)buf1) > *(((float*)buf2)) );
        }else if (rel == LTH){
            res = ( *((float*)buf1) < *(((float*)buf2)) );
        }else if (rel == GEQ){
            res = ( *((float*)buf1) >= *(((float*)buf2)) );
        }else if (rel == LEQ){
            res = ( *((float*)buf1) <= *(((float*)buf2)) );
        }

    }else{
        int tmp = stringCmp(buf1,buf2,size);
        if(rel == EQ){
            res = (tmp == 0);
        }else if (rel == NOT_EQ){
            res = (tmp != 0);
        }else if (rel == GTH){
            res = (tmp > 0);
        }else if (rel == LTH){
            res = (tmp < 0);
        }else if (rel == GEQ){
            res = (tmp >= 0);
        }else if (rel == LEQ){
            res = (tmp <= 0);
        }
    }
    return res;
}


/*
 * transform_dict_filter_and: merge the filter for dictionary-compressed predicate into the final filter.
 * @dictFilter: the filter for the dictionary compressed data
 * @dictFact: the compressed fact table column
 * @tupleNum: the number of tuples in the column
 * @filter: the filter for the uncompressed data
 */

__global__ static void transform_dict_filter_and(int * dictFilter, char *dictFact, long tupleNum, int dNum,  int * filter, int byteNum){

    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x*blockDim.x + threadIdx.x;

    int * fact = (int*)(dictFact + sizeof(struct dictHeader));

    int numInt = (tupleNum * byteNum +sizeof(int) - 1) / sizeof(int) ;

    for(long i=offset; i<numInt; i += stride){
        int tmp = fact[i];
        unsigned long bit = 1;

        for(int j=0; j< sizeof(int)/byteNum; j++){
            int t = (bit << ((j+1)*byteNum*8)) -1 - ((1<<(j*byteNum*8))-1);
            int fkey = (tmp & t)>> (j*byteNum*8) ;
            filter[i* sizeof(int)/byteNum + j] &= dictFilter[fkey];
        }
    }
}

__global__ static void transform_dict_filter_init(int * dictFilter, char *dictFact, long tupleNum, int dNum,  int * filter,int byteNum){

    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x*blockDim.x + threadIdx.x;

    int * fact = (int*)(dictFact + sizeof(struct dictHeader));
    int numInt = (tupleNum * byteNum +sizeof(int) - 1) / sizeof(int) ;

    for(long i=offset; i<numInt; i += stride){
        int tmp = fact[i];
        unsigned long bit = 1;

        for(int j=0; j< sizeof(int)/byteNum; j++){
            int t = (bit << ((j+1)*byteNum*8)) -1 - ((1<<(j*byteNum*8))-1);
            int fkey = (tmp & t)>> (j*byteNum*8) ;
            filter[i* sizeof(int)/byteNum + j] = dictFilter[fkey];
        }
    }
}

__global__ static void transform_dict_filter_or(int * dictFilter, char *fact, long tupleNum, int dNum,  int * filter,int byteNum){

    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x*blockDim.x + threadIdx.x;

    int numInt = (tupleNum * byteNum +sizeof(int) - 1) / sizeof(int) ;

    for(long i=offset; i<numInt; i += stride){
        int tmp = ((int *)fact)[i];
        unsigned long bit = 1;

        for(int j=0; j< sizeof(int)/byteNum; j++){
            int t = (bit << ((j+1)*byteNum*8)) -1 - ((1<<(j*byteNum*8))-1);
            int fkey = (tmp & t)>> (j*byteNum*8) ;
            filter[i* sizeof(int)/byteNum + j] |= dictFilter[fkey];
        }
    }
}

/*
 * genScanFilter_dict_init: generate the filter for dictionary-compressed predicate
 */

__global__ static void genScanFilter_dict_init(struct dictHeader *dheader, int colSize, int colType, int dNum, struct whereExp *where, int *dfilter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(int i=tid;i<dNum;i+=stride){
        int fkey = dheader->hash[i];
        con = testCon((char *)&fkey,where->content,colSize,colType,where->relation);
        dfilter[i] = con;
    }
}

__global__ static void genScanFilter_dict_or(struct dictHeader *dheader, int colSize, int colType, int dNum, struct whereExp *where, int *dfilter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(int i=tid;i<dNum;i+=stride){
        int fkey = dheader->hash[i];
        con = testCon((char *)&fkey,where->content,colSize,colType,where->relation);
        dfilter[i] |= con;
    }
}

__global__ static void genScanFilter_dict_and(struct dictHeader *dheader, int colSize, int colType, int dNum, struct whereExp *where, int *dfilter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(int i=tid;i<dNum;i+=stride){
        int fkey = dheader->hash[i];
        con = testCon((char *)&fkey,where->content,colSize,colType,where->relation);
        dfilter[i] &= con;
    }
}

__global__ static void genScanFilter_rle(char *col, int colSize, int colType, long tupleNum, struct whereExp *where, int andOr, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    struct rleHeader *rheader = (struct rleHeader *) col;
    int dNum = rheader->dictNum;

    for(int i = tid; i<dNum; i += stride){
        int fkey = ((int *)(col+sizeof(struct rleHeader)))[i];
        int fcount = ((int *)(col+sizeof(struct rleHeader)))[i + dNum];
        int fpos = ((int *)(col+sizeof(struct rleHeader)))[i + 2*dNum];

        con = testCon((char *)&fkey,where->content,colSize,colType,where->relation);

        for(int k=0;k<fcount;k++){
            if(andOr == AND)
                filter[fpos+k] &= con;
            else
                filter[fpos+k] |= con;
        }

    }
}

__global__ static void genScanFilter_and_in(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 0;
    int vlen = where->vlen;

    for(long i = tid; i < tupleNum; i += stride){
        con = 0;
        for(long j = 0; j < vlen; j++)
            con |= (stringCmp(col + colSize * i, *(char **)where->content + colSize * j, colSize) == 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_in_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 0;

    for(long i = tid; i < tupleNum; i += stride){
        int vlen = *((*(int ***)where->content)[i]);
        char *str_st = (*(char ***)where->content)[i] + sizeof(int);
        con = 0;
        for(long j = 0; j < vlen; j++)
            con |= (stringCmp(col + colSize * i, str_st + colSize * j, colSize) == 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_nin(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 0;
    int vlen = where->vlen;

    for(long i = tid; i < tupleNum; i += stride){
        con = 0;
        for(long j = 0; j < vlen; j++)
            con |= (stringCmp(col + colSize * i, *(char **)where->content + colSize * j, colSize) == 0);
        filter[i] &= (~con);
    }
}

__global__ static void genScanFilter_and_nin_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 0;

    for(long i = tid; i < tupleNum; i += stride){
        int vlen = *((*(int ***)where->content)[i]);
        char *str_st = (*(char ***)where->content)[i] + sizeof(int);
        con = 0;
        for(long j = 0; j < vlen; j++)
            con |= (stringCmp(col + colSize * i, str_st + colSize * j, colSize) == 0);
        filter[i] &= (~con);
    }
}

__global__ static void genScanFilter_and_like(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 1;
    int vlen = where->vlen;

    for(long i = tid; i < tupleNum; i += stride){
        int pos = 0, res;
        con = 1;
        for(long j = 0; j < vlen; j++){
            const char *str1 = col + colSize * i;
            const char *str2 = *(char **)where->content + colSize * j;
            int len1 = stringLen(str1);
            int len2 = stringLen(str2);
            len1 = len1 < colSize ? len1 : colSize;
            len2 = len2 < colSize ? len2 : colSize;

            if(j == 0){
                if(!len2) continue;
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res == 0);
                pos += len2;
            }else if(j == vlen - 1){
                if(!len2) continue;
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res + len2 == len1);
            }else{
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res != -1);
                pos = res + len2;
            }
            if(!con) break;
        }
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_nlike(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 1;
    int vlen = where->vlen;

    for(long i = tid; i < tupleNum; i += stride){
        int pos = 0, res;
        con = 1;
        for(long j = 0; j < vlen; j++){
            const char *str1 = col + colSize * i;
            const char *str2 = *(char **)where->content + colSize * j;
            int len1 = stringLen(str1);
            int len2 = stringLen(str2);
            len1 = len1 < colSize ? len1 : colSize;
            len2 = len2 < colSize ? len2 : colSize;

            if(j == 0){
                if(!len2) continue;
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res == -1);
                pos += len2;
            }else if(j == vlen - 1){
                if(!len2) continue;
                res = stringFind(str1, str2, len1, len2, pos);
                con &= !(res + len2 == len1);
            }else{
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res == -1);
                pos = res + len2;
            }
            if(!con) break;
        }
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_eq(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, where->content, colSize) == 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_neq(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, where->content, colSize) != 0);
        filter[i] &= con;
    }
}


__global__ static void genScanFilter_and_eq_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, str_vec + colSize * i, colSize) == 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_neq_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, str_vec + colSize * i, colSize) != 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_gth(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, where->content, colSize) > 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_gth_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, str_vec + colSize * i, colSize) > 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_lth(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, where->content, colSize) < 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_lth_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, str_vec + colSize * i, colSize) < 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_geq(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, where->content, colSize) >= 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_geq_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, str_vec + colSize * i, colSize) >= 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_leq(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, where->content, colSize) <= 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_leq_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, str_vec + colSize * i, colSize) <= 0);
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_init_int_eq(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] == where;
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_int_eq_idx(char *col, int *idx, int *st, int *ed, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = *st + tid; i < *ed; i += stride){
        con = ((int *)col)[idx[i]] == where;
        filter[idx[i]] = con;
    }

}

__global__ static void genScanFilter_init_int_neq(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] != where;
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_int_eq_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] == where[i];
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_int_neq_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] != where[i];
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_float_eq(char *col, long tupleNum, float where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] == where;
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_float_neq(char *col, long tupleNum, float where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] != where;
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_float_eq_vec(char *col, long tupleNum, float *where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] == where[i];
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_float_neq_vec(char *col, long tupleNum, float *where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] != where[i];
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_int_gth(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] > where;
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_int_gth_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] > where[i];
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_float_gth(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        // printf("((float*)col)[i] = %f\n", ((float*)col)[i]);
        con = ((float*)col)[i] > where;
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_float_gth_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] > where[i];
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_int_lth(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] < where;
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_int_lth_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] < where[i];
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_float_lth(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] < where;
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_float_lth_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] < where[i];
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_int_geq(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] >= where;
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_int_geq_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] >= where[i];
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_float_geq(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] >= where;
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_float_geq_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] >= where[i];
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_int_leq(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] <= where;
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_int_leq_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] <= where[i];
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_float_leq(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] <= where;
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_float_leq_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] <= where[i];
        filter[i] = con;
    }
}

__global__ static void genScanFilter_and_int_eq(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] == where;
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_int_neq(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] != where;
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_int_eq_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] == where[i];
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_int_neq_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] != where[i];
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_float_eq(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] == where;
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_float_neq(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] != where;
        filter[i] &= con;
    }
}


__global__ static void genScanFilter_and_float_eq_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] == where[i];
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_float_neq_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] != where[i];
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_int_geq(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] >= where;
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_int_geq_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] >= where[i];
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_float_geq(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] >= where;
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_float_geq_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] >= where[i];
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_int_leq(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] <= where;
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_int_leq_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] <= where[i];
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_float_leq(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] <= where;
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_float_leq_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] <= where[i];
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_int_gth(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] > where;
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_int_gth_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] > where[i];
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_float_gth(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] > where;
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_float_gth_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] > where[i];
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_int_lth(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] < where;
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_int_lth_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] < where[i];
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_float_lth(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] < where;
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_and_float_lth_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] < where[i];
        filter[i] &= con;
    }
}

__global__ static void genScanFilter_init_in(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 0;
    int vlen = where->vlen;

    for(long i = tid; i < tupleNum; i += stride){
        con = 0;
        for(long j = 0; j < vlen; j++)
            con |= (stringCmp(col + colSize * i, *(char **)where->content + colSize * j, colSize) == 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_in_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    // int stride = blockDim.x * gridDim.x;
    // int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // int con = 0;

    // for(long i = tid; i < tupleNum; i += stride){
    //     int vlen = *((*(int ***)where->content)[i]);
    //     char *str_st = (*(char ***)where->content)[i] + sizeof(int);
    //     con = 0;
    //     for(long j = 0; j < vlen; j++)
    //         con |= (stringCmp(col + colSize * i, str_st + colSize * j, colSize) == 0);
    //     filter[i] = con;
    // }

    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 0;
    // printf("tupleNum =%d\n", tupleNum);
    for(long i = tid; i < tupleNum; i += stride){
        int vlen = where->num;
        // printf("[in_vec scan] in_vec num = %d\n", vlen);
        char *str_st = where->column;
        // char *str_st = (*(char ***)where->content)[0] + sizeof(int);
        con = 0;
        // printf("con = %d\n", con);
        for(long j = 0; j < vlen; j++){
            con |= (stringCmp(col + colSize * i, str_st + colSize * j, colSize) == 0);
        }
        filter[i] = con;
        // if(con == 1)
        //     printf("[%d] = 1\n", i);
    }
}


__global__ static void genScanFilter_init_nin(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 0;
    int vlen = where->vlen;

    for(long i = tid; i < tupleNum; i += stride){
        con = 0;
        for(long j = 0; j < vlen; j++)
            con |= (stringCmp(col + colSize * i, *(char **)where->content + colSize * j, colSize) == 0);
        filter[i] = ~con;
    }
}

__global__ static void genScanFilter_init_nin_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 0;

    for(long i = tid; i < tupleNum; i += stride){
        int vlen = *((*(int ***)where->content)[i]);
        char *str_st = (*(char ***)where->content)[i] + sizeof(int);
        con = 0;
        for(long j = 0; j < vlen; j++)
            con |= (stringCmp(col + colSize * i, str_st + colSize * j, colSize) == 0);
        filter[i] = ~con;
    }
}

__global__ static void genScanFilter_init_like(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 1;
    int vlen = where->vlen;

    for(long i = tid; i < tupleNum; i += stride){
        int pos = 0, res;
        con = 1;
        for(long j = 0; j < vlen; j++){
            const char *str1 = col + colSize * i;
            const char *str2 = *(char **)where->content + colSize * j;
            int len1 = stringLen(str1);
            int len2 = stringLen(str2);
            len1 = len1 < colSize ? len1 : colSize;
            len2 = len2 < colSize ? len2 : colSize;

            if(j == 0){
                if(!len2) continue;
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res == 0);
                pos += len2;
            }else if(j == vlen - 1){
                if(!len2) continue;
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res + len2 == len1);
            }else{
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res != -1);
                pos = res + len2;
            }
            if(!con) break;
        }
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_nlike(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 1;
    int vlen = where->vlen;

    for(long i = tid; i < tupleNum; i += stride){
        int pos = 0, res;
        con = 1;
        for(long j = 0; j < vlen; j++){
            const char *str1 = col + colSize * i;
            const char *str2 = *(char **)where->content + colSize * j;
            int len1 = stringLen(str1);
            int len2 = stringLen(str2);
            len1 = len1 < colSize ? len1 : colSize;
            len2 = len2 < colSize ? len2 : colSize;

            if(j == 0){
                if(!len2) continue;
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res == -1);
                pos += len2;
            }else if(j == vlen - 1){
                if(!len2) continue;
                res = stringFind(str1, str2, len1, len2, pos);
                con &= !(res + len2 == len1);
            }else{
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res == -1);
                pos = res + len2;
            }
            if(!con) break;
        }
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_eq(char *col, int colSize,long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, where->content,colSize) == 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_neq(char *col, int colSize,long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, where->content,colSize) != 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_eq_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, str_vec + colSize * i,colSize) == 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_neq_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, str_vec + colSize * i,colSize) != 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_gth(char *col, int colSize,long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, where->content,colSize) > 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_gth_vec(char *col, int colSize,long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, str_vec + colSize * i,colSize) > 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_lth(char *col, int colSize,long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, where->content,colSize) < 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_lth_vec(char *col, int colSize,long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, str_vec + colSize * i, colSize) < 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_geq(char *col, int colSize,long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, where->content,colSize) >= 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_geq_vec(char *col, int colSize,long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, str_vec + colSize * i, colSize) >= 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_leq(char *col, int colSize,long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, where->content,colSize) <= 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_init_leq_vec(char *col, int colSize,long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize*i, str_vec + colSize * i, colSize) <= 0);
        filter[i] = con;
    }
}

__global__ static void genScanFilter_or_in(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 0;
    int vlen = where->vlen;

    for(long i = tid; i < tupleNum; i += stride){
        con = 0;
        for(long j = 0; j < vlen; j++)
            con |= (stringCmp(col + colSize * i, *(char **)where->content + colSize * j, colSize) == 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_in_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 0;

    for(long i = tid; i < tupleNum; i += stride){
        int vlen = *((*(int ***)where->content)[i]);
        char *str_st = (*(char ***)where->content)[i] + sizeof(int);
        con = 0;
        for(long j = 0; j < vlen; j++)
            con |= (stringCmp(col + colSize * i, str_st + colSize * j, colSize) == 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_nin(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 0;
    int vlen = where->vlen;

    for(long i = tid; i < tupleNum; i += stride){
        con = 0;
        for(long j = 0; j < vlen; j++)
            con |= (stringCmp(col + colSize * i, *(char **)where->content + colSize * j, colSize) == 0);
        filter[i] |= ~con;
    }
}

__global__ static void genScanFilter_or_nin_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 0;

    for(long i = tid; i < tupleNum; i += stride){
        int vlen = *((*(int ***)where->content)[i]);
        char *str_st = (*(char ***)where->content)[i] + sizeof(int);
        con = 0;
        for(long j = 0; j < vlen; j++)
            con |= (stringCmp(col + colSize * i, str_st + colSize * j, colSize) == 0);
        filter[i] |= ~con;
    }
}

__global__ static void genScanFilter_or_like(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 1;
    int vlen = where->vlen;

    for(long i = tid; i < tupleNum; i += stride){
        int pos = 0, res;
        con = 1;
        for(long j = 0; j < vlen; j++){
            const char *str1 = col + colSize * i;
            const char *str2 = *(char **)where->content + colSize * j;
            int len1 = stringLen(str1);
            int len2 = stringLen(str2);
            len1 = len1 < colSize ? len1 : colSize;
            len2 = len2 < colSize ? len2 : colSize;

            if(j == 0){
                if(!len2) continue;
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res == 0);
                pos += len2;
            }else if(j == vlen - 1){
                if(!len2) continue;
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res + len2 == len1);
            }else{
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res != -1);
                pos = res + len2;
            }
            if(!con) break;
        }
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_nlike(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con = 1;
    int vlen = where->vlen;

    for(long i = tid; i < tupleNum; i += stride){
        int pos = 0, res;
        con = 1;
        for(long j = 0; j < vlen; j++){
            const char *str1 = col + colSize * i;
            const char *str2 = *(char **)where->content + colSize * j;
            int len1 = stringLen(str1);
            int len2 = stringLen(str2);
            len1 = len1 < colSize ? len1 : colSize;
            len2 = len2 < colSize ? len2 : colSize;

            if(j == 0){
                if(!len2) continue;
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res == -1);
                pos += len2;
            }else if(j == vlen - 1){
                if(!len2) continue;
                res = stringFind(str1, str2, len1, len2, pos);
                con &= !(res + len2 == len1);
            }else{
                res = stringFind(str1, str2, len1, len2, pos);
                con &= (res == -1);
                pos = res + len2;
            }
            if(!con) break;
        }
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_eq(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize *i, where->content, colSize) == 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_neq(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize *i, where->content, colSize) != 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_eq_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize *i, str_vec + colSize * i, colSize) == 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_neq_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize *i, str_vec + colSize * i, colSize) != 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_gth(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize *i, where->content, colSize)> 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_gth_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize *i, str_vec + colSize * i, colSize)> 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_lth(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize *i, where->content, colSize) < 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_lth_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize *i, str_vec + colSize * i, colSize) < 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_geq(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize *i, where->content, colSize) >= 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_geq_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize *i, str_vec + colSize * i, colSize) >= 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_leq(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize *i, where->content, colSize) <= 0);
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_leq_vec(char *col, int colSize, long tupleNum, struct whereExp * where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    char *str_vec = *((char **) where->content);
    for(long i = tid; i<tupleNum;i+=stride){
        con = (stringCmp(col+colSize *i, str_vec + colSize * i, colSize) <= 0);
        filter[i] |= con;
    }
}


__global__ static void genScanFilter_or_int_eq(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] == where;
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_int_neq(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] != where;
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_int_eq_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] == where[i];
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_int_neq_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] != where[i];
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_float_eq(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] == where;
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_float_neq(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] != where;
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_float_eq_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] == where[i];
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_float_neq_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] != where[i];
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_int_gth(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] > where;
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_int_gth_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] > where[i];
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_float_gth(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] > where;
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_float_gth_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] > where[i];
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_int_lth(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] < where;
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_int_lth_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] < where[i];
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_float_lth(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] < where;
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_float_lth_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] < where[i];
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_int_geq(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] >= where;
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_int_geq_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] >= where[i];
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_float_geq(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] >= where;
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_float_geq_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] >= where[i];
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_int_leq(char *col, long tupleNum, int where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] <= where;
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_int_leq_vec(char *col, long tupleNum, int *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((int*)col)[i] <= where[i];
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_float_leq(char *col, long tupleNum, float where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] <= where;
        filter[i] |= con;
    }
}

__global__ static void genScanFilter_or_float_leq_vec(char *col, long tupleNum, float *where, int * filter){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con;

    for(long i = tid; i<tupleNum;i+=stride){
        con = ((float*)col)[i] <= where[i];
        filter[i] |= con;
    }
}

/*
 * countScanNum: count the number of results that each thread generates.
 */

__global__ static void countScanNum(int *filter, long tupleNum, int * count){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int localCount = 0;

    for(long i = tid; i<tupleNum; i += stride){
        localCount += filter[i];
    }

    count[tid] = localCount;

}

__global__ static void countScanNum_idx(int *filter, int *index, int *st, int *ed, int *count)
{
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int localCount = 0;

    for(long i = tid + *st; i< *ed; i += stride){
        localCount += filter[index[i]];
    }

    count[tid] = localCount;
}

/*
 * scan_dict_other: generate the result for dictionary-compressed column.
 */

__global__ static void scan_dict_other(char *col, struct dictHeader * dheader, int byteNum,int colSize, long tupleNum, int *psum, long resultNum, int * filter, char * result){

    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = psum[tid] * colSize;

    for(long i = tid; i<tupleNum; i+= stride){
        if(filter[i] == 1){
            int key = 0;
            memcpy(&key, col + sizeof(struct dictHeader) + i* dheader->bitNum/8, dheader->bitNum/8);
            memcpy(result+pos,&dheader->hash[key],colSize);
            pos += colSize;
        }
    }
}

__global__ static void scan_dict_int(char *col, struct dictHeader * dheader,int byteNum,int colSize, long tupleNum, int *psum, long resultNum, int * filter, char * result){

    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int localCount = psum[tid];

    for(long i = tid; i<tupleNum; i+= stride){
        if(filter[i] == 1){
            int key = 0;
            memcpy(&key, col + sizeof(struct dictHeader) + i*byteNum, byteNum);
            ((int *)result)[localCount] = dheader->hash[key];
            localCount ++;
        }
    }
}

/*
 * scan_other: generate scan result for uncompressed column.
 */

__global__ static void scan_other(char *col, int colSize, long tupleNum, int *psum, long resultNum, int * filter, char * result){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = psum[tid]  * colSize;

    for(long i = tid; i<tupleNum;i+=stride){

        if(filter[i] == 1){
            memcpy(result+pos,col+i*colSize,colSize);
            pos += colSize;
        }
    }
}

__global__ static void scan_other_idx(char *col, int colSize, int *idx, int *st, int *ed, int *psum, long resultNum, int * filter, char * result){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = psum[tid]  * colSize;

    for(long i = tid + *st; i < *ed;i+=stride){

        if(filter[idx[i]] == 1){
            memcpy(result+pos,col+idx[i]*colSize,colSize);
            pos += colSize;
        }
    }
}


__global__ static void scan_int(char *col, int colSize, long tupleNum, int *psum, long resultNum, int * filter, char * result){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int localCount = psum[tid] ;

    for(long i = tid; i<tupleNum;i+=stride){

        if(filter[i] == 1){
            ((int*)result)[localCount] = ((int*)col)[i];
            localCount ++;
        }
    }
}

__global__ static void scan_int_idx(char *col, int colSize, int *idx, int *st, int *ed, int *psum, long resultNum, int * filter, char * result){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int localCount = psum[tid] ;

    for(long i = tid + *st; i < *ed;i+=stride){
        if(filter[idx[i]] == 1){
            ((int*)result)[localCount] = ((int*)col)[idx[i]];
            localCount ++;
        }
    }
}

__global__ void static unpack_rle(char * fact, char * rle, long tupleNum, int dNum){

    int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i=offset; i<dNum; i+=stride){

        int fvalue = ((int *)(fact+sizeof(struct rleHeader)))[i];
        int fcount = ((int *)(fact+sizeof(struct rleHeader)))[i + dNum];
        int fpos = ((int *)(fact+sizeof(struct rleHeader)))[i + 2*dNum];

        for(int k=0;k<fcount;k++){
            ((int*)rle)[fpos+ k] = fvalue;
        }
    }
}

/*
 * Construct bitmap buffer based on the index results.
 */
__global__ void static indexScanPackResult(int* posIdx_d, int* bitmapRes_d, int l_offset, int h_offset, int tupleNum){
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int pos;
    for(int i = l_offset+tid; i<=h_offset; i+=stride){
        if (l_offset<=i && i<=h_offset){
            pos = posIdx_d[i];
            bitmapRes_d[pos] = 1;
        }
    }
}

/*
 * Performs index scan
 */
 int indexScanInt (struct tableNode *tn, int columnPos, int idxPos, int filterValue, int* bitmapRes_d, struct statistic *pp){

    //Check assumption (INT enum == 4)
    if (tn->attrType[columnPos] != 4 ){
        printf("[ERROR] Indexing is only supported for INT type!\n");
        exit(-1);
    }
    if (tn->attrSize[columnPos] != sizeof(int)){
        printf("[ERROR] Indexing is only supported for INT type (and size!)!\n");
        exit(-1); 
    }


    //Start timer for Index Step 1 - Get index position 
    struct timespec startInxS1, endInxS1;
    clock_gettime(CLOCK_REALTIME,&startInxS1);

    //Get data size
    long dataSize = tn->tupleNum * tn->attrSize[columnPos];

    //Get index
    int* contentIdx_d;
    if (tn->indexPos == MEM){
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&contentIdx_d, dataSize));      
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(contentIdx_d, tn->contentIdx[idxPos], dataSize, hipMemcpyHostToDevice));
    }else if (tn->indexPos == GPU){
        contentIdx_d = tn->contentIdx[idxPos]; 
    }else{
        printf("[ERROR] Index can be on either on host or device!\n");
        exit(-1);
    }

    //Stop timer for Index Step 1 - Get index position 
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endInxS1);
    pp->getIndexPos_idxS1 += (endInxS1.tv_sec - startInxS1.tv_sec)* BILLION + endInxS1.tv_nsec - startInxS1.tv_nsec;    

    
    // Implemenation with 2 binary searches
    // ----------------------------------
    // bool exists = thrust::binary_search(thrust::device, contentIdx_d, contentIdx_d + tn->tupleNum, filterValue); 
    // int l_offset = -1; 
    // int h_offset = -1;
    // if (exists){
    //     thrust::pair<int *, int *> range;
    //     range = thrust::equal_range(thrust::device, contentIdx_d, contentIdx_d + tn->tupleNum, filterValue); 
    //     l_offset = (int) (range.first - contentIdx_d); 
    //     h_offset = (int) (range.second - contentIdx_d) - 1;
    // }
    // ----------------------------------

    // Implemenation with 1 binary search
    // ---------------------------------- 
    
    //Start timer for Index Step 2 - Get range
    struct timespec startInxS2, endInxS2;
    clock_gettime(CLOCK_REALTIME,&startInxS2);
    
    //Get range
    thrust::pair<int *, int *> range; 
    range = thrust::equal_range(thrust::device, contentIdx_d, contentIdx_d + tn->tupleNum, filterValue);     
    int l_offset = -1; 
    int h_offset = -1;

    //Stop timer for Index Step 2 - Get range
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endInxS2);
    pp->getRange_idxS2 += (endInxS2.tv_sec - startInxS2.tv_sec)* BILLION + endInxS2.tv_nsec - startInxS2.tv_nsec;    


    //Start timer for Index Step 3 - Convert mem addrs to elements
    struct timespec startInxS3, endInxS3;
    clock_gettime(CLOCK_REALTIME,&startInxS3);

    //Check if value exists
    bool exists = false; 
    int* firstRangeValue = (int *) malloc(sizeof(int));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(firstRangeValue, range.first, sizeof(int), hipMemcpyDeviceToHost));  
    if (firstRangeValue[0] == filterValue){
        exists = true;
    }

    //Convert addrs to elements
    if (exists){ 
        l_offset = (int) (range.first - contentIdx_d); 
        h_offset = (int) (range.second - contentIdx_d) - 1;
    }  
    
    //Calculate number of selected nodes
    int countResult = (h_offset + 1) - l_offset ;
    
    //Stop timer for Index Step 3 - Convert mem addrs to elements
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endInxS3);
    pp->convertMemToElement_idxS3 += (endInxS3.tv_sec - startInxS3.tv_sec)* BILLION + endInxS3.tv_nsec - startInxS3.tv_nsec;    
        
    // ----------------------------------


    //Start timer for Index Step 4 - Get mapping position 
    struct timespec startInxS4, endInxS4;
    clock_gettime(CLOCK_REALTIME,&startInxS4);

    //Get mapping pos
    int* posIdx_d;
    if (tn->indexPos == MEM){
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&posIdx_d, dataSize));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(posIdx_d, tn->posIdx[idxPos], dataSize, hipMemcpyHostToDevice));  
    }else if (tn->indexPos == GPU){
        posIdx_d = tn->posIdx[idxPos];
    }else{
        printf("[ERROR] Index mapping can be on either on host or device!\n");
        exit(-1);
    }

    //Stop timer for Index Step 4 - Get mapping position 
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endInxS4);
    pp->getMapping_idxS4 += (endInxS4.tv_sec - startInxS4.tv_sec)* BILLION + endInxS4.tv_nsec - startInxS4.tv_nsec;    


    //Start timer for Index Step 5 - Set bitmap to zero
    struct timespec startInxS5, endInxS5;
    clock_gettime(CLOCK_REALTIME,&startInxS5);

    //Set everything to false
    CUDA_SAFE_CALL_NO_SYNC(hipMemset(bitmapRes_d, 0, sizeof(int)* tn->tupleNum)); 

    //Stop timer for Index Step 5 - Set bitmap to zero
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endInxS5);
    pp->setBitmapZeros_idxS5 += (endInxS5.tv_sec - startInxS5.tv_sec)* BILLION + endInxS5.tv_nsec - startInxS5.tv_nsec;    

    //Start timer for Index Step 6 - Build bitmap
    struct timespec startInxS6, endInxS6;
    clock_gettime(CLOCK_REALTIME,&startInxS6);

    //Define Grid and block size
    dim3 grid(2048);
    dim3 block(256);  

    //Construct bitmap filter result
    if (exists){
        indexScanPackResult<<<grid,block>>>(posIdx_d, bitmapRes_d, l_offset, h_offset, tn->tupleNum);
    }

    //Stop timer for Index Step 6 - Build bitmap
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endInxS6);
    pp->buildBitmap_idxS6 += (endInxS6.tv_sec - startInxS6.tv_sec)* BILLION + endInxS6.tv_nsec - startInxS6.tv_nsec;    

    //Return selected rows
    return countResult;
}

/*
 * tableScan Prerequisites:
 *  1. the input data can be fit into GPU device memory
 *  2. input data are stored in host memory
 *
 * Input:
 *  sn: contains the data to be scanned and the predicate information
 *  pp: records statistics such kernel execution time and PCIe transfer time
 *
 * Output:
 *  A new table node
 */


__global__ static void pr_int_array(int *arr, int size)
{
    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = offset; i < size; i += stride)
        printf("%d, ", arr[i]);
    printf("\n");
}

__global__ static void set_zero(int *filter, long  inNum){
    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    //int global_index = threadIdx.x + blockDim.x * threadIdx.y;
    for (int i = offset; i<inNum; i += stride){
        filter[i] = 0;
    }
}

__global__ void andArrays(int* arr1, const int* arr2, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        arr1[index] = arr1[index] & arr2[index];
    }
}

__global__ void L2distance(float* res, const float* search, const char * data, int dim, int num) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    float * dataf = (float *)data;

    if (idx < num) {
        float distance = 0.0;

        for (int i = 0; i < dim; ++i) {
            float diff = search[i] - dataf[idx * dim + i];
            distance += diff * diff;
        }

        res[idx] = sqrt(distance);
        // printf("[L2distance] res[%d] = %f\n", idx, res[idx]);
    }

    // if(idx == 0){
    //     for (int i = 0; i < 10; ++i) {
    //         float f = dataf[i];
    //         printf("d[%d] = %f   ", i, f);
    //         if(i % 10 == 0)
    //             printf("\n");
    //     }
    //     printf("\n");
    // }
}

__global__ void twoColL2Distance(char * res, char * col0, char * col1, int dim, int num) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float * f0 = (float *)col0;
    float * f1 = (float *)col1;
    float * fres = (float *)res;
    
    if (idx < num) {
        float distance = 0.0;

        for (int i = 0; i < dim; ++i) {
            float diff = f0[idx * dim + i] - f1[idx * dim + i];
            distance += diff * diff;
        }

        fres[idx] = distance;
        // printf("%f\n", distance);
    }
}

// std::vector<int> andVectors(const std::vector<int>& v1, const std::vector<int>& v2) {
//     std::vector<int> result;

//     for (std::size_t i = 0; i < v1.size(); ++i) {
//         result.push_back(v1[i] && v2[i]);
//     }

//     return result;
// }

__global__ void genScanFilter_and_distance(const char* col1, const char* col2, size_t vectorSize, int totalTupleNum, float threshold, int* filter) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < totalTupleNum) {
        const float* vec1 = (float*)col1 + index * vectorSize;
        const float* vec2 = (float*)col2 + index * vectorSize;
        float distance = 0.0f;

        for (size_t i = 0; i < vectorSize; ++i) {
            float diff = vec1[i] - vec2[i];
            distance += diff * diff;
        }

        distance = sqrtf(distance);

        if (distance < threshold) {
            filter[index] = 1;
        } else {
            filter[index] = 0;
        }
    }
}

__global__ void setFilter(int* globalFilter, size_t* nns, int k) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    globalFilter[idx] = 0;

    for (int i = 0; i < k; ++i) {
        if (idx == nns[i]) {
            globalFilter[idx] = 1;
            break;
        }
    }
}

__global__ void setZeroKernel(int* globalFilter, size_t numElements) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements) {
        globalFilter[idx] = 0;
    }
}


__global__ void setIndicesKernel(int* globalFilter, const size_t* nns, size_t k) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < k) {
        globalFilter[nns[idx]] = 1;
    }
}


struct distanceFilter {
    int colIndex;
    int nearestk;
    std::vector<float> searchVec; 
    int *filter;
};

struct distanceTwoCol {
    int firstColIndex;
    int secondColIndex;
    int projectIndex;
};


struct tableNode * tableScan(struct scanNode *sn, struct statistic *pp,
                             Mempool *host_mp = NULL, Mempool *dev_mp = NULL, Mempool *res_mp = NULL, int *idx = NULL, int *st = NULL, int *ed = NULL, faiss::Index *faiss_index = NULL){

    //Start timer (total tableScan())
    struct timespec startTableScanTotal,endTableScanTotal;
    clock_gettime(CLOCK_REALTIME,&startTableScanTotal);

    struct timespec annsAddStart, annsAddEnd;
    struct timespec annsSearchStart, annsSearchEnd;
    struct timespec readIndexStart, readIndexEnd;
    struct timespec cpu2gpuStart, cpu2gpuEnd;
    struct timespec annsStart, annsEnd;
    struct timespec s, e;
    double tt;
    clock_gettime(CLOCK_REALTIME,&s);

    // char intArray[16384*16];
    // printf("[tableScan.cu] the type of result[0] :%d\n", sn->tn->attrType[0]);
    // printf("[tableScan.cu] the type of result[1] :%d\n", sn->tn->attrType[1]);
    // hipMemcpy(intArray, sn->tn->content[1], sizeof(float) * 100, hipMemcpyDeviceToHost);
    // for(int i = 0; i < 100; i++){
    //     printf("\t%f", intArray[i]);
    //     if((i+1) % 10 == 0)
    //         printf("\n");
	// }

    printf("\n================ [tableScan.cu] Start ================\n");
    int numProject = sn->projectNum;
    // printf("[tableScan.cu] numProject = %d\n", numProject);
    int numOutputAttr = sn->outputNum;
    // printf("[tableScan.cu] res->attrNum = %d\n", numOutputAttr);

    int disFilterCount = 0, disTwoColProject = 0;
    for (int i = 0; i < numProject; i++) {
        // printf("[tableScan.cu] sn->project->exp[%d].funcType = %d\n", i, sn->project->exp[i].funcType);
        if(sn->project->exp[i].func == DISTANCE and (sn->project->exp[i].funcType == 0 or  sn->project->exp[i].funcType == 1)){
            disFilterCount += 1;
        }
        else if (sn->project->exp[i].func == DISTANCE and sn->project->exp[i].funcType == 2) {
            disTwoColProject += 1;
        }
        
    }
    printf("[tableScan.cu] disFilterCount = %d, disTwoColProject = %d\n", disFilterCount, disTwoColProject);


    struct distanceFilter *dFilter = (struct distanceFilter *)malloc(sizeof(struct distanceFilter) * disFilterCount);
    std::vector<float> searchV;
    struct distanceTwoCol *dTwoCol = (struct distanceTwoCol *)malloc(sizeof(struct distanceTwoCol) * disTwoColProject);
    int ctmp1 = 0, ctmp2 = 0;
    for (int i = 0; i < numProject; i++) {
        if(sn->project->exp[i].func == DISTANCE and (sn->project->exp[i].funcType == 0 or sn->project->exp[i].funcType == 1)){
            /* 
                funType = 0: DISTANCE(v1, EXTRACTION(path))
                funType = 1: DISTANCE(v1, path)
                project one vector column to one float column (both of these two types need filter first)
            */
            dFilter[ctmp1].colIndex = sn->project->exp[i].index;
            int limit = sn->project->exp[i].limit;
            if(limit == -1) //pass limit
                dFilter[ctmp1].nearestk = 5;
            else
                dFilter[ctmp1].nearestk = min(limit * 5, 1024);
            // readFloatArray(dFilter[ctmp1].searchVec, sn->project->exp[i].vecPath);
            readFloatArray(searchV, sn->project->exp[i].vecPath);
            // printf("searchV");
            // printf("%f", searchV[0]);
            // dFilter[ctmp1].searchVec.assign(searchV.begin(), searchV.end());
            // for(int i = 0; i < searchV.size(); i++){
            //     printf("searchV[%d] = %f\n", i, searchV[i]);
            // }

            ctmp1++;
        } 
        else if (sn->project->exp[i].func == DISTANCE and sn->project->exp[i].funcType == 2) {
            /* 
                funType = 2: DISTANCE(v1, v2)
                project two vector columns to one float column
            */
            dTwoCol[ctmp2].firstColIndex = sn->project->exp[i].index;
            dTwoCol[ctmp2].secondColIndex = sn->project->exp[i].secondIndex;
            dTwoCol[ctmp2].projectIndex = i;
            printf("dTwoCol[%d].projectIndex = %d\n", ctmp2, i);
            ctmp2++;
        }
    }

    clock_gettime(CLOCK_REALTIME,&s);

    // std::vector<int> globalFilter(sn->tn->tupleNum, 1);
    // int* globalFilter = (int*)malloc(sizeof(int) * sn->tn->tupleNum);
    int* globalFilter;
    int has_globalFilter = 0;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&globalFilter, sizeof(int) * sn->tn->tupleNum));
    const int blockSize = 256;
    const int gridSize = (sn->tn->tupleNum + blockSize - 1) / blockSize;
    setZeroKernel<<<gridSize, blockSize>>>(globalFilter, sn->tn->tupleNum);

    // std::vector<int> filter(sn->tn->tupleNum, 0);

    /*
        DISTANCE(PATH, v1)
        @filter: init to 0, set element to 1 while in nearest neighbor search
        @globalFilter: init to 1, globalFilter &= filter
    */
    for (int i = 0; i < disFilterCount; i++) {
        float *feature = (float *)(sn->tn->content[dFilter[i].colIndex]);
        
        clock_gettime(CLOCK_REALTIME,&e);
        tt = (e.tv_sec - s.tv_sec)* BILLION + e.tv_nsec - s.tv_nsec;
        printf("tt0 = %f\n", tt/(1000*1000));
        clock_gettime(CLOCK_REALTIME,&s);

        size_t feature_num = (size_t)(sn->tn->tupleNum);
        // initial index
        // size_t length = strlen(sn->tn->attrName[i]);
        char prestr[50] = "../../vector_src/init_vector/";
        char endstr[50] = "_index.faissindex";
        char *tmpstr = strcat(prestr, sn->tn->attrName[i]);
        const char* index_path = strcat( tmpstr, endstr);

        // std::cout << "[tableScan.cu] index path = \"" << index_path << "\"" << std::endl;
        int dev_no = 0;
        int k = dFilter[i].nearestk;
        int nq = 1;  //query num
        std::vector<faiss::idx_t> nns(k * nq);
        std::vector<float> dis(k * nq);
        int disColIndex = dFilter[i].colIndex;
        // if index exist
        //   read index from desk 
        // else
        //   train index and save it

        // anns Start
        clock_gettime(CLOCK_REALTIME,&annsStart);
        if(faiss_index != NULL){
            // searching...
            clock_gettime(CLOCK_REALTIME,&annsSearchStart);
            // index->search(nq, dFilter[i].searchVec.data(), k, dis.data(), nns.data());
            faiss_index->search(nq, searchV.data(), k, dis.data(), nns.data());
            clock_gettime(CLOCK_REALTIME,&annsSearchEnd);
            pp->indexSearch += (annsSearchEnd.tv_sec -  annsSearchStart.tv_sec)* BILLION + annsSearchEnd.tv_nsec - annsSearchStart.tv_nsec;
        }
        else if(fileExists(index_path)) {
            printf("[tableScan.cu] Index file exists\n");
            faiss::gpu::StandardGpuResources resources;

            // read faiss index from index_path
            clock_gettime(CLOCK_REALTIME,&readIndexStart);
            faiss::Index *index = faiss::read_index(index_path);
            clock_gettime(CLOCK_REALTIME,&readIndexEnd);
            pp->loadIndex += (readIndexEnd.tv_sec -  readIndexStart.tv_sec)* BILLION + readIndexEnd.tv_nsec - readIndexStart.tv_nsec;

            // index cpu to gpu
            clock_gettime(CLOCK_REALTIME,&cpu2gpuStart);
            index = faiss::gpu::index_cpu_to_gpu(&resources, dev_no, index);
            clock_gettime(CLOCK_REALTIME,&cpu2gpuEnd);
            pp->indexCPU2GPU += (cpu2gpuEnd.tv_sec -  cpu2gpuStart.tv_sec)* BILLION + cpu2gpuEnd.tv_nsec - cpu2gpuStart.tv_nsec;
            
            // add...
            // clock_gettime(CLOCK_REALTIME,&annsAddStart);
            // index->add(feature_num, feature);
            // clock_gettime(CLOCK_REALTIME,&annsAddEnd);
            // pp->indexAddData += (annsAddEnd.tv_sec -  annsAddStart.tv_sec)* BILLION + annsAddEnd.tv_nsec - annsAddStart.tv_nsec;

            // searching...
            clock_gettime(CLOCK_REALTIME,&annsSearchStart);
            // index->search(nq, dFilter[i].searchVec.data(), k, dis.data(), nns.data());
            index->search(nq, searchV.data(), k, dis.data(), nns.data());
            clock_gettime(CLOCK_REALTIME,&annsSearchEnd);
            pp->indexSearch += (annsSearchEnd.tv_sec -  annsSearchStart.tv_sec)* BILLION + annsSearchEnd.tv_nsec - annsSearchStart.tv_nsec;
        }
        else
        {
            printf("[tableScan.cu] Index doesn't exists\n");
            
            // train index
            printf("Train faiss index\n");
            printf("feature_num = %ld\n", feature_num);
            printf("dn->tn->attrSize[%d] = %d\n", disColIndex, sn->tn->attrSize[disColIndex]);
            int d = sn->tn->attrSize[disColIndex] / sizeof(float);
            printf("dimensions = %d\n", d);
            size_t nb = feature_num;
            printf("disColIndex = %d\n", disColIndex);
            int ncentroids = 16384;
            printf("Initial faiss member parameters\n");
            faiss::gpu::StandardGpuResources resources;
            faiss::gpu::GpuIndexIVFPQConfig config;
            config.device = dev_no;
            int m = 16;
            faiss::gpu::GpuIndexIVFPQ index(&resources, d, ncentroids, m, 8, faiss::METRIC_L2, config);
            index.nprobe = 512;

            double t0 = elapsed();
            printf(" Generating %ld vectors in %dD for training\n", nb, d);

            // train index
            struct timespec trainStart, trainEnd;
            clock_gettime(CLOCK_REALTIME,&trainStart);
            index.train(feature_num, feature);
            clock_gettime(CLOCK_REALTIME, &trainEnd);
            pp->trainIndex += (trainEnd.tv_sec - trainStart.tv_sec)* BILLION + trainEnd.tv_nsec - trainStart.tv_nsec;

            // add data to index
            clock_gettime(CLOCK_REALTIME,&annsAddStart);
            index.add(feature_num, feature);
            clock_gettime(CLOCK_REALTIME,&annsAddEnd);
            pp->indexAddData += (annsAddEnd.tv_sec -  annsAddStart.tv_sec)* BILLION + annsAddEnd.tv_nsec - annsAddStart.tv_nsec;

            // save index
            struct timespec saveStart,saveEnd;
            clock_gettime(CLOCK_REALTIME,&saveStart);
            write_index(faiss::gpu::index_gpu_to_cpu(&index), index_path);
            clock_gettime(CLOCK_REALTIME, &saveEnd);
            pp->saveIndex += (saveEnd.tv_sec - saveStart.tv_sec)* BILLION + saveEnd.tv_nsec - saveStart.tv_nsec;
            
            // searching...
            clock_gettime(CLOCK_REALTIME,&annsSearchStart);
            // index.search(nq, dFilter[i].searchVec.data(), k, dis.data(), nns.data());
            index.search(nq, searchV.data(), k, dis.data(), nns.data());
            clock_gettime(CLOCK_REALTIME,&annsSearchEnd);
            pp->indexSearch += (annsSearchEnd.tv_sec -  annsSearchStart.tv_sec)* BILLION + annsSearchEnd.tv_nsec - annsSearchStart.tv_nsec;
        }

        // anns end
        clock_gettime(CLOCK_REALTIME,&annsEnd);
        pp->annsTotalTime += (annsEnd.tv_sec -  annsStart.tv_sec)* BILLION + annsEnd.tv_nsec - annsStart.tv_nsec;

        // for(int j = 0; j < nns.size(); j++){
        //     globalFilter[nns[j]] = 1;
        //     // printf("nns[%d] = %d\n", j, nns[j]);
        // }
        size_t* nns_dev;
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&nns_dev, sizeof(size_t) * k));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(nns_dev, nns.data(), sizeof(size_t)*k, hipMemcpyHostToDevice));
        // setFilter<<<1024, 1024>>>(globalFilter, nns_dev, k);
        const int blockSize = 256;
        const int gridSize = (sn->tn->tupleNum + blockSize - 1) / blockSize;
        setZeroKernel<<<gridSize, blockSize>>>(globalFilter, sn->tn->tupleNum);
        const int gridSizeForNNS = (k + blockSize - 1) / blockSize;
        setIndicesKernel<<<gridSizeForNNS, blockSize>>>(globalFilter, nns_dev, k);
        has_globalFilter = 1;

        // save filter[] in dFilter
        // dFilter[i].filter = (int*)malloc(sizeof(int) * sn->tn->tupleNum);
        // memcpy(dFilter[i].filter, filter.data(), sizeof(int) * sn->tn->tupleNum);
        
        // globalFilter &= filter
        // std::transform(globalFilter.begin(), globalFilter.end(), filter.begin(), globalFilter.begin(), [](int a, int b) {
        //     return a & b;
        // });
        // globalFilter = andVectors(globalFilter, filter);

        // reset filter to 0
        // std::fill(filter.begin(), filter.end(), 0);
    }

    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME,&e);
    tt = (e.tv_sec - s.tv_sec)* BILLION + e.tv_nsec - s.tv_nsec;
    printf("tt1 = %f\n", tt/(1000*1000));
    clock_gettime(CLOCK_REALTIME,&s);

    /**
     * now we have distance filter : globalFilter
     * next get where filter and fuse all filters 
     **/


    //Start timer for Other - 01 (tableScan)
    struct timespec startS01,endS01;
    clock_gettime(CLOCK_REALTIME,&startS01);

    struct tableNode *res = NULL;
    int tupleSize = 0;

    if(host_mp == NULL){
        res = (struct tableNode *) malloc(sizeof(struct tableNode));
        CHECK_POINTER(res);
    }else
        res = (struct tableNode *) host_mp->alloc(sizeof(struct tableNode));

    res->totalAttr = sn->outputNum;
    // printf("\nsn->outputNum = %d\n", sn->outputNum);

    if(host_mp == NULL) {
        res->attrType = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->attrType);
        res->attrSize = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->attrSize);
        res->attrTotalSize = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->attrTotalSize);
        res->attrIndex = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->attrIndex);
        res->dataPos = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->dataPos);
        res->dataFormat = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->dataFormat);
        res->content = (char **) malloc(sizeof(char *) * res->totalAttr);
        CHECK_POINTER(res->content);
    }else{
        res->attrType = (int *) host_mp->alloc(sizeof(int) * res->totalAttr);
        res->attrSize = (int *) host_mp->alloc(sizeof(int) * res->totalAttr);
        res->attrTotalSize = (int *) host_mp->alloc(sizeof(int) * res->totalAttr);
        res->attrIndex = (int *) host_mp->alloc(sizeof(int) * res->totalAttr);
        res->dataPos = (int *) host_mp->alloc(sizeof(int) * res->totalAttr);
        res->dataFormat = (int *) host_mp->alloc(sizeof(int) * res->totalAttr);
        res->content = (char **) host_mp->alloc(sizeof(char *) * res->totalAttr);
    }

    res->colIdxNum = 0;

    for(int i=0;i<res->totalAttr;i++){
        int index = sn->outputIndex[i];
        res->attrType[i] = sn->tn->attrType[index];
        res->attrSize[i] = sn->tn->attrSize[index];
        // printf("res->attrSize[%d] = %d\n", i, res->attrSize[i]);
    }

    int *gpuCount = NULL, *gpuFilter = NULL, *gpuPsum = NULL;

    dim3 grid(2048);
    dim3 block(256);

    long totalTupleNum = sn->tn->tupleNum;
    // printf("[InputTable] tupleNum = %ld\n", totalTupleNum);
    int blockNum = totalTupleNum / block.x + 1;

    if(blockNum<2048)
        grid = blockNum;

    if(idx != NULL) {
        int h_st, h_ed;
        CUDA_SAFE_CALL( hipMemcpy(&h_st, st, sizeof(int), hipMemcpyDeviceToHost) );
        CUDA_SAFE_CALL( hipMemcpy(&h_ed, ed, sizeof(int), hipMemcpyDeviceToHost) );
        CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());
        grid = (h_ed - h_st) / block.x + 1;
    }

    int threadNum = grid.x * block.x;
    int attrNum = 0;
    if(sn->filter != NULL and sn->hasWhere != 0)
        attrNum = sn->whereAttrNum;

    char **column;
    int *whereFree, *colWherePos;
    int count;
    if(host_mp == NULL){
        // printf("attrNum = %d\n", attrNum);
        column = (char **) malloc(attrNum * sizeof(char *));
        // printf("Breakpoint\n");
        CHECK_POINTER(column);
        // printf("Breakpoint\n");

        whereFree = (int *)malloc(attrNum * sizeof(int));
        CHECK_POINTER(whereFree);

        colWherePos = (int *)malloc(sn->outputNum * sizeof(int));
        CHECK_POINTER(colWherePos);
    }else{
        column = (char **)host_mp->alloc(attrNum * sizeof(char *));
        whereFree = (int *)host_mp->alloc(attrNum * sizeof(int));
        colWherePos = (int *)host_mp->alloc(sn->outputNum * sizeof(int));
    }

    if(sn->filter != NULL and sn->hasWhere != 0){
        printf("sn->hasWhere != 0\n");
        for(int i=0;i<sn->outputNum;i++)
            colWherePos[i] = -1;

        for(int i=0;i<attrNum;i++){
            whereFree[i] = 1;
            for(int j=0;j<sn->outputNum;j++){
                if(sn->whereIndex[i] == sn->outputIndex[j]){
                    whereFree[i] = -1;
                    colWherePos[j] = i;
                }
            }
        }


        if(dev_mp == NULL){
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuFilter,sizeof(int) * totalTupleNum));
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuPsum,sizeof(int)*threadNum));
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuCount,sizeof(int)*threadNum));
        }else{
            gpuFilter = (int *) dev_mp->alloc(sizeof(int) * totalTupleNum);
            gpuPsum   = (int *) dev_mp->alloc(sizeof(int) * threadNum);
            gpuCount  = (int *) dev_mp->alloc(sizeof(int) * threadNum);
        }
    }

    // assert(sn->hasWhere !=0);
    // assert(sn->filter != NULL);
    struct whereCondition *where = sn->filter;

    //Stop timer for Other - 01 (tableScan)
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS01);
    pp->create_tableNode_S01 += (endS01.tv_sec - startS01.tv_sec)* BILLION + endS01.tv_nsec - startS01.tv_nsec;
    //Keeps index col pos
    int idxPos = -1;

    /*
     * The first step is to evaluate the selection predicates and generate a vetor to form the final results.
    */
    if(sn->filter != NULL and sn->hasWhere != 0){
        printf("sn->filter != NULL and sn->hasWhere != 0\n");

        //Start timer for Step 1 - Copy where clause 
        struct timespec startS1, endS1;
        clock_gettime(CLOCK_REALTIME,&startS1);

        struct whereExp * gpuExp = NULL;
        // printf("gpuExp address: %p\n", &gpuExp);
        
        // if(dev_mp == NULL)
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuExp, sizeof(struct whereExp)));
        // else
        //     gpuExp = (struct whereExp *) dev_mp->alloc(sizeof(struct whereExp));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuExp, &where->exp[0], sizeof(struct whereExp), hipMemcpyHostToDevice));
        // printf("gpuExp pointer address: %p\n", gpuExp);
        /*
         * Currently we evaluate the predicate one by one.
         * When consecutive predicates are accessing the same column, we don't release the GPU device memory
         * that store the accessed column until all these predicates have been evaluated. Otherwise the device
         * memory will be released immediately after the corresponding predicate has been evaluated.
         *
         * (@whereIndex, @prevWhere), (@index,  @prevIndex), (@format, @prevFormat) are used to decide
         * whether two consecutive predicates access the same column with the same format.
         */

        int whereIndex = where->exp[0].index;
        int index = sn->whereIndex[whereIndex];
        int prevWhere = whereIndex;
        int prevIndex = index;

        int format = sn->tn->dataFormat[index];
        int prevFormat = format;

        if( (where->exp[0].relation == EQ_VEC || where->exp[0].relation == NOT_EQ_VEC || where->exp[0].relation == GTH_VEC || where->exp[0].relation == LTH_VEC || where->exp[0].relation == GEQ_VEC || where->exp[0].relation == LEQ_VEC) &&
            (where->exp[0].dataPos == MEM || where->exp[0].dataPos == MMAP || where->exp[0].dataPos == PINNED) )
        {
            char vec_addr_g[32];
            size_t vec_size = sn->tn->attrSize[index] * sn->tn->tupleNum;
            CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **) vec_addr_g, vec_size) );
            CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(*((void **)vec_addr_g), *((void **) where->exp[0].content), vec_size, hipMemcpyHostToDevice) );

            /* free memory here? */
            free( *((void **) where->exp[0].content) );
            memcpy(where->exp[0].content, vec_addr_g, 32);
            where->exp[0].dataPos = GPU;
        }

        if( (where->exp[0].relation == IN || where->exp[0].relation == LIKE) &&
            (where->exp[0].dataPos == MEM || where->exp[0].dataPos == MMAP || where->exp[0].dataPos == PINNED) )
        {
            char vec_addr_g[32];
            size_t vec_size = sn->tn->attrSize[index] * where->exp[0].vlen;
            CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **) vec_addr_g, vec_size) );
            CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(*((void **)vec_addr_g), *((void **) where->exp[0].content), vec_size, hipMemcpyHostToDevice) );

            free( *((void **) where->exp[0].content) );
            memcpy(where->exp[0].content, vec_addr_g, 32);
            where->exp[0].dataPos = GPU;
        }

        if( where->exp[0].relation == IN_VEC &&
            (where->exp[0].dataPos == MEM || where->exp[0].dataPos == MMAP || where->exp[0].dataPos == PINNED) )
        {
            // char vec_addr_g[32];
            // size_t vec1_size = sizeof(void *) * sn->tn->tupleNum;
            // void **vec1_addrs = (void **)malloc(vec1_size);
            // for(int i = 0; i < sn->tn->tupleNum; i++) {
            //     // [int: vec_len][char *: string1][char *: string2] ...
            //     size_t vec2_size = *((*(int ***)(where->exp[0].content))[i]) * sn->tn->attrSize[index] + sizeof(int);
            //     CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **) &vec1_addrs[i], vec2_size) );
            //     CUDA_SAFE_CALL_NO_SYNC( hipMemcpy( vec1_addrs[i], (*(char ***)where->exp[0].content)[i], vec2_size, hipMemcpyHostToDevice) );
            //     free( (*(char ***)where->exp[0].content)[i] );
            // }
            // CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **) vec_addr_g, vec1_size) );
            // CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(*((void **)vec_addr_g), vec1_addrs, vec1_size, hipMemcpyHostToDevice) );
            // free(vec1_addrs);

            // free(*(char ***)where->exp[0].content);
            // memcpy(where->exp[0].content, vec_addr_g, 32);
            // where->exp[0].dataPos = GPU;


            clock_gettime(CLOCK_REALTIME,&e);
            tt = (e.tv_sec - s.tv_sec)* BILLION + e.tv_nsec - s.tv_nsec;
            printf("tt1.1 = %f\n", tt/(1000*1000));
            clock_gettime(CLOCK_REALTIME,&s);
            printf("[IN_VEC0]\n");
            // char vec_addr_g[32];
            // size_t vec1_size = sizeof(void *);
            // printf("[IN_VEC0]vec1_size = %ld\n", vec1_size);
            // void **vec1_addrs = (void **)malloc(vec1_size);
            // printf("index = %d\n", index);
            // size_t vec2_size = *((*(int ***)(where->exp[0].content))[0]) * sn->tn->attrSize[index] + sizeof(int);
            // size_t vec2_size = *((*(int ***)(where->exp[0].content))[0]) * sn->tn->attrSize[index];
            

            // distinct
            std::unordered_set<std::string> uniqueElements;
            int dataSize = sn->tn->attrSize[index]; // 每个元素的大小
            int numElements = *((*(int ***)(where->exp[0].content))[0]);
            printf("numElements = %d\n", numElements);
            char *data = (*(char ***)where->exp[0].content)[0] + 4;
            
            for (int i = 0; i < numElements; ++i) {
                // 计算当前元素的起始位置
                int elementStart = i * dataSize;
                
                // 从 binaryData 中提取当前元素的数据
                std::string elementData(data + elementStart, dataSize);

                // 如果数据不在哈希集中，添加它
                if (uniqueElements.find(elementData) == uniqueElements.end()) {
                    uniqueElements.insert(elementData);
                }
            }

            where->exp[0].num = uniqueElements.size();
            printf("where->exp[0].num = %d\n", where->exp[0].num);
            // where->exp[0].num = *((*(int ***)(where->exp[0].content))[0]);

            // for (const std::string& element : uniqueElements) {
            //     std::cout << element << std::endl;
            // }
            std::vector<std::string> setVector(uniqueElements.begin(), uniqueElements.end());
            for (size_t i = 0; i < setVector.size(); ++i) {
                std::cout << setVector[i] << "\n";
            }

            std::string concatenatedStrings;
            for (const auto& str : uniqueElements) {
                concatenatedStrings += str;
            }

            size_t elementsSize = setVector.size() * dataSize;
            printf("elementsSize =%ld\n", elementsSize);
            CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **) &(where->exp[0].column), elementsSize));
            CUDA_SAFE_CALL_NO_SYNC( hipMemcpy( where->exp[0].column, concatenatedStrings.data(), elementsSize, hipMemcpyHostToDevice) );
            // char* host_string = (char*)malloc(2415);
            // hipMemcpy(host_string, where->exp[0].column, elementsSize, hipMemcpyDeviceToHost);
            // for(int i = 0; i < 2415; i++){
            //     printf("%c", host_string[i]);
            // }
            // printf("\n");

            where->exp[0].dataPos = GPU;
            
            // add new grammar
            // CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **) &(where->exp[0].column), vec2_size) );
            // CUDA_SAFE_CALL_NO_SYNC( hipMemcpy( where->exp[0].column, (*(char ***)where->exp[0].content)[0] + 4, vec2_size, hipMemcpyHostToDevice) );

            // printf("[IN_VEC0]vec2_size = %ld\n", vec2_size);
            // CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **) &vec1_addrs[0], vec2_size) );
            // CUDA_SAFE_CALL_NO_SYNC( hipMemcpy( vec1_addrs[0], (*(char ***)where->exp[0].content)[0], vec2_size, hipMemcpyHostToDevice) );
            // free( (*(char ***)where->exp[0].content)[0] );
            
            // CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **) vec_addr_g, vec1_size) );
            // CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(*((void **)vec_addr_g), vec1_addrs, vec1_size, hipMemcpyHostToDevice) );
            // printf("vec1_addrs address: %p\n", &(vec1_addrs));
            // printf("vec1_addrs point address: %p\n", vec1_addrs);
            // printf("vec1_addrs point address: %p\n", vec1_addrs[0]);
            // free(vec1_addrs);

            // free(*(char ***)where->exp[0].content);
            // memcpy(where->exp[0].content, vec_addr_g, 32);
            // printf("content address: %p\n", where->exp[0].content);
            // char *addr;
            // memcpy(&addr, where->exp[0].content, sizeof(char *));
            // printf("address: %p\n", addr);
            // char **g_addr;
            // memcpy(&g_addr, vec_addr_g, sizeof(char **));
            // // printf("g_addr: %p\n", g_addr);
            // printf("g_addr p: %p\n",*g_addr);
            // printf("content point address: %p\n", (*(int ***)where->exp[0].content));
            // int vlen;
            // hipMemcpy(&vlen, *(int**)g_addr, sizeof(int), hipMemcpyDeviceToHost);
            // printf("vlen = %d\n", vlen);
            
        }

        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuExp, &where->exp[0], sizeof(struct whereExp), hipMemcpyHostToDevice));

        //Stop for Step 1 - Copy where clause 
        CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
        clock_gettime(CLOCK_REALTIME, &endS1);
        pp->whereMemCopy_s1 += (endS1.tv_sec - startS1.tv_sec)* BILLION + endS1.tv_nsec - startS1.tv_nsec;
                 
       /*   
         * @dNum, @byteNum and @gpuDictFilter are for predicates that need to access dictionary-compressed columns.
         */
        int dNum;
        int byteNum;
        int *gpuDictFilter = NULL;

        /*
         * We will allocate GPU device memory for a column if it is stored in the host pageable or pinned memory.
         * If it is configured to utilize the UVA technique, no GPU device memory will be allocated.
         */

        if(sn->tn->dataPos[index] == MEM || sn->tn->dataPos[index] == MMAP || sn->tn->dataPos[index] == PINNED)
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &column[whereIndex], sn->tn->attrTotalSize[index]));

        if(format == UNCOMPRESSED){
            
            //Start timer for Step 2 - Copy data
            struct timespec startS2, endS2;
            clock_gettime(CLOCK_REALTIME,&startS2);

            if(sn->tn->dataPos[index] == MEM || sn->tn->dataPos[index] == MMAP || sn->tn->dataPos[index] == PINNED)
                CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[whereIndex], sn->tn->content[index], sn->tn->attrTotalSize[index], hipMemcpyHostToDevice));
            else if (sn->tn->dataPos[index] == UVA || sn->tn->dataPos[index] == GPU)
                column[whereIndex] = sn->tn->content[index];

            //Stop timer for Step 2 - Copy data
            CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
            clock_gettime(CLOCK_REALTIME, &endS2);
            pp->dataMemCopy_s2 += (endS2.tv_sec - startS2.tv_sec)* BILLION + endS2.tv_nsec - startS2.tv_nsec;
                
            // printf("BreakPoint\n");

            int rel = where->exp[0].relation;
            printf("rel =%d\n", rel);
            if(sn->tn->attrType[index] == INT){
                int whereValue;
                int *whereVec;
                if(rel == EQ || rel == NOT_EQ || rel == GTH || rel == LTH || rel == GEQ || rel == LEQ)
                    whereValue = *((int*) where->exp[0].content);
                else
                    whereVec = *((int **) where->exp[0].content);

                if(rel==EQ){
                    //Check if this column is indexed
                    if (sn->tn->colIdxNum != 0){
                        for (int k=0; k<sn->tn->colIdxNum; k++){
                            if (sn->tn->colIdx[k] == index){
                                idxPos = k; 
                            }
                        }
                    }

                    //Start timer for Step 3 - Scan
                    struct timespec startS3, endS3;
                    clock_gettime(CLOCK_REALTIME,&startS3);
                    
                    // if (idxPos >= 0){

                        //Regular scan
                        //genScanFilter_init_int_eq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);

                        //Index scan!
                        // res->tupleNum = indexScanInt(sn->tn, index, idxPos, whereValue, gpuFilter, pp);
                        
                        /* DEBUG CODE - DO NOT REMOVE FOR NOW */
                        // int* originalRes = (int *)malloc(sizeof(int) * totalTupleNum);
                        // CHECK_POINTER(originalRes); 
                        // CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(originalRes, gpuFilter, sizeof(int) * totalTupleNum, hipMemcpyDeviceToHost));
                        // printf("This is what is being returned by serial scan:\n");
                        // for (int i=0;i<totalTupleNum;i++){
                        //     if (originalRes[i] != 0 ){
                        //         printf ("Value[%d]: %d \n",i,originalRes[i]);
                        //     }
                        // }
                        // free(originalRes);
                    if(idx != NULL){
                        genScanFilter_init_int_eq_idx<<<grid, block>>>(column[whereIndex], idx, st, ed, whereValue, gpuFilter);
                    }else{
                        genScanFilter_init_int_eq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                    }

                    //Stop timer for Step 3 - Scan
                    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
                    clock_gettime(CLOCK_REALTIME, &endS3);
                    pp->scanTotal_s3 += (endS3.tv_sec - startS3.tv_sec)* BILLION + endS3.tv_nsec - startS3.tv_nsec;

                }else if(rel == NOT_EQ)
                    genScanFilter_init_int_neq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                else if(rel == GTH)
                    genScanFilter_init_int_gth<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                else if(rel == LTH)
                    genScanFilter_init_int_lth<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                else if(rel == GEQ)
                    genScanFilter_init_int_geq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                else if (rel == LEQ)
                    genScanFilter_init_int_leq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                else if (rel == EQ_VEC)
                    genScanFilter_init_int_eq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                else if (rel == NOT_EQ_VEC)
                    genScanFilter_init_int_neq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                else if (rel == GTH_VEC)
                    genScanFilter_init_int_gth_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                else if (rel == LTH_VEC)
                    genScanFilter_init_int_lth_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                else if (rel == GEQ_VEC)
                    genScanFilter_init_int_geq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                else if (rel == LEQ_VEC)
                    genScanFilter_init_int_leq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);

            }else if (sn->tn->attrType[index] == FLOAT){
                // printf("[INFO] FLOAT filter\n");
                float whereValue, *whereVec;
                if(rel == EQ || rel == NOT_EQ || rel == GTH || rel == LTH || rel == GEQ || rel == LEQ){
                    whereValue = *((float*) where->exp[0].content);
                    printf("whereValue = %f\n", whereValue);
                }
                else
                    whereVec = *((float **) where->exp[0].content);

                // parameters for DISTANCE(v1, v2)
                int firstIndex;
                int secondIndex;
                float parameter;
                if(rel == DISTANCE_LTH_VEC){
                    firstIndex = where->exp[0].index;
                    secondIndex = where->exp[0].secondIndex;
                    parameter = where->exp[0].parameter;
                }

                if(rel==EQ)
                    genScanFilter_init_float_eq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                else if(rel==NOT_EQ)
                    genScanFilter_init_float_neq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                else if(rel == GTH)
                    genScanFilter_init_float_gth<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                else if(rel == LTH)
                    genScanFilter_init_float_lth<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                else if(rel == GEQ)
                    genScanFilter_init_float_geq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                else if (rel == LEQ)
                    genScanFilter_init_float_leq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                else if(rel==EQ_VEC)
                    genScanFilter_init_float_eq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                else if(rel==NOT_EQ_VEC)
                    genScanFilter_init_float_neq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                else if(rel == GTH_VEC)
                    genScanFilter_init_float_gth_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                else if(rel == LTH_VEC)
                    genScanFilter_init_float_lth_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                else if(rel == GEQ_VEC)
                    genScanFilter_init_float_geq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                else if (rel == LEQ_VEC)
                    genScanFilter_init_float_leq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                else if(rel == DISTANCE_LTH_VEC){
                    int size = sn->tn->attrSize[firstIndex]/sizeof(float);
                    genScanFilter_and_distance<<<grid,block>>>(column[firstIndex],column[secondIndex], size, totalTupleNum, parameter, gpuFilter);
                }

            }else if (sn->tn->attrType[index] == VECTOR) {
                if(rel == KNN){
                    int k;
                    memcpy(&k, where->exp[0].content, sizeof(int));
                    // get vecPath, load vec
                    readFloatArray(searchV, where->exp[0].videoFeaturePath);
                    // faiss knn search
                    float *feature = (float *)(sn->tn->content[index]);
                    size_t feature_num = (size_t)(sn->tn->tupleNum);
                    // initial index
                    // size_t length = strlen(sn->tn->attrName[i]);
                    char prestr[50] = "../../vector_src/init_vector/";
                    char endstr[50] = "_index.faissindex";
                    char *tmpstr = strcat(prestr, sn->tn->attrName[index]);
                    const char* index_path = strcat( tmpstr, endstr);
                    std::cout << "[tableScan.cu] index path = \"" << index_path << "\"" << std::endl;
                    int dev_no = 0;
                    int nq = searchV.size() * sizeof(float) / sn->tn->attrSize[index];  //query num
                    // for(int i = 0; i < searchV.size(); i++){
                    //     printf("searchV[%d] = %f\n", i, searchV[i]);
                    // }
                    printf("[KNN]search vector num = %d\n", nq);
                    printf("[KNN]search num = %d\n", k);
                    std::vector<faiss::idx_t> nns(k * nq);
                    std::vector<float> dis(k * nq);
                    int disColIndex = index;
                    // if index exist
                    //   read index from desk 
                    // else
                    //   train index and save it

                    // ANNs start
                    clock_gettime(CLOCK_REALTIME, &annsStart);
                    if(faiss_index != NULL){
                        // searching...
                        clock_gettime(CLOCK_REALTIME,&annsSearchStart);
                        // index->search(nq, dFilter[i].searchVec.data(), k, dis.data(), nns.data());
                        faiss_index->search(nq, searchV.data(), k, dis.data(), nns.data());
                        clock_gettime(CLOCK_REALTIME,&annsSearchEnd);
                        pp->indexSearch += (annsSearchEnd.tv_sec -  annsSearchStart.tv_sec)* BILLION + annsSearchEnd.tv_nsec - annsSearchStart.tv_nsec;
                    }
                    else if (fileExists(index_path)) {
                        // printf("[tableScan.cu] Index file exists\n");
                        faiss::gpu::StandardGpuResources resources;

                        // read faiss index from index_path
                        clock_gettime(CLOCK_REALTIME,&readIndexStart);
                        faiss::Index *index = faiss::read_index(index_path);
                        clock_gettime(CLOCK_REALTIME,&readIndexEnd);
                        pp->loadIndex += (readIndexEnd.tv_sec -  readIndexStart.tv_sec)* BILLION + readIndexEnd.tv_nsec - readIndexStart.tv_nsec;
                        
                        clock_gettime(CLOCK_REALTIME,&cpu2gpuStart);
                        index = faiss::gpu::index_cpu_to_gpu(&resources, dev_no, index);
                        clock_gettime(CLOCK_REALTIME,&cpu2gpuEnd);
                        pp->indexCPU2GPU += (cpu2gpuEnd.tv_sec -  cpu2gpuStart.tv_sec)* BILLION + cpu2gpuEnd.tv_nsec - cpu2gpuStart.tv_nsec;
                        
                        // add...
                        // clock_gettime(CLOCK_REALTIME,&annsAddStart);
                        // index->add(feature_num, feature);
                        // clock_gettime(CLOCK_REALTIME,&annsAddEnd);
                        // pp->indexAddData = (annsAddEnd.tv_sec -  annsAddStart.tv_sec)* BILLION + annsAddEnd.tv_nsec - annsAddStart.tv_nsec;
                        
                        // searching...
                        clock_gettime(CLOCK_REALTIME,&annsSearchStart);
                        // index->search(nq, dFilter[i].searchVec.data(), k, dis.data(), nns.data());
                        index->search(nq, searchV.data(), k, dis.data(), nns.data());
                        clock_gettime(CLOCK_REALTIME,&annsSearchEnd);
                        pp->indexSearch += (annsSearchEnd.tv_sec -  annsSearchStart.tv_sec)* BILLION + annsSearchEnd.tv_nsec - annsSearchStart.tv_nsec;
                    }
                    else
                    {
                        // printf("[tableScan.cu] Index doesn't exists\n");
                        
                        // train index
                        // printf("Train faiss index\n");
                        // printf("feature_num = %ld\n", feature_num);
                        // printf("dn->tn->attrSize[%d] = %d\n", disColIndex, sn->tn->attrSize[disColIndex]);
                        int d = sn->tn->attrSize[disColIndex] / sizeof(float);
                        // printf("dimensions = %d\n", d);
                        size_t nb = feature_num;
                        // printf("disColIndex = %d\n", disColIndex);
                        // int ncentroids = int(sqrt(feature_num));
                        int nlist = 16;
                        // printf("Initial faiss member parameters\n");
                        faiss::gpu::StandardGpuResources resources;
                        faiss::gpu::GpuIndexIVFFlatConfig config;
                        config.device = dev_no;
                        faiss::gpu::GpuIndexIVFFlat index(&resources, d, nlist, faiss::METRIC_L2, config);

                        double t0 = elapsed();
                        // printf(" Generating %ld vectors in %dD for training\n", nb, d);

                        // train index
                        struct timespec trainStart, trainEnd;
                        clock_gettime(CLOCK_REALTIME,&trainStart);
                        index.train(feature_num, feature);
                        clock_gettime(CLOCK_REALTIME,&trainEnd);
                        pp->trainIndex += (trainEnd.tv_sec -  trainStart.tv_sec)* BILLION + trainEnd.tv_nsec - trainStart.tv_nsec;
                        

                        // add data to index
                        clock_gettime(CLOCK_REALTIME,&annsAddStart);
                        index.add(feature_num, feature);
                        clock_gettime(CLOCK_REALTIME,&annsAddEnd);
                        pp->indexAddData += (annsAddEnd.tv_sec -  annsAddStart.tv_sec)* BILLION + annsAddEnd.tv_nsec - annsAddStart.tv_nsec;


                        // save index
                        struct timespec saveStart,saveEnd;
                        clock_gettime(CLOCK_REALTIME,&saveStart);
                        write_index(faiss::gpu::index_gpu_to_cpu(&index), index_path);
                        clock_gettime(CLOCK_REALTIME, &saveEnd);
                        pp->saveIndex += (saveEnd.tv_sec - saveStart.tv_sec)* BILLION + saveEnd.tv_nsec - saveStart.tv_nsec;
                        

                        // searching...
                        clock_gettime(CLOCK_REALTIME,&annsSearchStart);
                        // index.search(nq, dFilter[i].searchVec.data(), k, dis.data(), nns.data());
                        index.search(nq, searchV.data(), k, dis.data(), nns.data());
                        clock_gettime(CLOCK_REALTIME,&annsSearchEnd);
                        pp->indexSearch += (annsSearchEnd.tv_sec -  annsSearchStart.tv_sec)* BILLION + annsSearchEnd.tv_nsec - annsSearchStart.tv_nsec;
                    }

                    // for (int i = 0; i < nq; i++) {
                    //     for (int j = 0; j < k; j++) {
                    //         globalFilter[nns[j + i * k]] = 1;
                    //         // printf("nns[%d] = 1\n", j+i*k);
                    //         // printf("index = %d", nns[j + i * k]);
                    //     }
                    // }
                    size_t* nns_dev;
                    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&nns_dev, sizeof(size_t) * nq * k));
                    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(nns_dev, nns.data(), sizeof(size_t) * nq * k, hipMemcpyHostToDevice));
                    int blockSize = 256;
                    int numBlocks = (totalTupleNum + blockSize - 1) / blockSize;                    
                    setFilter<<<numBlocks, blockSize>>>(globalFilter, nns_dev, nq * k);
                    // CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gFilter, sizeof(int) * sn->tn->tupleNum));
                    has_globalFilter = 1;
                }
            }
            
            else{
                if(rel == EQ)
                    genScanFilter_init_eq<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                else if(rel == NOT_EQ)
                    genScanFilter_init_neq<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                else if (rel == GTH)
                    genScanFilter_init_gth<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                else if (rel == LTH)
                    genScanFilter_init_lth<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                else if (rel == GEQ)
                    genScanFilter_init_geq<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                else if (rel == LEQ)
                    genScanFilter_init_leq<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                else if(rel == EQ_VEC)
                    genScanFilter_init_eq_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                else if(rel == NOT_EQ_VEC)
                    genScanFilter_init_neq_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                else if (rel == GTH_VEC)
                    genScanFilter_init_gth_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                else if (rel == LTH_VEC)
                    genScanFilter_init_lth_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                else if (rel == GEQ_VEC)
                    genScanFilter_init_geq_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                else if (rel == LEQ_VEC)
                    genScanFilter_init_leq_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                else if (rel == IN)
                    genScanFilter_init_in<<<grid, block>>>(column[whereIndex], sn->tn->attrSize[index], totalTupleNum, gpuExp, gpuFilter);
                else if (rel == IN_VEC){
                    printf("sn->tn->attrSize[index] = %d\n", sn->tn->attrSize[index]);
                    printf("whereIndex = %d\n", whereIndex);
                    printf("totalTupleNum = %ld\n", totalTupleNum);
                    clock_gettime(CLOCK_REALTIME,&e);
                    tt = (e.tv_sec - s.tv_sec)* BILLION + e.tv_nsec - s.tv_nsec;
                    printf("tt1.2 = %f\n", tt/(1000*1000));
                    clock_gettime(CLOCK_REALTIME,&s);
                    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
                    genScanFilter_init_in_vec<<<grid, block>>>(column[whereIndex], sn->tn->attrSize[index], totalTupleNum, gpuExp, gpuFilter);
                    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
                }
                else if (rel == LIKE)
                    genScanFilter_init_like<<<grid, block>>>(column[whereIndex], sn->tn->attrSize[index], totalTupleNum, gpuExp, gpuFilter);
            }
        }

        clock_gettime(CLOCK_REALTIME,&e);
        tt = (e.tv_sec - s.tv_sec)* BILLION + e.tv_nsec - s.tv_nsec;
        printf("tt1.3 = %f\n", tt/(1000*1000));
        clock_gettime(CLOCK_REALTIME,&s);

        int dictFilter = 0;
        int dictFinal = OR;
        int dictInit = 1;

        //Start timer for Step 1 - Copy where clause (Part2)
        struct timespec startS12, endS12;
        clock_gettime(CLOCK_REALTIME,&startS12);

        for(int i=1;i<where->expNum;i++){
            whereIndex = where->exp[i].index;
            index = sn->whereIndex[whereIndex];
            format = sn->tn->dataFormat[index];

            // parameters for DISTANCE(v1, v2) < dis
            int firstIndex;
            int secondIndex;
            float parameter;
            if(where->exp[i].relation == DISTANCE_LTH_VEC){
                firstIndex = where->exp[i].index;
                secondIndex = where->exp[i].secondIndex;
                parameter = where->exp[i].parameter;
            }

            if( (where->exp[i].relation == EQ_VEC || where->exp[i].relation == NOT_EQ_VEC || where->exp[i].relation == GTH_VEC || where->exp[i].relation == LTH_VEC || where->exp[i].relation == GEQ_VEC || where->exp[i].relation == LEQ_VEC) &&
                (where->exp[i].dataPos == MEM || where->exp[i].dataPos == MMAP || where->exp[i].dataPos == PINNED) )
            {
                char vec_addr_g[32];
                size_t vec_size = sn->tn->attrSize[index] * sn->tn->tupleNum;
                CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **) vec_addr_g, vec_size) );
                CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(*((void **)vec_addr_g), *((void **) where->exp[i].content), vec_size, hipMemcpyHostToDevice) );

                free( *((void **) where->exp[i].content) );
                memcpy(where->exp[i].content, vec_addr_g, 32);
                where->exp[i].dataPos = GPU;
            }

            if( (where->exp[i].relation == IN || where->exp[i].relation == LIKE) &&
                (where->exp[i].dataPos == MEM || where->exp[i].dataPos == MMAP || where->exp[i].dataPos == PINNED) )
            {
                char vec_addr_g[32];
                size_t vec_size = sn->tn->attrSize[index] * where->exp[i].vlen;
                CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **) vec_addr_g, vec_size) );
                CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(*((void **)vec_addr_g), *((void **) where->exp[i].content), vec_size, hipMemcpyHostToDevice) );

                free( *((void **) where->exp[i].content) );
                memcpy(where->exp[i].content, vec_addr_g, 32);
                where->exp[i].dataPos = GPU;
            }

            if( where->exp[i].relation == IN_VEC &&
                (where->exp[i].dataPos == MEM || where->exp[i].dataPos == MMAP || where->exp[i].dataPos == PINNED) )
            {
                char vec_addr_g[32];
                size_t vec1_size = sizeof(void *) * sn->tn->tupleNum;
                void **vec1_addrs = (void **)malloc(vec1_size);
                for(int j = 0; j < sn->tn->tupleNum; j++) {
                    // [int: vec_len][char *: string1][char *: string2] ...
                    size_t vec2_size = *((*(int ***)(where->exp[i].content))[j]) * sn->tn->attrSize[index] + sizeof(int);
                    CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **) &vec1_addrs[j], vec2_size) );
                    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy( vec1_addrs[j], (*(char ***)where->exp[i].content)[j], vec2_size, hipMemcpyHostToDevice) );
                    free( (*(char ***)where->exp[i].content)[j] );
                }
                CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **) vec_addr_g, vec1_size) );
                CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(*((void **)vec_addr_g), vec1_addrs, vec1_size, hipMemcpyHostToDevice) );
                free(vec1_addrs);

                free(*(char ***)where->exp[i].content);
                memcpy(where->exp[i].content, vec_addr_g, 32);
                where->exp[i].dataPos = GPU;
            }

            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuExp, &where->exp[i], sizeof(struct whereExp), hipMemcpyHostToDevice));

            if(prevIndex != index){

                 /*When the two consecutive predicates access different columns*/

                if(prevFormat == DICT){
                    if(dictInit == 1){
                        transform_dict_filter_init<<<grid,block>>>(gpuDictFilter, column[prevWhere], totalTupleNum, dNum, gpuFilter,byteNum);
                        dictInit = 0;
                    }else if(dictFinal == OR)
                        transform_dict_filter_or<<<grid,block>>>(gpuDictFilter, column[prevWhere], totalTupleNum, dNum, gpuFilter,byteNum);
                    else
                        transform_dict_filter_and<<<grid,block>>>(gpuDictFilter, column[prevWhere], totalTupleNum, dNum, gpuFilter,byteNum);

                    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuDictFilter));
                    dictFinal = where->andOr;
                }

                if(whereFree[prevWhere] == 1 && (sn->tn->dataPos[prevIndex] == MEM || sn->tn->dataPos[prevIndex] == MMAP || sn->tn->dataPos[prevIndex] == PINNED))
                    CUDA_SAFE_CALL_NO_SYNC(hipFree(column[prevWhere]));

                if(sn->tn->dataPos[index] == MEM || sn->tn->dataPos[index] == MMAP || sn->tn->dataPos[index] == PINNED)
                    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &column[whereIndex] , sn->tn->attrTotalSize[index]));

                if(format == DICT){
                    if(sn->tn->dataPos[index] == MEM || sn->tn->dataPos[index] == MMAP || sn->tn->dataPos[index] == PINNED)
                        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[whereIndex], sn->tn->content[index], sn->tn->attrTotalSize[index], hipMemcpyHostToDevice));
                    else if (sn->tn->dataPos[index] == UVA || sn->tn->dataPos[index] == GPU)
                        column[whereIndex] = sn->tn->content[index];

                    struct dictHeader * dheader = (struct dictHeader *)sn->tn->content[index];
                    dNum = dheader->dictNum;
                    byteNum = dheader->bitNum/8;

                    struct dictHeader * gpuDictHeader = NULL;
                    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuDictHeader,sizeof(struct dictHeader)));
                    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuDictHeader,dheader,sizeof(struct dictHeader), hipMemcpyHostToDevice));
                    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuDictFilter, dNum * sizeof(int)));

                    genScanFilter_dict_init<<<grid,block>>>(gpuDictHeader,sn->tn->attrSize[index],sn->tn->attrType[index],dNum, gpuExp,gpuDictFilter);
                    dictFilter= -1;
                    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuDictHeader));

                }else{
                    if(sn->tn->dataPos[index] == MEM || sn->tn->dataPos[index] == MMAP || sn->tn->dataPos[index] == PINNED)
                        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[whereIndex], sn->tn->content[index], sn->tn->attrTotalSize[index], hipMemcpyHostToDevice));
                    else if (sn->tn->dataPos[index] == UVA || sn->tn->dataPos[index] == GPU)
                        column[whereIndex] = sn->tn->content[index];
                }

                prevIndex = index;
                prevWhere = whereIndex;
                prevFormat = format;
            }


            if(format == UNCOMPRESSED){
                int rel = where->exp[i].relation;
                if(sn->tn->attrType[index] == INT){
                    int whereValue;
                    int *whereVec;
                    if(rel == EQ || rel == NOT_EQ || rel == GTH || rel == LTH || rel == GEQ || rel == LEQ)
                        whereValue = *((int*) where->exp[i].content);
                    else
                        whereVec = *((int **) where->exp[i].content);

                    if(where->andOr == AND){
                        if(rel==EQ)
                            genScanFilter_and_int_eq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel==NOT_EQ)
                            genScanFilter_and_int_neq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel == GTH)
                            genScanFilter_and_int_gth<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel == LTH)
                            genScanFilter_and_int_lth<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel == GEQ)
                            genScanFilter_and_int_geq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if (rel == LEQ)
                            genScanFilter_and_int_leq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel==EQ_VEC)
                            genScanFilter_and_int_eq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel==NOT_EQ_VEC)
                            genScanFilter_and_int_neq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel == GTH_VEC)
                            genScanFilter_and_int_gth_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel == LTH_VEC)
                            genScanFilter_and_int_lth_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel == GEQ_VEC)
                            genScanFilter_and_int_geq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if (rel == LEQ_VEC)
                            genScanFilter_and_int_leq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                    }else{
                        if(rel==EQ)
                            genScanFilter_or_int_eq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel==NOT_EQ)
                            genScanFilter_or_int_neq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel == GTH)
                            genScanFilter_or_int_gth<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel == LTH)
                            genScanFilter_or_int_lth<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel == GEQ)
                            genScanFilter_or_int_geq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if (rel == LEQ)
                            genScanFilter_or_int_leq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel==EQ_VEC)
                            genScanFilter_or_int_eq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel==NOT_EQ_VEC)
                            genScanFilter_or_int_neq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel == GTH_VEC)
                            genScanFilter_or_int_gth_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel == LTH_VEC)
                            genScanFilter_or_int_lth_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel == GEQ_VEC)
                            genScanFilter_or_int_geq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if (rel == LEQ_VEC)
                            genScanFilter_or_int_leq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);

                    }

                } else if (sn->tn->attrType[index] == FLOAT){
                    float whereValue, *whereVec;
                    if(rel == EQ || rel == NOT_EQ || rel == GTH || rel == LTH || rel == GEQ || rel == LEQ)
                        whereValue = *((float*) where->exp[i].content);
                    else
                        whereVec = *((float**) where->exp[i].content);

                    if(where->andOr == AND){
                        if(rel==EQ)
                            genScanFilter_and_float_eq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel==NOT_EQ)
                            genScanFilter_and_float_neq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel == GTH)
                            genScanFilter_and_float_gth<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel == LTH)
                            genScanFilter_and_float_lth<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel == GEQ)
                            genScanFilter_and_float_geq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if (rel == LEQ)
                            genScanFilter_and_float_leq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel==EQ_VEC)
                            genScanFilter_and_float_eq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel==NOT_EQ_VEC)
                            genScanFilter_and_float_neq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);    
                        else if(rel == GTH_VEC)
                            genScanFilter_and_float_gth_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel == LTH_VEC)
                            genScanFilter_and_float_lth_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel == GEQ_VEC)
                            genScanFilter_and_float_geq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if (rel == LEQ_VEC)
                            genScanFilter_and_float_leq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if (rel == DISTANCE_LTH_VEC){
                            int size = sn->tn->attrSize[firstIndex]/sizeof(float);
                            genScanFilter_and_distance<<<grid,block>>>(column[firstIndex],column[secondIndex], size, totalTupleNum, parameter, gpuFilter);
                        }
                    }else if(where->andOr == OR){
                        if(rel==EQ)
                            genScanFilter_or_float_eq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel==NOT_EQ)
                            genScanFilter_or_float_neq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel == GTH)
                            genScanFilter_or_float_gth<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel == LTH)
                            genScanFilter_or_float_lth<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel == GEQ)
                            genScanFilter_or_float_geq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if (rel == LEQ)
                            genScanFilter_or_float_leq<<<grid,block>>>(column[whereIndex],totalTupleNum, whereValue, gpuFilter);
                        else if(rel==EQ_VEC)
                            genScanFilter_or_float_eq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel==NOT_EQ_VEC)
                            genScanFilter_or_float_neq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel == GTH_VEC)
                            genScanFilter_or_float_gth_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel == LTH_VEC)
                            genScanFilter_or_float_lth_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if(rel == GEQ_VEC)
                            genScanFilter_or_float_geq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);
                        else if (rel == LEQ_VEC)
                            genScanFilter_or_float_leq_vec<<<grid,block>>>(column[whereIndex],totalTupleNum, whereVec, gpuFilter);

                    }
                }else{
                    if(where->andOr == AND){
                        if (rel == EQ)
                            genScanFilter_and_eq<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == NOT_EQ)
                            genScanFilter_and_neq<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == GTH)
                            genScanFilter_and_gth<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == LTH)
                            genScanFilter_and_lth<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == GEQ)
                            genScanFilter_and_geq<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == LEQ)
                            genScanFilter_and_leq<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == EQ_VEC)
                            genScanFilter_and_eq_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == NOT_EQ_VEC)
                            genScanFilter_and_neq_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == GTH_VEC)
                            genScanFilter_and_gth_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == LTH_VEC)
                            genScanFilter_and_lth_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == GEQ_VEC)
                            genScanFilter_and_geq_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == LEQ_VEC)
                            genScanFilter_and_leq_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == IN)
                            genScanFilter_and_in<<<grid, block>>>(column[whereIndex], sn->tn->attrSize[index], totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == IN_VEC)
                            genScanFilter_and_in_vec<<<grid, block>>>(column[whereIndex], sn->tn->attrSize[index], totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == LIKE)
                            genScanFilter_and_like<<<grid, block>>>(column[whereIndex], sn->tn->attrSize[index], totalTupleNum, gpuExp, gpuFilter);
                    }else{
                        if (rel == EQ)
                            genScanFilter_or_eq<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == NOT_EQ)
                            genScanFilter_or_neq<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == GTH)
                            genScanFilter_or_gth<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == LTH)
                            genScanFilter_or_lth<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == GEQ)
                            genScanFilter_or_geq<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == LEQ)
                            genScanFilter_or_leq<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == EQ_VEC)
                            genScanFilter_or_eq_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == NOT_EQ_VEC)
                            genScanFilter_or_neq_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == GTH_VEC)
                            genScanFilter_or_gth_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == LTH_VEC)
                            genScanFilter_or_lth_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == GEQ_VEC)
                            genScanFilter_or_geq_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == LEQ_VEC)
                            genScanFilter_or_leq_vec<<<grid,block>>>(column[whereIndex],sn->tn->attrSize[index],totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == IN)
                            genScanFilter_or_in<<<grid, block>>>(column[whereIndex], sn->tn->attrSize[index], totalTupleNum, gpuExp, gpuFilter);
                        else if (rel == IN_VEC){
                            // printf("sn->tn->attrSize[index] = %d\n", sn->tn->attrSize[index]);
                            genScanFilter_or_in_vec<<<grid, block>>>(column[whereIndex], sn->tn->attrSize[index], totalTupleNum, gpuExp, gpuFilter);
                        }
                        else if (rel == LIKE)
                            genScanFilter_or_like<<<grid, block>>>(column[whereIndex], sn->tn->attrSize[index], totalTupleNum, gpuExp, gpuFilter);
                    }
                }
            }
        }

        //Stop timer for Step 1 - Copy where clause (Part2)
        CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
        clock_gettime(CLOCK_REALTIME, &endS12);
        pp->whereMemCopy_s1 += (endS12.tv_sec - startS12.tv_sec)* BILLION + endS12.tv_nsec - startS12.tv_nsec;            

        if(prevFormat == DICT){
            if (dictInit == 1){
                transform_dict_filter_init<<<grid,block>>>(gpuDictFilter, column[prevWhere], totalTupleNum, dNum, gpuFilter, byteNum);
                dictInit = 0;
            }else if(dictFinal == AND)
                transform_dict_filter_and<<<grid,block>>>(gpuDictFilter, column[prevWhere], totalTupleNum, dNum, gpuFilter, byteNum);
            else
                transform_dict_filter_or<<<grid,block>>>(gpuDictFilter, column[prevWhere], totalTupleNum, dNum, gpuFilter, byteNum);
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuDictFilter));
        }

        if(whereFree[prevWhere] == 1 && (sn->tn->dataPos[prevIndex] == MEM || sn->tn->dataPos[prevIndex] == MMAP || sn->tn->dataPos[prevIndex] == PINNED))
            CUDA_SAFE_CALL_NO_SYNC(hipFree(column[prevWhere]));

        if(dev_mp == NULL)
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuExp));

    }

    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME,&e);
    tt = (e.tv_sec - s.tv_sec)* BILLION + e.tv_nsec - s.tv_nsec;
    printf("tt2 = %f\n", tt/(1000*1000));
    clock_gettime(CLOCK_REALTIME,&s);


    //Start timer for Step 4 - Count result (PreScan)
    struct timespec startS4, endS4;
    clock_gettime(CLOCK_REALTIME,&startS4);


    // int cntgf = 0;
    // for(int i = 0; i < globalFilter.size(); i++) {
    //     if(globalFilter[i] == 1){
    //         // printf("globalFilter[%d] = 1\n", i);
    //         cntgf += 1;
    //     }
    // }
    // printf("[globalFilter] there are %d tuples to be choosen\n", cntgf);
    
    // if we have where exp, gpuFilter is not NULL
    // we need to AND gpuFilter/globalFilter
    if(gpuFilter != NULL and has_globalFilter == 1){
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuFilter, globalFilter, sizeof(int) * totalTupleNum, hipMemcpyDeviceToDevice));
    }
    else if(gpuFilter != NULL){
        ;
    }
    else { // else, initial gpuFilter and copy globalFilter to gpuFilter
        if(dev_mp == NULL){
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuFilter,sizeof(int) * totalTupleNum));
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuPsum,sizeof(int)*threadNum));
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuCount,sizeof(int)*threadNum));
        }else{
            gpuFilter = (int *) dev_mp->alloc(sizeof(int) * totalTupleNum);
            gpuPsum   = (int *) dev_mp->alloc(sizeof(int) * threadNum);
            gpuCount  = (int *) dev_mp->alloc(sizeof(int) * threadNum);
        }
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuFilter, globalFilter, sizeof(int) * totalTupleNum, hipMemcpyDeviceToDevice));
        // printf("[tableScan] no where clause, copy globalFilter to gpuFilter\n\n");
    }

    /* --Optimization Note--
    * We cannot remove this one even if we know the number of selected tuples
    * because counts the tuples per thread (not only total number)
    */

    /* 
    * Count the number of tuples that meets the predicats for each thread
    * and calculate the prefix sum.
    */

    //Start timer for Count Step 4.1 - Count selected rows kernels
    struct timespec startCountS1, endCountS1;
    clock_gettime(CLOCK_REALTIME,&startCountS1);

    // countScanNum<<<grid,block>>>(gpuFilter,totalTupleNum,gpuCount);

    //Stop timer for Count Step 4.1 - Count selected rows kernels


    if(idx == NULL)
        countScanNum<<<grid,block>>>(gpuFilter,totalTupleNum,gpuCount);
    else{
        countScanNum_idx<<<grid, block>>>(gpuFilter, idx, st, ed, gpuCount);
    }
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endCountS1);
    pp->countScanKernel_countS1 += (endCountS1.tv_sec - startCountS1.tv_sec)* BILLION + endCountS1.tv_nsec - startCountS1.tv_nsec;

    //Start timer for Count Step 4.2 - scanImpl time
    struct timespec startCountS2, endCountS2;
    clock_gettime(CLOCK_REALTIME,&startCountS2);
    scanImpl(gpuCount,threadNum, gpuPsum, pp);

    // int * testPsum = (int*)malloc(sizeof(int) * threadNum);
    // hipMemcpy(testPsum, gpuPsum, sizeof(int) * threadNum, hipMemcpyDeviceToHost);
    // for(int i = 0; i < threadNum; i++){
    //     printf("gpuPsum[%d] = %d", i, testPsum[i]);
    // }

    //Stop timer for Count Step 4.2 - scanImpl time
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endCountS2);
    pp->scanImpl_countS2 += (endCountS2.tv_sec - startCountS2.tv_sec)* BILLION + endCountS2.tv_nsec - startCountS2.tv_nsec;

    //Stop timer for Step 4 - Count result (PreScan)
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS4);
    pp->preScanTotal_s4 += (endS4.tv_sec -  startS4.tv_sec)* BILLION + endS4.tv_nsec - startS4.tv_nsec;
    pp->preScanCount_s4++;

    int tmp1, tmp2;

    //Start timer for Step 5 - Count result (PreScan)
    struct timespec startS5, endS5;
    clock_gettime(CLOCK_REALTIME,&startS5);

    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&tmp1, &gpuCount[threadNum-1], sizeof(int), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&tmp2, &gpuPsum[threadNum-1], sizeof(int), hipMemcpyDeviceToHost));    
    count = tmp1+tmp2;
    printf("[tableScan] count = %d\n", count);
    
    if(disTwoColProject != 0)
        count = totalTupleNum;
    res->tupleNum = count;

    

    //End timer for Step 5 - Count result (PreScan)
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS5);
    pp->preScanResultMemCopy_s5 += (endS5.tv_sec -  startS5.tv_sec)* BILLION + endS5.tv_nsec - startS5.tv_nsec;
    
    //Start timer for Other - 02 (mallocRes)
    struct timespec startS02,endS02;
    clock_gettime(CLOCK_REALTIME,&startS02);

    if(dev_mp == NULL)
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuCount));

    char **result = NULL, **scanCol = NULL;

    attrNum = sn->outputNum;

    if(host_mp == NULL){
        scanCol = (char **) malloc(attrNum * sizeof(char *));
        CHECK_POINTER(scanCol);
        result = (char **) malloc(attrNum * sizeof(char *));
        CHECK_POINTER(result);
    }else{
        scanCol = (char **)host_mp->alloc(attrNum * sizeof(char *));
        result = (char **)host_mp->alloc(attrNum * sizeof(char *));
    }

    //Stop timer for Other - 02 (mallocRes)
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS02);
    pp->mallocRes_S02 += (endS02.tv_sec - startS02.tv_sec)* BILLION + endS02.tv_nsec - startS02.tv_nsec;

    //Start timer for Step 6 - Copy to device all other columns
    struct timespec startS6, endS6;
    clock_gettime(CLOCK_REALTIME,&startS6);

    // printf("[tableScan] attrNum = %d\n", attrNum);
    
    if(sn->filter != NULL and sn->hasWhere != 0){
        // printf("sn->hasWhere != NULL and sn->hasWhere != 0\n");
        for(int i=0;i<attrNum;i++){

            int pos = colWherePos[i];
            int index = sn->outputIndex[i];
            tupleSize += sn->tn->attrSize[index];

            if(pos != -1){
                scanCol[i] = column[pos];
            }else{
                if(sn->tn->dataPos[index] == MEM || sn->tn->dataPos[index] == MMAP || sn->tn->dataPos[index] == PINNED)
                    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &scanCol[i] , sn->tn->attrTotalSize[index]));

                if(sn->tn->dataFormat[index] != DICT){
                    if(sn->tn->dataPos[index] == MEM || sn->tn->dataPos[index] == MMAP || sn->tn->dataPos[index] == PINNED){
                        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(scanCol[i], sn->tn->content[index], sn->tn->attrTotalSize[index], hipMemcpyHostToDevice));
                    }
                    else
                        scanCol[i] = sn->tn->content[index];

                }else{
                    if(sn->tn->dataPos[index] == MEM || sn->tn->dataPos[index] == MMAP || sn->tn->dataPos[index] == PINNED){
                        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(scanCol[i], sn->tn->content[index], sn->tn->attrTotalSize[index], hipMemcpyHostToDevice));
                    }
                    else
                        scanCol[i] = sn->tn->content[index];
                }
            }

            if(res_mp == NULL)
                CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &result[i], count * sn->tn->attrSize[index]));
            else
                result[i] = (char *) res_mp->alloc(count * sn->tn->attrSize[index]);
        }
    }
    else{
        // printf("[Info] there is no where clause\n");
        int index = 0;
        for(int i=0;i<attrNum;i++){
            if(i < sn->outputNum){
                index = sn->outputIndex[i];
                // printf("index = %d, attrSize = %d\n", sn->outputIndex[i], sn->tn->attrSize[index]);
            } else {
                index = dTwoCol[0].secondColIndex;
            }
            
            // printf("outputIndex[%d] = %d, attrSize = %d\n", i, index, sn->tn->attrSize[index]);
            tupleSize += sn->tn->attrSize[index];
            // printf("tupleSize = %d\n", tupleSize);

            if(sn->tn->dataPos[index] == MEM || sn->tn->dataPos[index] == MMAP || sn->tn->dataPos[index] == PINNED){
                // printf("copy data from host to device\n");
                // printf("sn->tn->attrTotalSize[%d] = %d\n", index, sn->tn->attrTotalSize[index]);
                CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &scanCol[i] , sn->tn->attrTotalSize[index]));
                // printf("Breakpoint\n");
                CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(scanCol[i], sn->tn->content[index], sn->tn->attrTotalSize[index], hipMemcpyHostToDevice));
                // printf("attrTotalSize[%d] = %d\n", index, sn->tn->attrTotalSize[index]);
                // printf("totalTupleNum[%ld] = %d\n", index, totalTupleNum);
                // printf("\n");
                // if(sn->tn->attrType[index] == VECTOR){
                //     float *data = (float*)(sn->tn->content[index]);
                //     for(int i = 0; i < 128; ++i){
                //         printf("data[%d] = %f\n", i, data[i]);
                //     }
                // }
            }
            else
                scanCol[i] = sn->tn->content[index];

            // printf("count = %d\n", count);
            if(res_mp == NULL)
                CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &result[i], count * sn->tn->attrSize[index]));
            else
                result[i] = (char *) res_mp->alloc(count * sn->tn->attrSize[index]);
        }
    }
    //End timer for Step 6 - Copy to device all other columns
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS6);
    pp->dataMemCopyOther_s6 += (endS6.tv_sec -  startS6.tv_sec)* BILLION + endS6.tv_nsec - startS6.tv_nsec;

    //Start timer for Step 7 - Materialize result
    struct timespec startS7, endS7;
    clock_gettime(CLOCK_REALTIME,&startS7);

    if(sn->filter != NULL or count != totalTupleNum){
        // printf("[Info] scan column\n");
        // printf("attrNum = %d\n", attrNum);
        for(int i=0; i<attrNum; i++){
            int index = sn->outputIndex[i];
            int format = sn->tn->dataFormat[index];
            if(format == UNCOMPRESSED){
                if (sn->tn->attrSize[index] == sizeof(int)){
                    if(idx == NULL)
                        scan_int<<<grid,block>>>(scanCol[i], sn->tn->attrSize[index], totalTupleNum, gpuPsum, count, gpuFilter, result[i]);
                    else
                        scan_int_idx<<<grid,block>>>(scanCol[i], sn->tn->attrSize[index], idx, st, ed, gpuPsum, count, gpuFilter, result[i]);
                }else{
                    if(idx == NULL)
                        scan_other<<<grid,block>>>(scanCol[i], sn->tn->attrSize[index], totalTupleNum,gpuPsum, count, gpuFilter,result[i]);
                    else
                        scan_other_idx<<<grid,block>>>(scanCol[i], sn->tn->attrSize[index], idx, st, ed, gpuPsum, count, gpuFilter,result[i]);
                }
            }
        }
    } else {
        // printf("[Info] copy column\n");
        for(int i=0; i<attrNum; i++){
            result[i] = scanCol[i];
        }
    }

    // float * test = (float *)malloc(sizeof(float) * 128 * count);
    // hipMemcpy(test, result[1], sizeof(float) * 128 * count, hipMemcpyDeviceToHost);
    // for(int i = 0; i < 128; i++){
    //     printf("test[%d] = %f\n", i, test[i]);
    // }

    //End timer for Step 7 - Materialize result
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS7);
    pp->materializeResult_s7 += (endS7.tv_sec -  startS7.tv_sec)* BILLION + endS7.tv_nsec - startS7.tv_nsec;
   
        
    //Start timer for Step 8 - Copy final result
    struct timespec startS8, endS8;
    clock_gettime(CLOCK_REALTIME,&startS8);

    res->tupleSize = tupleSize;

    for(int i=0;i<attrNum;i++){

        int index = sn->outputIndex[i];
        // printf("dataPos[%d] = %d (49 MEM)\n", index, sn->tn->dataPos[index]);

        if(sn->tn->dataPos[index] == MEM || sn->tn->dataPos[index] == MMAP || sn->tn->dataPos[index] == PINNED)
            CUDA_SAFE_CALL_NO_SYNC(hipFree(scanCol[i]));

        int colSize = res->tupleNum * res->attrSize[i];

        res->attrTotalSize[i] = colSize;
        res->dataFormat[i] = UNCOMPRESSED;

        if(sn->keepInGpu == 1){
            // printf("keepInGpu == 1 (yes)\n");
            res->dataPos[i] = GPU;
            res->content[i] = result[i];
        }else{
            res->dataPos[i] = MEM;
            res->content[i] = (char *)malloc(colSize);
            CHECK_POINTER(res->content[i]);
            memset(res->content[i],0,colSize);
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(res->content[i],result[i],colSize ,hipMemcpyDeviceToHost));
            CUDA_SAFE_CALL(hipFree(result[i]));
        }
    }

    //End timer for Step 8 - Copy final result
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS8);
    pp->finalResultMemCopy_s8 += (endS8.tv_sec -  startS8.tv_sec)* BILLION + endS8.tv_nsec - startS8.tv_nsec;

    // project result
    // 1.project distance_filter
    float * projectRes  = NULL;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &projectRes , sizeof(float) * count));

    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing

    for (int i = 0; i < disFilterCount; i++) {
        int projectIndex = dFilter[i].colIndex;
        // printf("\n[project] projectIndex = %d (1?)\n", projectIndex);
        // std::vector<float> search = dFilter[i].searchVec;
        std::vector<float> search = searchV;
        // for(int i = 0; i < search.size(); i++){
        //     printf("search[%d] = %f", i, search[i]);
        //     if(i % 10 == 0)
        //         printf("\n");
        // }
        // printf("\n");

        // cal L2 distance
        // tupleNum = count
        float *gpuSearch = NULL;
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &gpuSearch , sizeof(float) * search.size()));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuSearch, search.data(), sizeof(float) * search.size(), hipMemcpyHostToDevice));
        int dim = res->attrSize[projectIndex] / sizeof(float);
        // printf("\n[project] dim = %d (128?)\n", dim);
        // printf("[project] count = %d (250?)\n\n", count);

        int blockSize = 256;
        int numBlocks = (count + blockSize - 1) / blockSize;
        L2distance<<<numBlocks, blockSize>>>(projectRes, gpuSearch, res->content[projectIndex], dim, count);
        CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
        // printf("After project res calculation!!! check the parameters and results!!!\n");

        // initial new "result[projectIndex]" column
        res->content[projectIndex] = (char*)projectRes;
        tupleSize = res->attrSize[projectIndex];
        res->attrType[projectIndex] = FLOAT;
        res->attrSize[projectIndex] = sizeof(float);
        res->attrTotalSize[projectIndex] = count * sizeof(float); 
        res->tupleSize += -tupleSize + sizeof(float);
    }

    // printf("there are %d tuples after where clause\n", res->tupleNum);
    // 2.project distance(v1, v2)
    struct disTwoColIndex2Res{
        int projectIndex; /*  */
        char *distanceTwoColRes = NULL;
    };
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing

    struct disTwoColIndex2Res *dTwoColIndex2Res = (struct disTwoColIndex2Res *)malloc(sizeof(struct disTwoColIndex2Res) * disTwoColProject);
    int projectTupleSize = 0;
    for (int i = 0; i < disTwoColProject; i++) {
        
        // test input
        // int output = 10;
        // float * test = (float *)malloc(sizeof(float) * 2048 * output);
        // hipMemcpy(test, res->content[1], sizeof(float) * 2048 * output, hipMemcpyDeviceToHost);
        // for(int i = 0; i < 2048 * output; i++){
        //     printf("[tableScan] column[%d] = %f\n", i, test[i]);
        // }
        // hipMemcpy(test, res->content[2], sizeof(float) * 2048 * output, hipMemcpyDeviceToHost);
        // for(int i = 0; i < 2048 * output; i++){
        //     printf("[tableScan] column[%d] = %f\n", i, test[i]);
        // }

        dTwoColIndex2Res[i].projectIndex = dTwoCol[i].projectIndex;
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &(dTwoColIndex2Res[i].distanceTwoColRes) , totalTupleNum * sizeof(float)));
        int dim = res->attrSize[dTwoCol[i].firstColIndex] / sizeof(float);
        CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing

        // printf("dTwoColIndex2Res[i].projectIndex = %d\n", dTwoColIndex2Res[i].projectIndex);
        // printf("sizeof resCol = %d * 4\n", count);
        // printf("dim = %d\n", dim);

        // printf("dTwoCol[i].firstColIndex = %d\n", dTwoCol[i].firstColIndex);
        // printf("dTwoCol[i].secondColIndex = %d\n", dTwoCol[i].secondColIndex);
        twoColL2Distance<<<grid, block>>>(dTwoColIndex2Res[i].distanceTwoColRes, res->content[dTwoCol[i].firstColIndex], res->content[dTwoCol[i].secondColIndex], dim, count);
        CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    }

    struct tableNode *projectResTable = NULL;
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    if(disTwoColProject != 0){
        if(host_mp == NULL){
            projectResTable = (struct tableNode *) malloc(sizeof(struct tableNode));
            CHECK_POINTER(projectResTable);
        }else
            projectResTable = (struct tableNode *) host_mp->alloc(sizeof(struct tableNode));

        projectResTable->totalAttr = sn->outputNum - disTwoColProject;
        projectResTable->tupleNum = totalTupleNum;

        if(host_mp == NULL) {
            projectResTable->attrType = (int *) malloc(sizeof(int) * projectResTable->totalAttr);
            CHECK_POINTER(projectResTable->attrType);
            projectResTable->attrSize = (int *) malloc(sizeof(int) * projectResTable->totalAttr);
            CHECK_POINTER(projectResTable->attrSize);
            projectResTable->attrTotalSize = (int *) malloc(sizeof(int) * projectResTable->totalAttr);
            CHECK_POINTER(projectResTable->attrTotalSize);
            projectResTable->attrIndex = (int *) malloc(sizeof(int) * projectResTable->totalAttr);
            CHECK_POINTER(projectResTable->attrIndex);
            projectResTable->dataPos = (int *) malloc(sizeof(int) * projectResTable->totalAttr);
            CHECK_POINTER(projectResTable->dataPos);
            projectResTable->dataFormat = (int *) malloc(sizeof(int) * projectResTable->totalAttr);
            CHECK_POINTER(res->dataFormat);
            projectResTable->content = (char **) malloc(sizeof(char *) * projectResTable->totalAttr);
            CHECK_POINTER(projectResTable->content);
        }else{
            projectResTable->attrType = (int *) host_mp->alloc(sizeof(int) * projectResTable->totalAttr);
            projectResTable->attrSize = (int *) host_mp->alloc(sizeof(int) * projectResTable->totalAttr);
            projectResTable->attrTotalSize = (int *) host_mp->alloc(sizeof(int) * projectResTable->totalAttr);
            projectResTable->attrIndex = (int *) host_mp->alloc(sizeof(int) * projectResTable->totalAttr);
            projectResTable->dataPos = (int *) host_mp->alloc(sizeof(int) * projectResTable->totalAttr);
            projectResTable->dataFormat = (int *) host_mp->alloc(sizeof(int) * projectResTable->totalAttr);
            projectResTable->content = (char **) host_mp->alloc(sizeof(char *) * projectResTable->totalAttr);
        }

        CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing

        printf("\n[projectResTable] totalAttr = %d\n", projectResTable->totalAttr);

        for(int i=0;i<projectResTable->totalAttr;i++){
            int proIndex = sn->projectIndex[i];
            if(proIndex < 100){
                int index = sn->outputIndex[i];
                projectResTable->attrType[i] = res->attrType[index];
                projectResTable->attrSize[i] = res->attrSize[index];
                projectResTable->attrTotalSize[i] = res->attrTotalSize[index];
                projectResTable->attrIndex[i] = res->attrIndex[index];
                projectResTable->dataPos[i] = res->dataPos[index];
                projectResTable->dataFormat[i] = res->dataFormat[index];
                projectResTable->content[i] = res->content[index];
                res->content[index] = NULL;
                projectTupleSize += res->attrSize[index];
            }
            else{
                int index = proIndex-100;
                projectResTable->attrType[i] = FLOAT;
                printf("[projectResTable] attrType[%d] = FLOAT\n", i);
                projectResTable->attrSize[i] = sizeof(float);
                projectResTable->attrTotalSize[i] =  projectResTable->tupleNum * sizeof(float);
                projectResTable->attrIndex[i] = i; // right??
                projectResTable->dataPos[i] = GPU;
                projectResTable->dataFormat[i] = UNCOMPRESSED;
                int tmp = -1;
                for(int k = 0; k < disTwoColProject; k++){
                    if(dTwoColIndex2Res[k].projectIndex == index){
                        tmp = k;
                        break;
                    }
                }
                // printf("tmp = %d\n", tmp);
                projectResTable->content[i] = dTwoColIndex2Res[tmp].distanceTwoColRes;
                dTwoColIndex2Res[tmp].distanceTwoColRes = NULL;
                projectTupleSize += sizeof(float);
            }
        }
        projectResTable->tupleSize = projectTupleSize;
    }

    //Start timer for Other - 03 (deallocate buffers)
    struct timespec startS03,endS03;
    clock_gettime(CLOCK_REALTIME,&startS03);
    if(dev_mp == NULL){
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuPsum));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuFilter));
    }
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    
    if(host_mp == NULL){
        free(whereFree);
        free(colWherePos);
        free(column);
        free(scanCol);
        free(result);
    }
    //Stop timer for Other - 01 (tableScan)
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS03);
    pp->deallocateBuffs_S03 += (endS03.tv_sec - startS03.tv_sec)* BILLION + endS03.tv_nsec - startS03.tv_nsec;

    //Stop timer (Global tableScan())
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME,&endTableScanTotal);
    pp->tableScanTotal += (endTableScanTotal.tv_sec -  startTableScanTotal.tv_sec)* BILLION + endTableScanTotal.tv_nsec - startTableScanTotal.tv_nsec;
    pp->tableScanCount ++;

    // if(sn->filter != NULL and sn->filter->exp[0].relation == KNN){
    //     char disArray[2048];
    //     char intArray[2048];
    //     printf("[tableScan.cu] the type of result[0] :%d\n", res->attrType[0]);
    //     hipMemcpy(disArray, res->content[0], sizeof(char) * res->attrSize[0] * res->tupleNum, hipMemcpyDeviceToHost);
    //     for(int i = 0; i < res->tupleNum; i++){
    //         printf("[tableScan.cu] tableName[%d] = ", i);
    //         for(int j = 0; j < 23; j++){
    //             printf("%c", disArray[i*23 + j]);
    //         }
    //         printf("\n");
    //     }
    // }
    

    if(disTwoColProject != 0){
        printf("[tableScan.cu] outputTable tupleNum = %ld\n", projectResTable->tupleNum);
        clock_gettime(CLOCK_REALTIME,&e);
        tt = (e.tv_sec - s.tv_sec)* BILLION + e.tv_nsec - s.tv_nsec;
        printf("tt = %f\n", tt/(1000*1000));
        printf("TableScan Time: %lf\n", pp->tableScanTotal/(1000*1000));
        return projectResTable;
    }

    printf("res->tupleNum = %ld\n", res->tupleNum);

    clock_gettime(CLOCK_REALTIME,&e);
    tt = (e.tv_sec - s.tv_sec)* BILLION + e.tv_nsec - s.tv_nsec;
    printf("tt = %f\n", tt/(1000*1000));
    printf("<--TableScan Time -->    : %lf\n\n", pp->tableScanTotal/(1000*1000));
    return res;
}

/*
 * Creates sorted index for the selected column
 * 
 * --Input--
 * tn -> Table node
 * idxPos -> Position of the indexed column in contentIdx and posIdx
 * columnPos -> Column to be sorted
 *
 * --Output--
 * tn.contentIdx (sorted values)
 * tn.posIdx (position in the original table )
 */
void createIndex (struct tableNode *tn, int idxPos, int columnPos, struct statistic *pp){

    //Start timer for build index
    struct timespec startIdxBuildTime, endIdxBuildTime;
    clock_gettime(CLOCK_REALTIME,&startIdxBuildTime);
 
    //Check assumption (INT enum == 4)
    if (tn->attrType[columnPos] != 4 ){
        printf("[ERROR] Indexing is only supported for INT type!\n");
        exit(-1);
    }
    if (tn->attrSize[columnPos] != sizeof(int)){
        printf("[ERROR] Indexing is only supported for INT type (and size!)!\n");
        exit(-1); 
    }

    //Define Grid and block size
    dim3 grid(2048);
    dim3 block(256);

    //Get data size
    long dataSize = sizeof(int) * tn->tupleNum;

    //Allocate memory for the index (in host)
    tn->contentIdx[idxPos] = (int *)malloc(dataSize); 
    CHECK_POINTER(tn->contentIdx[idxPos]);
    tn->posIdx[idxPos] = (int *)malloc(sizeof(int) * tn->tupleNum); 
    CHECK_POINTER(tn->posIdx[idxPos]);


    //Assign index (in device)
    int* posIdx_d;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&posIdx_d, sizeof(int) * tn->tupleNum));
    assign_index<<<grid,block>>>(posIdx_d, tn->tupleNum);


    //Copy values (in device)
    int* contentIdx_d;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&contentIdx_d, dataSize));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(contentIdx_d, tn->content[columnPos], dataSize, hipMemcpyHostToDevice));

    //Sort columns
    thrust::sort_by_key(thrust::device, contentIdx_d, contentIdx_d + tn->tupleNum, posIdx_d); //thrust inplace sorting
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait for short (or SEGFAULT)

    if (tn->keepInGpuIdx){

        //De-allocate host memory
        free(tn->contentIdx[idxPos]);
        free(tn->posIdx[idxPos]);

        //Store Device pointers
        tn->contentIdx[idxPos] = contentIdx_d;
        tn->posIdx[idxPos] = posIdx_d;

        //Set position
        tn->indexPos = GPU;

    }else{
        
        //Copy index to host
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(tn->contentIdx[idxPos], contentIdx_d, dataSize, hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(tn->posIdx[idxPos], posIdx_d, sizeof(int) * tn->tupleNum, hipMemcpyDeviceToHost)); 
    
        //De-allocate device memory
        CUDA_SAFE_CALL_NO_SYNC(hipFree(contentIdx_d));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(posIdx_d));
        
        //Set position
        tn->indexPos = MEM;
    }

    //Stop and add 
    clock_gettime(CLOCK_REALTIME, &endIdxBuildTime);
    pp->buildIndexTotal += (endIdxBuildTime.tv_sec -  startIdxBuildTime.tv_sec)* BILLION + endIdxBuildTime.tv_nsec - startIdxBuildTime.tv_nsec;
}

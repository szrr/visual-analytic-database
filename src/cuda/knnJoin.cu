#include "hip/hip_runtime.h"
/*
   Copyright (c) 2012-2013 The Ohio State University.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/


#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "../include/common.h"
#include "../include/utils.h"
#include "../include/hashJoin.h"
#include "../include/gpuCudaLib.h"
#include "scanImpl.cu"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#include "../include/Mempool.h"

#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <random>

#include <faiss/gpu/GpuAutoTune.h>
#include <faiss/gpu/GpuCloner.h>
#include <faiss/gpu/GpuIndexIVFPQ.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/index_io.h>
#include <faiss/IndexFlat.h>
#include <faiss/IndexIVFPQ.h>

__global__ static void copy_left_table(const char* data, size_t size, int num, int k, char* result) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < num) {
        const char* src = data + index * size;
        char* dest = result + index * size * k;

        for (int i = 0; i < k; i++) {
            memcpy(dest, src, size);
            dest += size;
        }
    }
}

__global__ static void copy_right_table(const char* data, size_t size, int num, const long* nns, char* result) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < num) {
        long nn = nns[index];
        const char* src = data + nn * size;
        char* dest = result + index * size;

        memcpy(dest, src, size);
    }
}

struct tableNode * knnJoin(struct joinNode *jNode, struct statistic *pp,
                            Mempool *host_mp = NULL, Mempool *dev_mp = NULL, Mempool *res_mp = NULL, int *rightTableHash = NULL, faiss::Index *faiss_index = NULL){

    // float *disArray = (float*)malloc(1024*1024);
    // printf("[driver.cu] the type of result[0] = %d\n", jNode->rightTable->attrType[0]);
    // printf("dataset_videoTable->attrSize[0] = %d\n", jNode->rightTable->attrSize[0]);
    // printf("dataset_videoTable->tupleNum = %d\n", jNode->rightTable->tupleNum);
    // memcpy(disArray, jNode->rightTable->content[0], sizeof(char) * jNode->rightTable->attrSize[0] * jNode->rightTable->tupleNum);
    // for(int i = 0; i < 100; i++){
    //     printf("search feature[%d] = ", i);
    //     for(int j = 0; j < 19; j++){
    //         printf("%f\t", disArray[i*19 + j]);
    //     }
    //     printf("\n");
    // }

    printf("================ [knnJoin.cu] Start ================\n");
    struct timespec annsAddStart, annsAddEnd;
    struct timespec annsSearchStart, annsSearchEnd;
    struct timespec readIndexStart, readIndexEnd;
    struct timespec start, end;
    struct timespec annsStart, annsEnd;

    //Start total timer
    struct timespec startS0, endS0;
    clock_gettime(CLOCK_REALTIME,&startS0);

    struct tableNode * res = NULL;

    char *gpu_result = NULL;
    int *gpu_count = NULL;
    long *gpu_nns = NULL;

    int defaultBlock = 4096;
    dim3 grid(defaultBlock);
    dim3 block(256);

    int blockNum;
    int threadNum;

    // blockNum = jNode->leftTable->tupleNum / block.x + 1;
    // if(blockNum < defaultBlock)
    //     grid = blockNum;
    // else
    //     grid = defaultBlock;
    grid = 1;

    if(host_mp == NULL) {
        res = (struct tableNode*) malloc(sizeof(struct tableNode));
        CHECK_POINTER(res);
    }else
        res = (struct tableNode *) host_mp->alloc(sizeof(struct tableNode));

    
    res->totalAttr = jNode->totalAttr;
    res->tupleSize = jNode->tupleSize;
    res->tupleNum = jNode->nearestk * jNode->leftTable->tupleNum;
    if(host_mp == NULL) {
        res->attrType = (int *) malloc(res->totalAttr * sizeof(int));
        CHECK_POINTER(res->attrType);
        res->attrSize = (int *) malloc(res->totalAttr * sizeof(int));
        CHECK_POINTER(res->attrSize);
        res->attrIndex = (int *) malloc(res->totalAttr * sizeof(int));
        CHECK_POINTER(res->attrIndex);
        res->attrTotalSize = (int *) malloc(res->totalAttr * sizeof(int));
        CHECK_POINTER(res->attrTotalSize);
        res->dataPos = (int *) malloc(res->totalAttr * sizeof(int));
        CHECK_POINTER(res->dataPos);
        res->dataFormat = (int *) malloc(res->totalAttr * sizeof(int));
        CHECK_POINTER(res->dataFormat);
        res->content = (char **) malloc(res->totalAttr * sizeof(char *));
        CHECK_POINTER(res->content);
    }else{
        res->attrType = (int *) host_mp->alloc(res->totalAttr * sizeof(int));
        res->attrSize = (int *) host_mp->alloc(res->totalAttr * sizeof(int));
        res->attrIndex = (int *) host_mp->alloc(res->totalAttr * sizeof(int));
        res->attrTotalSize = (int *) host_mp->alloc(res->totalAttr * sizeof(int));
        res->dataPos = (int *) host_mp->alloc(res->totalAttr * sizeof(int));
        res->dataFormat = (int *) host_mp->alloc(res->totalAttr * sizeof(int));
        res->content = (char **) host_mp->alloc(res->totalAttr * sizeof(char *));
    }

    for(int i=0;i<jNode->leftOutputAttrNum;i++){
        int pos = jNode->leftPos[i];
        res->attrType[pos] = jNode->leftOutputAttrType[i];
        int index = jNode->leftOutputIndex[i];
        res->attrSize[pos] = jNode->leftTable->attrSize[index];
        res->dataFormat[pos] = UNCOMPRESSED;
        res->attrTotalSize[pos] = res->attrSize[pos] * res->tupleNum;
    }

    for(int i=0;i<jNode->rightOutputAttrNum;i++){
        int pos = jNode->rightPos[i];
        res->attrType[pos] = jNode->rightOutputAttrType[i];
        int index = jNode->rightOutputIndex[i];
        res->attrSize[pos] = jNode->rightTable->attrSize[index];
        res->dataFormat[pos] = UNCOMPRESSED;
        res->attrTotalSize[pos] = res->attrSize[pos] * res->tupleNum;
    }
    

    /* generate nearest neighbor index
       search vector : left table join index
       search data   : right table join index
       index length  : left table tupleNum * nearestk
    */

    printf("[knnJoin] left tupleNum = %ld\n", jNode->leftTable->tupleNum);

    // int searchIndex = jNode->leftKeyIndex;
    float *searhVec = (float *)(jNode->leftTable->content[jNode->leftKeyIndex]);
    float *feature = (float *)(jNode->rightTable->content[jNode->rightKeyIndex]);
    size_t feature_num = (size_t)(jNode->rightTable->tupleNum);
    char index_path[256];
    memset(index_path, '\0', sizeof(index_path));
    if(jNode->rightTable->attrName != NULL) {
        strcat(index_path, jNode->rightTable->attrName[jNode->rightKeyIndex]);
        strcat(index_path, "_index.faissindex");
    } else {
        printf("attrName not exist\n");
        exit(1);
    }
    int dev_no = 0;
    int k = jNode->nearestk;
    printf("k = %d\n", k);
    int nq = jNode->leftTable->tupleNum;  //query num
    std::vector<faiss::idx_t> nns(k * nq);
    std::vector<float> dis(k * nq);
    std::cout << "index path : " << index_path << std::endl;
    std::cout << "fileExists(index_path) = " << fileExists(index_path) << std::endl;

    clock_gettime(CLOCK_REALTIME,&annsStart);
    if(faiss_index != NULL){
        // searching...
        clock_gettime(CLOCK_REALTIME,&annsSearchStart);
        // index->search(nq, dFilter[i].searchVec.data(), k, dis.data(), nns.data());
        faiss_index->search(nq, searhVec, k, dis.data(), nns.data());
        clock_gettime(CLOCK_REALTIME,&annsSearchEnd);
        pp->indexSearch += (annsSearchEnd.tv_sec -  annsSearchStart.tv_sec)* BILLION + annsSearchEnd.tv_nsec - annsSearchStart.tv_nsec;
    }
    else if (fileExists(index_path)) {
        // read faiss index from index_path
        faiss::gpu::StandardGpuResources resources;
        clock_gettime(CLOCK_REALTIME,&readIndexStart);
        faiss::Index *index = faiss::read_index(index_path);
        clock_gettime(CLOCK_REALTIME,&readIndexEnd);
        double readIndexTotal = (readIndexEnd.tv_sec -  readIndexStart.tv_sec)* BILLION + readIndexEnd.tv_nsec - readIndexStart.tv_nsec;
        printf("read_index time: %f\n", readIndexTotal/(1000*1000));
        clock_gettime(CLOCK_REALTIME,&start);
        index = faiss::gpu::index_cpu_to_gpu(&resources, dev_no, index);
        clock_gettime(CLOCK_REALTIME,&end);
        double total = (end.tv_sec -  start.tv_sec)* BILLION + end.tv_nsec - start.tv_nsec;
        printf("index_cpu_to_gpu time: %f\n", total/(1000*1000));
        
        // add...
        // clock_gettime(CLOCK_REALTIME,&annsAddStart);
        // index->add(feature_num, feature);
        // clock_gettime(CLOCK_REALTIME,&annsAddEnd);
        // double annsAddTotal = (annsAddEnd.tv_sec -  annsAddStart.tv_sec)* BILLION + annsAddEnd.tv_nsec - annsAddStart.tv_nsec;
        // printf("Faiss add time: %f\n", annsAddTotal/(1000*1000));
        // searching...
        clock_gettime(CLOCK_REALTIME,&annsSearchStart);
        index->search(nq, searhVec, k, dis.data(), nns.data());
        clock_gettime(CLOCK_REALTIME,&annsSearchEnd);
        double annsSearchTotal = (annsSearchEnd.tv_sec -  annsSearchStart.tv_sec)* BILLION + annsSearchEnd.tv_nsec - annsSearchStart.tv_nsec;
        printf("Faiss Search time: %f\n", annsSearchTotal/(1000*1000));
    }
    else
    {
        // train index
        printf("Train faiss index\n");
        printf("feature_num = %ld\n", feature_num);

        // vector dims
        int d = jNode->leftTable->attrSize[jNode->leftKeyIndex] / sizeof(float);
        printf("dimensions = %d\n", d);

        // train feature num
        size_t nb = feature_num;
        
        printf("Initial faiss member parameters\n");
        faiss::gpu::StandardGpuResources resources;
        
        int remain = d % 4;
        // if d is not a multiple of 4, use FlatL2 index
        // else use IVFPQ
        if(remain or jNode->rightTable->tupleNum < 1000){
            // printf("dims is not a multiple of 4, use FlatL2\n");
            faiss::gpu::GpuIndexFlatConfig config;
            config.device = 0;
            faiss::gpu::GpuIndexFlatL2 gpuIndex(&resources, d, config);
            gpuIndex.add(nb, feature);

            // save index
            double t0 = elapsed();
            if(jNode->rightTable->attrName != NULL)
                write_index(faiss::gpu::index_gpu_to_cpu(&gpuIndex), index_path);
            printf("[%.3f s] Finish write_index() \n", elapsed() - t0);

            // searching...
            clock_gettime(CLOCK_REALTIME,&annsSearchStart);
            gpuIndex.search(nq, searhVec, k, dis.data(), nns.data());
            clock_gettime(CLOCK_REALTIME,&annsSearchEnd);
            double annsSearchTotal = (annsSearchEnd.tv_sec -  annsSearchStart.tv_sec)* BILLION + annsSearchEnd.tv_nsec - annsSearchStart.tv_nsec;
            printf("Faiss Search time: %f\n", annsSearchTotal/(1000*1000));
        }else{
            int ncentroids = int(sqrt(feature_num));
            faiss::gpu::GpuIndexIVFPQConfig config;
            config.device = dev_no;
            faiss::gpu::GpuIndexIVFPQ index(&resources, d, ncentroids, 4, 8, faiss::METRIC_L2, config);

            double t0 = elapsed();
            printf(" Generating %ld vectors in %dD for training\n",
                nb,
                d);

            index.train(feature_num, feature);
            printf("[%.3f s] Finish training \n", elapsed() - t0);
            
            index.add(feature_num, feature);

            // save index
            t0 = elapsed();
            if(jNode->rightTable->attrName != NULL)
                write_index(faiss::gpu::index_gpu_to_cpu(&index), index_path);
            printf("[%.3f s] Finish write_index() \n", elapsed() - t0);
            
            clock_gettime(CLOCK_REALTIME,&annsSearchStart);
            
            // searching...
            index.search(nq, searhVec, k, dis.data(), nns.data());

            clock_gettime(CLOCK_REALTIME,&annsSearchEnd);
            double annsSearchTotal = (annsSearchEnd.tv_sec -  annsSearchStart.tv_sec)* BILLION + annsSearchEnd.tv_nsec - annsSearchStart.tv_nsec;
            printf("Faiss Search time: %f\n", annsSearchTotal/(1000*1000));
        }
        
    }

    // for(int i = 0; i < nq; i++){
    //     float distance = 0.0;
    //     for(int j = 0; j < k; j++){
    //         distance += dis[i*k + j];
    //     }
    //     printf("%f\n", distance);
    // }

    clock_gettime(CLOCK_REALTIME,&annsEnd);
    pp->knn_ANNs_time += (annsEnd.tv_sec -  annsStart.tv_sec)* BILLION + annsEnd.tv_nsec - annsStart.tv_nsec;

    // for (int i = 0; i < nq; i++) {
    // for (int i = 0; i < 1; i++) {
    //     printf("query %2d: ", i);
    //     for (int j = 0; j < k; j++) {
    //         printf("%7ld ", nns[j + i * k]);
    //         printf(" %f ", disArray[nns[j + i * k]*19]);
    //     }
    //     printf("\n");
    // }

    // for(int i = 0; i < 100; i++){
    //     printf("search feature[%d] = ", i);
    //     for(int j = 0; j < 19; j++){
    //         printf("%f\t", disArray[i*19 + j]);
    //     }
    //     printf("\n");
    // }

    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_nns, sizeof(faiss::idx_t) * nns.size()));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_nns, nns.data(), sizeof(faiss::idx_t) * nns.size(), hipMemcpyHostToDevice));

    clock_gettime(CLOCK_REALTIME,&start);
    
    /* join two table by nns */
    for(int i=0; i<res->totalAttr; i++){
        int index, pos;
        long colSize = 0, resSize = 0;
        int leftRight = 0;

        int attrSize, attrType;
        char * table = NULL;
        int found = 0 , dataPos, format;

        if (jNode->keepInGpu[i] == 1)
            res->dataPos[i] = GPU;
        else
            res->dataPos[i] = MEM;

        for(int j=0;j<jNode->leftOutputAttrNum;j++){
            if (jNode->leftPos[j] == i){
                found = 1;
                leftRight = 0;
                pos = j;
                break;
            }
        }
        if(!found){
            for(int j=0;j<jNode->rightOutputAttrNum;j++){
                if(jNode->rightPos[j] == i){
                    found = 1;
                    leftRight = 1;
                    pos = j;
                    break;
                }
            }
        }

        if(leftRight == 0){
            index = jNode->leftOutputIndex[pos];
            dataPos = jNode->leftTable->dataPos[index];
            format = jNode->leftTable->dataFormat[index];
            attrSize  = jNode->leftTable->attrSize[index];
            attrType  = jNode->leftTable->attrType[index];
            colSize = jNode->leftTable->attrTotalSize[index];

            if(dataPos == MEM) {
                CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&table, colSize));
                CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(table, jNode->leftTable->content[index], colSize, hipMemcpyHostToDevice));
            } 
            else {
                table = jNode->leftTable->content[index];
            }

            resSize = res->tupleNum * attrSize;
        }else{
            index = jNode->rightOutputIndex[pos];
            dataPos = jNode->rightTable->dataPos[index];
            format = jNode->rightTable->dataFormat[index];

            table = jNode->rightTable->content[index];
            attrSize = jNode->rightTable->attrSize[index];
            attrType = jNode->rightTable->attrType[index];
            colSize = jNode->rightTable->attrTotalSize[index];

            if(dataPos == MEM) {
                CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&table, colSize));
                CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(table, jNode->rightTable->content[index], colSize, hipMemcpyHostToDevice));
            } 
            else {
                table = jNode->rightTable->content[index];
            }

            resSize = attrSize * res->tupleNum;
            leftRight = 1;
        }

        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_result,resSize));

        if(leftRight == 0){

            if(format == UNCOMPRESSED){
                /* left table make k copies by tuple */
                // int numBlocks = 1;
                // int threadsPerBlock = jNode->leftTable->tupleNum;
                // copy_left_table<<<numBlocks, threadsPerBlock>>>(table, attrSize, jNode->leftTable->tupleNum, k, gpu_result);
                int threads = 256;
                int blocks = (jNode->leftTable->tupleNum + threads - 1) / threads;
                copy_left_table<<<blocks, threads>>>(table, attrSize, jNode->leftTable->tupleNum, k, gpu_result);
                CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());
            }
            else{
                printf("Datatype isn't compressed!");
                exit(1);
            }

        }else{

            if(format == UNCOMPRESSED){
                /* right table copy by nns */
                CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());
                // int threadsPerBlock = res->tupleNum;
                // copy_right_table<<<1, threadsPerBlock>>>(table, attrSize, res->tupleNum, gpu_nns, gpu_result);
                int threads = 256;
                int blocks = (res->tupleNum + threads - 1) / threads;
                copy_right_table<<<blocks, threads>>>(table, attrSize, res->tupleNum, gpu_nns, gpu_result);
                CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

            }else{
                printf("Datatype isn't compressed!");
                exit(1);
            }

        }
        
        CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

        res->dataFormat[i] = UNCOMPRESSED;
        if(res->dataPos[i] == MEM){
            res->content[i] = (char *) malloc(resSize);
            CHECK_POINTER(res->content[i]);
            memset(res->content[i],0,resSize);
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(res->content[i],gpu_result,resSize,hipMemcpyDeviceToHost));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_result));

        }else if(res->dataPos[i] == GPU){
            res->content[i] = gpu_result;
        }
    }
    CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());
    clock_gettime(CLOCK_REALTIME,&end);
    double total = (end.tv_sec -  start.tv_sec)* BILLION + end.tv_nsec - start.tv_nsec;
    printf("copy data time: %f\n", total/(1000*1000));

    //Stop total timer
    clock_gettime(CLOCK_REALTIME, &endS0);
    pp->knnJoin_totalTime += (endS0.tv_sec - startS0.tv_sec)* BILLION + endS0.tv_nsec - startS0.tv_nsec;
    printf("[Info] knnJoin_totalTime = %lf\n", pp->knnJoin_totalTime/(1000*1000));
    
    //Increase count call
    pp->join_callTimes++;

    printf("[knnJoin] res->tupleNum = %ld\n", res->tupleNum);
    // float disArray[1024*1024];
    // char intArray[2048];
    // printf("[knnJoin] the type of result[2] :%d\n", res->attrType[3]);
    // hipMemcpy(disArray, res->content[3], res->attrSize[3] * res->tupleNum, hipMemcpyDeviceToHost);
    // for(int i = 0; i < 10; i++){
    //     printf("test feature[%d] = ", i);
    //     for(int j = 0; j < 19; j++){
    //         printf("%f\t", disArray[i*19 + j]);
    //     }
    //     printf("\n");
    // }
    return res;

}

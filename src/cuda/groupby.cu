#include "hip/hip_runtime.h"
/*
   Copyright (c) 2012-2013 The Ohio State University.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <time.h>
#include "../include/common.h"
#include "../include/gpuCudaLib.h"
#include "../include/cudaHash.h"
#include "../include/Mempool.h"
#include "scanImpl.cu"

#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <faiss/IndexFlat.h>
#include <faiss/gpu/GpuIndexFlat.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/impl/IndexUtils.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <vector>
#include <string>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <random>
#include <set>
#include <thread>
#include <map>
#include <queue>
#include <functional>
#include <mutex>
#include <condition_variable>
#include <algorithm>

const float THRESHOLD = 2.8;

const int MATCH = 3;
const int MISMATCH = -3;
const int GAP = -2;

std::mutex resmap_mutex;

int threshold(float num){
    if(num < THRESHOLD){
        return 1;
    }
    return 0;
}

/*
    ThreadPool for multi-thread VIDEO_SIMILARITY()
*/
class ThreadPool {
public:
    ThreadPool(size_t threads);
    ~ThreadPool();

    template<class F, class... Args>
    void enqueue(F&& f, Args&&... args);

private:
    std::vector<std::thread> workers;
    std::queue<std::function<void()>> tasks;

    std::mutex queue_mutex;
    std::condition_variable condition;
    bool stop;
    int activeThreads = 0;  // 跟踪当前正在执行任务的线程数

public:
    bool allTasksCompleted();  // 函数检查所有任务是否完成

};

ThreadPool::ThreadPool(size_t threads) : stop(false) {
    for(size_t i = 0; i < threads; ++i)
        workers.emplace_back([this] {
            for(;;) {
                std::function<void()> task;
                {
                    std::unique_lock<std::mutex> lock(this->queue_mutex);
                    this->condition.wait(lock, [this] { return this->stop || !this->tasks.empty(); });
                    if(this->stop && this->tasks.empty()) return;
                    task = std::move(this->tasks.front());
                    this->tasks.pop();

                    this->activeThreads++;  // 开始执行任务，增加activeThreads计数
                }
                task();
                {
                    std::unique_lock<std::mutex> lock(this->queue_mutex);
                    this->activeThreads--;  // 任务完成，减少activeThreads计数
                }
            }
        });
}

ThreadPool::~ThreadPool() {
    {
        std::unique_lock<std::mutex> lock(queue_mutex);
        stop = true;
    }
    condition.notify_all();
    for(std::thread &worker: workers)
        worker.join();
}

bool ThreadPool::allTasksCompleted() {
    std::unique_lock<std::mutex> lock(queue_mutex);
    return tasks.empty() && activeThreads == 0;
}

template<class F, class... Args>
void ThreadPool::enqueue(F&& f, Args&&... args) {
    {
        std::unique_lock<std::mutex> lock(queue_mutex);
        tasks.emplace(std::bind(std::forward<F>(f), std::forward<Args>(args)...));
    }
    condition.notify_one();
}

float computeSumForSegment(const float* dist, int start, int k) {
    float sum = 0.0f;
    for (int i = 1; i < k; ++i) {
        sum += dist[start + i];
    }
    return sum;
}

int matchScore(int sim) {
    return sim == 1 ? MATCH : MISMATCH;
}

void SWAlign(std::vector<std::vector<int>> &simVec, int search_start, int search_end, int ref_start, int ref_end, int &max_score, std::vector<int> &seq){

    if(search_start < search_end && ref_start < ref_end){
        /* Backtracking: cal subsequence and score, add subsequence to seq
            1. initial dp matrix, get max score and address
            2. backtracking and record subsequence of search video and reference video
        */
        int len1 = search_end - search_start + 1;
        int len2 = ref_end - ref_start + 1;
        
        std::vector<std::vector<int>> dp(len1 + 1, std::vector<int>(len2 + 1, 0));
        
        int max_i = 0, max_j = 0;
        int start_i = 0, start_j = 0;

        // Fill the dp matrix
        for(int i = 1; i <= len1; i++) {
            for(int j = 1; j <= len2; j++) {
                int match = dp[i - 1][j - 1] + matchScore(simVec[i-1 + search_start][j-1 + ref_start]);
                int delete_op = dp[i - 1][j] + GAP;
                int insert_op = dp[i][j - 1] + GAP;
                
                dp[i][j] = std::max({0, match, delete_op, insert_op});
                
                if(dp[i][j] > max_score) {
                    max_score = dp[i][j];
                    max_i = i;
                    max_j = j;
                }
            }
        }

        int i = max_i;
        int j = max_j;
        while(i > 0 && j > 0 && dp[i][j] != 0) {
            if(dp[i][j] == dp[i-1][j-1] + matchScore(simVec[i-1 + search_start][j-1 + ref_start])) {
                i--;
                j--;
                start_i = i;
                start_j = j;
            } else if(dp[i][j] == dp[i-1][j] + GAP) {
                i--;
                start_i = i;
                start_j = j;
            } else {
                j--;
                start_i = i;
                start_j = j;
            }
        }
        seq[0] = start_i + search_start;
        seq[1] = max_i + search_start;
        seq[2] = start_j + ref_start;
        seq[3] = max_j + ref_start;
    } else {
        max_score = 0;
    }
}

int RSWAlign(std::vector<std::vector<int>> &simVec, int search_start, int search_end, int ref_start, int ref_end, std::vector<std::vector<int>> &subsequence, std::vector<int> &seqScore){
    int score = 0;
    std::vector<int> seq{0, 0, 0, 0};
    
    // printf("Begin RSWAlign\n");
    // printf("search_start = %d, search_end = %d, ref_start = %d, ref_end = %d\n", search_start, search_end, ref_start, ref_end);
    // cal seq
    SWAlign(simVec, search_start, search_end, ref_start, ref_end, score, seq);
    // printf("\nSWAlign in RSWAlign\n");
    // printf("seq[0] = %d, seq[1] = %d, seq[2] = %d, seq[3] = %d\n", seq[0], seq[1], seq[2], seq[3]);

    if(seq[0] < seq[1] && seq[2] < seq[3]){
        subsequence.push_back(seq);
        seqScore.push_back(score);
        // left
        // printf("\nleft in RSWAlign\n");
        score += RSWAlign(simVec, search_start, seq[0]-1, ref_start, seq[2]-1, subsequence, seqScore);

        // right
        // printf("\nright in RSWAlign\n");
        score += RSWAlign(simVec, seq[1]+1, search_end, seq[3]+1, ref_end, subsequence, seqScore);
    }
    return score;
}

std::vector<std::vector<int>> convertToSimVec(const std::vector<float>& distVec, int numRows, int numCols) {
    std::vector<std::vector<int>> simVec(numRows, std::vector<int>(numCols));

    for(int i = 0; i < numRows; i++) {
        for(int j = 0; j < numCols; j++) {
            simVec[i][j] = threshold(distVec[i * numCols + j]);
        }
    }

    return simVec;
}

int calScore(std::vector<float> &distVec, int num, std::vector<std::vector<int>> &seq, std::vector<int> &seqScore) {
    // cal score
    int refLen = distVec.size() / num;
    std::vector<std::vector<int>> similarityVec = convertToSimVec(distVec, num, refLen);

    // printf("Breakpoint in calScore\n");
    int score = RSWAlign(similarityVec, 0, num-1, 0, refLen-1, seq, seqScore);
    return score;
}

void normalize(float* array, int size) {
    if (size <= 0) {
        return;  // 防止除以零
    }

    // 找到最小值和最大值
    float min = array[0];
    float max = array[0];

    for (int i = 1; i < size; i++) {
        if (array[i] < min) {
            min = array[i];
        }
        if (array[i] > max) {
            max = array[i];
        }
    }

    printf("[normalize] max = %f, min = %f\n", max, min);

    // 归一化数组元素
    for (int i = 0; i < size; i++) {
        array[i] = (array[i] - min) / (max - min);
    }
}

__global__ void copyElement(float* data, int size, float* result) {
    int index = (int)(0.999 * size); // 99.9% position
    *result = data[index];
}

void sortAndFindElement(float* data, int size, float* result) {
    // Use thrust to sort
    thrust::device_ptr<float> dev_ptr(data);
    thrust::sort(dev_ptr, dev_ptr + size);

    // Launch kernel to copy 90% element to result
    copyElement<<<1, 1>>>(data, size, result);

    hipDeviceSynchronize();
}

/*
 * Combine the group by columns to build the group by keys. 
 */

__global__ static void build_groupby_key(char ** content, int gbColNum, int * gbIndex, int * gbType, int * gbSize, long tupleNum, int * key, int *num, int* groupNum){

    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    for(long i = offset; i< tupleNum; i+= stride){
        char buf[128] = {0};
        int hkey;
        // if(gbColNum == 1 && gbIndex[0] != -1 && gbType[0] == INT) {
        //     int k = ((int *)(content[gbIndex[0]]))[i];
        //     hkey = k % HSIZE;
        // }else
        {
            for (int j=0; j< gbColNum; j++){
                char tbuf[32]={0};
                int index = gbIndex[j];

                if (index == -1){
                    gpuItoa(1,tbuf,10);
                    gpuStrncat(buf,tbuf,1);

                }else if (gbType[j] == STRING){
                    gpuStrncat(buf, content[index] + i*gbSize[j], gbSize[j]);

                }else if (gbType[j] == INT){
                    int key = ((int *)(content[index]))[i];
                    gpuItoa(key,tbuf,10);
                    gpuStrcat(buf,tbuf);
                }
            }
            if(gbColNum == 1 && gbIndex[0] != -1 && gbType[0] == INT)
                hkey = StringHashInt(buf) % HSIZE;
            else
                hkey = StringHash(buf) % HSIZE;
        }
        key[i]= hkey;
        num[hkey] = 1;
        atomicAdd(&(groupNum[hkey]), 1);
    }
}


/*
 * Count the number of groups 
 */

__global__ static void count_group_num(int *num, int tupleNum, int *totalCount){
        int stride = blockDim.x * gridDim.x;
        int offset = blockIdx.x * blockDim.x + threadIdx.x;
        int localCount = 0;

        for(int i=offset; i<tupleNum; i+= stride){
                if(num[i] == 1){
                        localCount ++;
                }
        }

        atomicAdd(totalCount,localCount);
}

/*
 * Calculate the groupBy expression.
 */

__device__ static float calMathExp(char **content, struct mathExp exp, int pos){
    float res ;

    if(exp.op == NOOP){
        if (exp.opType == CONS)
            res = exp.opValue;
        else if(exp.opType == COLUMN_INTEGER){
            int index = exp.opValue;
            res = ((int *)(content[index]))[pos];
        }else if(exp.opType == COLUMN_DECIMAL){
            int index = exp.opValue;
            res = ((float *)(content[index]))[pos];
        }
    
    }else if(exp.op == PLUS ){
        res = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos) + calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);

    }else if (exp.op == MINUS){
        res = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos) - calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);

    }else if (exp.op == MULTIPLY){
        res = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos) * calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);

    }else if (exp.op == DIVIDE){
        res = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos) / calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);
    }

    return res;
}


__device__ __forceinline__ float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

__device__ __forceinline__ float atomicMinFloat (float * addr, float value) {
        float old;
        old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
             __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));

        return old;
}

/*
 * group by constant. Currently only support SUM function.
 */

__global__ static void agg_cal_cons(char ** content, int colNum, struct groupByExp* exp, long tupleNum, char ** result){

    int stride = blockDim.x * gridDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float buf[32];
    for(int i = 0; i < colNum; i++){
        int func = exp[i].func;
        if(func == MAX)
            buf[i] = FLOAT_MIN;
        else if(func == MIN)
            buf[i] = FLOAT_MAX;
        else
            buf[i] = 0;
    }

    for(int i=index;i<tupleNum;i+=stride){
        for(int j=0;j<colNum;j++){
            int func = exp[j].func;
            if (func == SUM){
                float tmpRes = calMathExp(content, exp[j].exp, i);
                buf[j] += tmpRes;
            }else if (func == AVG){

                float tmpRes = calMathExp(content, exp[j].exp, i)/tupleNum;
                buf[j] += tmpRes;
            }else if (func == MAX){

                float tmpRes = calMathExp(content, exp[j].exp, i);
                buf[j] = buf[j] > tmpRes ? buf[j] : tmpRes;
            }else if (func == MIN){

                float tmpRes = calMathExp(content, exp[j].exp, i);
                buf[j] = buf[j] < tmpRes ? buf[j] : tmpRes;
            }
        }
    }

    for(int i=0;i<colNum;i++)
    {
        int func = exp[i].func;
        if (func == SUM)
            atomicAdd(&((float *)result[i])[0], buf[i]);
        else if (func == MAX)
            atomicMaxFloat(&((float *)result[i])[0], buf[i]);
        else if (func == MIN)
            atomicMinFloat(&((float *)result[i])[0], buf[i]);
    }
}

/*
 * gropu by
 */

__global__ static void agg_cal(char ** content, int colNum, struct groupByExp* exp, int * gbType, int * gbSize, long tupleNum, int * key, \
    int *psum, int * groupNum, char ** result, int *dkeys = nullptr, float* dvalues = nullptr, int dnum = 0){

    int stride = blockDim.x * gridDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=index;i<tupleNum;i+=stride){

        int hKey = key[i];
//        int offset = atomicAdd(&(psum[hKey]), 1);
        int offset = psum[hKey];


        for(int j=0;j<colNum;j++){
            int func = exp[j].func;
            if(func ==NOOP){
                int type = exp[j].exp.opType;

                if(type == CONS){
                    int value = exp[j].exp.opValue;
                    ((int *)result[j])[offset] = value;
                }else{
                    int index = exp[j].exp.opValue;
                    int attrSize = gbSize[j];
                    if(attrSize == sizeof(int))
                        ((int *)result[j])[offset] = ((int*)content[index])[i];
                    else
                        memcpy(result[j] + offset*attrSize, content[index] + i * attrSize, attrSize);
                }

            }else if (func == SUM ){
                float tmpRes = calMathExp(content, exp[j].exp, i);
                atomicAdd(& ((float *)result[j])[offset], tmpRes);
            }else if (func == MAX ){
                float tmpRes = calMathExp(content, exp[j].exp, i);
                atomicMaxFloat(& ((float *)result[j])[offset], tmpRes);

            }else if (func == MIN ){
                float tmpRes = calMathExp(content, exp[j].exp, i);
                atomicMinFloat(& ((float *)result[j])[offset], tmpRes);

            }else if (func == AVG){
                float tmpRes = calMathExp(content, exp[j].exp, i)/groupNum[hKey];
                atomicAdd(& ((float *)result[j])[offset], tmpRes);
            }else if (func == OUTLIER_SCORE){
                float tmpRes = (( (float*) (content[j]) )[i]);
                atomicMaxFloat(& ((float *)result[j])[offset], tmpRes);
            }
        }
    }
    if(dkeys != NULL){
        for(int i=0; i<colNum; i++){
            int func = exp[i].func;
            if (func == VIDEO_SIMILARITY){
                for(int j=0; j<dnum; j++){
                    int hKey = dkeys[j];
                    int offset = psum[hKey];
                    ((float *)result[i])[offset] = dvalues[j];
                }
            }
        }
    }
}

void loadFloatArray(std::vector<float>& buffer, char* filename) {
    FILE* file = fopen(filename, "rb");
    if (file != nullptr) {
        // 获取文件大小
        fseek(file, 0, SEEK_END);
        long fileSize = ftell(file);
        fseek(file, 0, SEEK_SET);
        // 计算数组元素数量
        std::size_t count = fileSize / sizeof(float);
        printf("[DISTANCE Info]searchVec dimensions = %ld\n", count);
        // 调整向量大小以容纳数据
        buffer.resize(count);
        // buffer[0] = 1;
        // 读取数据到向量
        fread(buffer.data(), sizeof(float), count, file);
        fclose(file);
    } else {
        std::cerr << "Failed to open file." << std::endl;
    }
}

__global__ static void init_int_array(int *array, int array_size, int init_value)
{
    int stride = blockDim.x * gridDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = index; i < array_size; i += stride)
        array[i] = init_value;
}

__global__ void mergeAndSum(float* data, int num, int k, float* res) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < num) {
        float sum = 0.0f;
        float* vector = &data[tid * k];

        for (int i = 1; i < k; i++) {
            sum += vector[i];
        }

        res[tid] = sum;
    }
}

__global__ void findValue(float* data, int size, float* result) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // 每个线程处理一个数据
    if (tid < size) {
        // 在共享内存中拷贝数据
        extern __shared__ float sharedData[];
        sharedData[tid] = data[tid];
        __syncthreads();

        // 使用并行排序算法对数据进行排序
        for (int stride = 1; stride < size; stride *= 2) {
            int index = 2 * stride * tid;
            if (index < size) {
                // 进行合并排序
                float temp = sharedData[index];
                if (index + stride < size) {
                    float nextValue = sharedData[index + stride];
                    if (temp > nextValue) {
                        sharedData[index] = nextValue;
                        sharedData[index + stride] = temp;
                    }
                }
            }
            __syncthreads();
        }

        // 将位置在 90% 处的值输出
        if (tid == static_cast<int>(size * 0.9) - 1) {
            *result = sharedData[tid];
        }
    }
}

float calcL2Distance(const std::vector<float>& vec1, const std::vector<float>& vec2, int startIdx1, int startIdx2, int vectorSize) {
    float dist = 0.0f;
    for (int i = 0; i < vectorSize; i++) {
        float diff = vec1[startIdx1 + i] - vec2[startIdx2 + i];
        dist += diff * diff;
    }
    return std::sqrt(dist);
}

std::map<int, std::vector<float>> calcDistances(std::vector<float>& searchV, std::map<int, std::vector<float>>& hashmap, int vectorSize) {
    std::map<int, std::vector<float>> result;
    for (auto& pair : hashmap) {
        int key = pair.first;
        std::vector<float>& vectors = pair.second;
        for (int i = 0; i < searchV.size(); i += vectorSize) {
            for (int j = 0; j < vectors.size(); j += vectorSize) {
                float dist = calcL2Distance(searchV, vectors, i, j, vectorSize);
                result[key].push_back(dist);
            }
        }
    }
    return result;
}

std::map<int, float> calVideoSimlarity(std::map<int, std::vector<float>> distmap, int searchFrameNum){
    // fake algorithm
    std::map<int, float> res;
    float count = 1.0;
    for(auto &pair : distmap){
        printf("pair.first = %d\n", pair.first);
        res[pair.first] = count;
        count += 1.0;
    }

    // thinking... true algorithm

    return res;
}

/* 
 * groupBy: group by the data and calculate. 
 * 
 * Prerequisite:
 *  input data are not compressed
 *
 * Input:
 *  gb: the groupby node which contains the input data and groupby information
 *  pp: records the statistics such as kernel execution time 
 *
 * Return:
 *  a new table node
 */

struct tableNode * groupBy(struct groupByNode * gb, struct statistic * pp,
                           Mempool *host_mp = NULL, Mempool *dev_mp = NULL){

    
    printf("\n=============== [groupby.cu] start =============\n");

    float disArray[1024*1024];
    int intArray[40960];
    // printf("[groupBy] attrType[2] :%d\n", gb->table->attrType[2]);
    // hipMemcpy(disArray, gb->table->content[2], gb->table->attrSize[2] * gb->table->tupleNum, hipMemcpyDeviceToHost);
    // for(int i = 0; i < 10; i++){
    //     printf("test feature[%d] = ", i);
    //     for(int j = 0; j < 10; j++){
    //         printf("%f\t", disArray[i*10 + j]);
    //     }
    //     printf("\n");
    // }

    // printf("[groupBy] attrType[0] :%d\n", gb->table->attrType[0]);
    // hipMemcpy(intArray, gb->table->content[0], gb->table->attrSize[0] * gb->table->tupleNum, hipMemcpyDeviceToHost);
    // for(int i = 0; i < 100; i++){
    //     for(int j = 0; j < 10; j++){
    //         printf("%d\t", intArray[i * 10 + j]);
    //     }
    //     printf("\n");
    // }

    // printf("[groupBy] attrType[1] :%d\n", gb->table->attrType[1]);
    // hipMemcpy(intArray, gb->table->content[1], gb->table->attrSize[1] * gb->table->tupleNum, hipMemcpyDeviceToHost);
    // for(int i = 0; i < 10; i++){
    //     for(int j = 0; j < 10; j++){
    //         printf("%d\t", intArray[i * 10 + j]);
    //     }
    //     printf("\n");
    // }

    // char intArray[16384*16];
    // printf("[groupby.cu] the type of result[0] :%d\n", gb->table->attrType[0]);
    // printf("[groupby.cu] the type of result[1] :%d\n", gb->table->attrType[1]);
    // hipMemcpy(intArray, gb->table->content[0], sizeof(char) * 23 * 400*16, hipMemcpyDeviceToHost);
    // for(int i = 0; i < 400*16; i++){
    //     printf("video_name[%d] = ", i);
	// 	for(int j = 0; j < 23; j++){
    //         printf("%c", intArray[i*23 + j]);
    //     }
    //     printf("\n");
	// }

    //Start total timer
    struct timespec startS0, endS0;
    struct timespec s, e;
    double tt;
    clock_gettime(CLOCK_REALTIME,&s);
    clock_gettime(CLOCK_REALTIME,&startS0);

    //Start timer for Step 1 - Allocate memory for intermediate results
    struct timespec startS1, endS1;
    clock_gettime(CLOCK_REALTIME,&startS1);

    int *gpuGbIndex = NULL, gpuTupleNum, gpuGbColNum;
    int *gpuGbType = NULL, *gpuGbSize = NULL;

    int *gpuGbKey = NULL;
    char ** gpuContent = NULL, **column = NULL;

    /*
     * @gbCount: the number of groups
     * gbConstant: whether group by constant
     */

    int gbCount;
    int gbConstant = 0;

    struct tableNode *res = NULL;
    if(host_mp == NULL) {
        res = (struct tableNode *) malloc(sizeof(struct tableNode));
        CHECK_POINTER(res);
    } else
        res = (struct tableNode *) host_mp->alloc(sizeof(struct tableNode));

    res->tupleSize = gb->tupleSize;
    res->totalAttr = gb->outputAttrNum;

    if(host_mp == NULL) {
        res->attrType = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->attrType);
        res->attrSize = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->attrSize);
        res->attrTotalSize = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->attrTotalSize);
        res->dataPos = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->dataPos);
        res->dataFormat = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->dataFormat);
        res->content = (char **) malloc(sizeof(char **) * res->totalAttr);
        CHECK_POINTER(res->content);
    } else {
        res->attrType = (int *) host_mp->alloc((sizeof(int) * res->totalAttr));
        res->attrSize = (int *) host_mp->alloc((sizeof(int) * res->totalAttr));
        res->attrTotalSize = (int *) host_mp->alloc((sizeof(int) * res->totalAttr));
        res->dataPos = (int *) host_mp->alloc((sizeof(int) * res->totalAttr));
        res->dataFormat = (int *) host_mp->alloc(sizeof(int) * res->totalAttr);
        res->content = (char **) host_mp->alloc(sizeof(char **) * res->totalAttr);
    }

    for(int i=0;i<res->totalAttr;i++){
        res->attrType[i] = gb->attrType[i];
        res->attrSize[i] = gb->attrSize[i];
        res->dataFormat[i] = UNCOMPRESSED;
        printf("res->attrType[%d] = %d\n", i, gb->attrType[i]);
    }

    gpuTupleNum = gb->table->tupleNum;
    int alignedGpuTupleNum = gpuTupleNum;
    NP2(alignedGpuTupleNum);
    printf("[groupby] tupleNum = %d\n", gpuTupleNum);
    gpuGbColNum = gb->groupByColNum;

    // if video_similarity() function exist = 1, else = 0
    int video_similarity_exist = 0;
    // the index of video_similarity() function input column, if not exist = -1
    int video_similarity_index = -1;
    // the index of video_similarity() function, 0:pathVideo 1:tableVideo
    int video_similarity_type = -1;

    int outlier_score_exist = 0;
    int outlier_score_index = -1;

    int searchFrameNum;

    for(int i = 0; i < res->totalAttr; i++) {
        if (gb->gbExp[i].func == OUTLIER_LINE) {
            int colIndex = gb->gbExp[i].index;
            res->attrType[colIndex] = FLOAT;
            res->attrSize[colIndex] = sizeof(float);
        }
        else if (gb->gbExp[i].func == VIDEO_SIMILARITY) {
            video_similarity_exist = 1;
            video_similarity_index = gb->gbExp[i].index;
            video_similarity_type = gb->gbExp[i].type;
            if(video_similarity_type == 1)
                searchFrameNum = gb->gbExp[i].searchFrameNum;
            res->attrType[video_similarity_index] = FLOAT;
            res->attrSize[video_similarity_index] = sizeof(float);
        }
        else if (gb->gbExp[i].func == OUTLIER_SCORE) {
            outlier_score_exist = 1;
            int colIndex = gb->gbExp[i].index;
            outlier_score_index = colIndex;
            res->attrType[colIndex] = FLOAT;
            res->attrSize[colIndex] = sizeof(float);
        }
    }

    if(gpuGbColNum == 1 && gb->groupByIndex[0] == -1){
        gbConstant = 1;
    }

    dim3 grid(1024);
    dim3 block(128);
    int blockNum = gpuTupleNum / block.x + 1;
    if(blockNum < 1024)
        grid = blockNum;

    //hashTable: gpu_hashNum[hkey] = 1, hkey = hash(element)
    int *gpu_hashNum = NULL;
    
    int *gpu_psum = NULL;
    
    //the number of groups
    int *gpuGbCount = NULL;

    //hashTable: gpuGbCount[hkey] += 1, hkey = hash(element)
    int *gpu_groupNum = NULL;

    if(dev_mp == NULL)
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuContent, gb->table->totalAttr * sizeof(char *)));
    else
        gpuContent = (char **) dev_mp->alloc(gb->table->totalAttr * sizeof(char *));

    if(host_mp == NULL) {
        column = (char **) malloc(sizeof(char *) * gb->table->totalAttr);
        CHECK_POINTER(column);
    } else
        column = (char **) host_mp->alloc(sizeof(char *) * gb->table->totalAttr);

    printf("gb->table->totalAttr = %d\n", gb->table->totalAttr);

    //Stop for Step 1 - Allocate memory for intermediate results
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS1);
    pp->groupby_step1_allocMem += (endS1.tv_sec - startS1.tv_sec)* BILLION + endS1.tv_nsec - startS1.tv_nsec;
    
    //Start timer for Step 2 - Copy data to GPU
    struct timespec startS2, endS2;
    clock_gettime(CLOCK_REALTIME,&startS2);

    for(int i=0;i<gb->table->totalAttr;i++){
        int attrSize = gb->table->attrSize[i];
        if(gb->table->dataPos[i]==MEM){
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)& column[i], attrSize * gb->table->tupleNum));
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[i], gb->table->content[i], attrSize * gb->table->tupleNum, hipMemcpyHostToDevice));

            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuContent[i], &column[i], sizeof(char *), hipMemcpyHostToDevice));
        }else{
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuContent[i], &gb->table->content[i], sizeof(char *), hipMemcpyHostToDevice));
        }
    }

    //Stop for Step 2 - Copy data to GPU
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS2);
    pp->groupby_step2_copyToDevice += (endS2.tv_sec - startS2.tv_sec)* BILLION + endS2.tv_nsec - startS2.tv_nsec;
            
    if(gbConstant != 1){

        //Start timer for Step 3 - build_groupby_key kernel
        struct timespec startS3, endS3;
        clock_gettime(CLOCK_REALTIME,&startS3);

        if (dev_mp == NULL)
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbType, sizeof(int) * gb->groupByColNum));
        else
            gpuGbType = (int *) dev_mp->alloc(sizeof(int) * gb->groupByColNum);
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbType,gb->groupByType, sizeof(int) * gb->groupByColNum, hipMemcpyHostToDevice));

        if (dev_mp == NULL)
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbSize, sizeof(int) * gb->groupByColNum));
        else
            gpuGbSize = (int *) dev_mp->alloc(sizeof(int) * gb->groupByColNum);
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbSize,gb->groupBySize, sizeof(int) * gb->groupByColNum, hipMemcpyHostToDevice));

        if (dev_mp == NULL)
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbKey, alignedGpuTupleNum * sizeof(int)));
        else
            gpuGbKey = (int *) dev_mp->alloc(alignedGpuTupleNum * sizeof(int));

        if (dev_mp == NULL)
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbIndex, sizeof(int) * gb->groupByColNum));
        else
            gpuGbIndex = (int *) dev_mp->alloc(sizeof(int) * gb->groupByColNum);
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbIndex, gb->groupByIndex,sizeof(int) * gb->groupByColNum, hipMemcpyHostToDevice));

        if (dev_mp == NULL)
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_hashNum,sizeof(int)*HSIZE));
        else
            gpu_hashNum = (int *)dev_mp->alloc(sizeof(int) * HSIZE);
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpu_hashNum,0,sizeof(int)*HSIZE));

        if (dev_mp == NULL)
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_groupNum,sizeof(int)*HSIZE));
        else
            gpu_groupNum = (int *) dev_mp->alloc(sizeof(int) * HSIZE);
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpu_groupNum,0,sizeof(int)*HSIZE));

        build_groupby_key<<<grid,block>>>(gpuContent,gpuGbColNum, gpuGbIndex, gpuGbType,gpuGbSize,gpuTupleNum, gpuGbKey, gpu_hashNum, gpu_groupNum);
        CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

        if (dev_mp == NULL) {
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbType));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbSize));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbIndex));
        }

        gbCount = 1;

        if (dev_mp == NULL)
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbCount,sizeof(int)));
        else
            gpuGbCount = (int *) dev_mp->alloc(sizeof(int));
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpuGbCount, 0, sizeof(int)));

        //Stop for Step 3 - build_groupby_key kernel
        CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
        clock_gettime(CLOCK_REALTIME, &endS3);
        pp->groupby_step3_buildGroupByKey += (endS3.tv_sec - startS3.tv_sec)* BILLION + endS3.tv_nsec - startS3.tv_nsec;
    
        //Start timer for Step 4 - count_group_num
        struct timespec startS4, endS4;
        clock_gettime(CLOCK_REALTIME,&startS4);

        count_group_num<<<grid,block>>>(gpu_hashNum, HSIZE, gpuGbCount);
        CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gbCount, gpuGbCount, sizeof(int), hipMemcpyDeviceToHost));

        if (dev_mp == NULL)
            CUDA_SAFE_CALL(hipMalloc((void**)&gpu_psum,HSIZE*sizeof(int)));
        else
            gpu_psum = (int *) dev_mp->alloc(HSIZE * sizeof(int));
        scanImpl(gpu_hashNum,HSIZE,gpu_psum,pp);

        if (dev_mp == NULL) {
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbCount));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_hashNum));
        }

        //Stop for 4 - Count number of groups
        CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
        clock_gettime(CLOCK_REALTIME, &endS4);
        pp->groupby_step4_groupCount += (endS4.tv_sec - startS4.tv_sec)* BILLION + endS4.tv_nsec - startS4.tv_nsec;
    }

    if(gbConstant == 1)
        res->tupleNum = 1;
    else
        res->tupleNum = gbCount;
    printf("[INFO]Number of groupBy results: %ld\n",res->tupleNum);

    // init set<hkey,vector>
    int *gbKey;
    int d_num; /* number of videos */
    int* d_keys; /* keys of videos */
    float* d_values; /* values of videos */

    printf("video_similarity_exist = %d\n", video_similarity_exist);
    if (video_similarity_exist == 1 and video_similarity_type == 0) {
        // video_similarity(extraction(path), v1) 
        printf("video_similarity(extraction(path), v1)\n");
        gbKey = (int*)malloc(alignedGpuTupleNum * sizeof(int));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gbKey, gpuGbKey, alignedGpuTupleNum * sizeof(int), hipMemcpyDeviceToHost));
        printf("vecCol attrSize = %d\n", gb->table->attrSize[video_similarity_index]);
        printf("tupleNum = %ld\n", gb->table->tupleNum);
        int colSize = gb->table->attrSize[video_similarity_index] * gb->table->tupleNum;
        std::vector<float> vecCol(colSize);
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(vecCol.data(), gb->table->content[video_similarity_index], colSize, hipMemcpyDeviceToHost));
        // for(int i = 0; i < alignedGpuTupleNum; i++){
        //     printf("[gbKey] gbKey[%d] = %d\n", i, gbKey[i]);
        // }
        int vecSize = gb->table->attrSize[video_similarity_index] / sizeof(float);
        printf("vecSize = %d\n", vecSize);
        std::map<int, std::vector<float>> hashmap;
        for(int i = 0; i < gb->table->tupleNum; i++){
            int key = gbKey[i];
            std::vector<float> vectorToAdd;
            int startIdx = i * vecSize;
            // printf("startIdx = %d(i * 2048?)\n", startIdx);
            for (int j = 0; j < vecSize; ++j) {
                vectorToAdd.push_back(vecCol[startIdx + j]);
            }
            hashmap[key].insert(hashmap[key].end(), vectorToAdd.begin(), vectorToAdd.end());
        }
        std::vector<float> searchV;
        loadFloatArray(searchV, gb->gbExp[video_similarity_index].videoFeaturePath);
        searchFrameNum = searchV.size() / vecSize;
        printf("searchFrameNum = %d\n", searchFrameNum);
        // for(int i = 0; i < searchFrameNum; i++){
        //     printf("searchV[%d] = %f\n", i, searchV[i]);
        // }
        // process distance
        std::map<int, std::vector<float>> distmap;

        // for searchV
        //      for datasetV
        distmap = calcDistances(searchV, hashmap, vecSize);
        // for (const auto& pair : distmap) {
        //     int key = pair.first;
        //     const std::vector<float>& value = pair.second;

        //     printf("key = %d\n", key);
        //     printf("vec size = %d\n", value.size());
        //     printf("key video size = %d\n", value.size() * 2048 / searchV.size());
        // }
        
        //cal video_similarity
        ThreadPool pool(16);
        std::map<int, float> scoremap;

        
        // for(auto& pair : distmap) {
        //     int refLen = (pair.second).size() / searchFrameNum;
        //     for (int i = 0; i < searchFrameNum; ++i) {
        //         for (int j = 0; j < refLen; ++j) {
        //             std::cout << (pair.second)[i * refLen + j] << "\t";
        //         }
        //     std::cout << std::endl;
        //     }
        // }

        for(auto& pair : distmap) {
            pool.enqueue([&scoremap, &pair, searchFrameNum]() {
                std::vector<std::vector<int>> seq;
                std::vector<int> seqScore;
                int res = calScore(pair.second, searchFrameNum, seq, seqScore);
                // printf("score[%d] = %d\n", pair.first, res);
                std::lock_guard<std::mutex> lock(resmap_mutex);
                scoremap[pair.first] = res;
            });
        }

        while(1){
            if(pool.allTasksCompleted()) {
                std::cout << "All tasks are done!" << std::endl;
                break;
            }
        }

        std::vector<int> keys;
        std::vector<float> values;
        for (const auto& pair : scoremap) {
            keys.push_back(pair.first);
            values.push_back(pair.second);
        }
        d_num = keys.size();

        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_keys, keys.size() * sizeof(int)));
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_values, values.size() * sizeof(float)));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_keys, keys.data(), keys.size() * sizeof(int), hipMemcpyHostToDevice));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_values, values.data(), values.size() * sizeof(float), hipMemcpyHostToDevice));
    
        // for(int i = 0; i < values.size(); i++){
        //     printf("values[%d] = %f\n", i, values[i]);
        // }
    } else if (video_similarity_exist == 1 and video_similarity_type == 1) {
        // video_similarity(distance(v1, v2)) 
        printf("video_similarity(distance(v1, v2))\n");
        gbKey = (int*)malloc(alignedGpuTupleNum * sizeof(int));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gbKey, gpuGbKey, alignedGpuTupleNum * sizeof(int), hipMemcpyDeviceToHost));
        printf("vecCol attrSize = %d\n", gb->table->attrSize[video_similarity_index]);
        printf("tupleNum = %ld\n", gb->table->tupleNum);
        int colSize = gb->table->attrSize[video_similarity_index] * gb->table->tupleNum;
        std::vector<float> vecCol(colSize);
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(vecCol.data(), gb->table->content[video_similarity_index], colSize, hipMemcpyDeviceToHost));
        std::map<int, std::vector<float>> distmap;
        for(int i = 0; i < gb->table->tupleNum; i++){
            int key = gbKey[i];
            distmap[key].push_back(vecCol[i]);
        }

        std::cout << std::fixed << std::setprecision(7);
        // for(auto& pair : distmap) {
        //     int refLen = (pair.second).size() / searchFrameNum;
        //     for (int i = 0; i < searchFrameNum; ++i) {
        //         for (int j = 0; j < refLen; ++j) {
        //             std::cout << (pair.second)[i * refLen + j] << "\t";
        //         }
        //     std::cout << std::endl;
        //     }
        //     std::cout << std::endl;
        // }

        // multi threads
        ThreadPool pool(16);
        std::map<int, float> scoremap;
        for(auto& pair : distmap) {
            pool.enqueue([&scoremap, &pair, searchFrameNum]() {
                std::vector<std::vector<int>> seq;
                std::vector<int> seqScore;
                int res = calScore(pair.second, searchFrameNum, seq, seqScore);
                printf("score[%d] = %d\n", pair.first, res);
                std::lock_guard<std::mutex> lock(resmap_mutex);
                scoremap[pair.first] = res;
            });
        }
        while(1){
            if(pool.allTasksCompleted()) {
                std::cout << "All tasks are done!" << std::endl;
                break;
            }
        }

        // single thread
        // std::map<int, float> scoremap;
        // for(auto& pair : distmap) {
        //     std::vector<std::vector<int>> seq;
        //     std::vector<int> seqScore;
        //     printf("dist vec size = %d, searchFrameNum = %d\n", pair.second.size(), searchFrameNum);
        //     int res = calScore(pair.second, searchFrameNum, seq, seqScore);
        //     printf("score[%d] = %d\n", pair.first, res);
        //     scoremap[pair.first] = res;
        // }

        std::vector<int> keys;
        std::vector<float> values;
        for (const auto& pair : scoremap) {
            keys.push_back(pair.first);
            values.push_back(pair.second);
        }
        d_num = keys.size();

        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_keys, keys.size() * sizeof(int)));
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_values, values.size() * sizeof(float)));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_keys, keys.data(), keys.size() * sizeof(int), hipMemcpyHostToDevice));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_values, values.data(), values.size() * sizeof(float), hipMemcpyHostToDevice));
    
        for(int i = 0; i < values.size(); i++){
            printf("values[%d] = %f\n", i, values[i]);
        }
    }

    

    //Start timer for Step 5 - Allocate memory for result
    struct timespec startS5, endS5;
    clock_gettime(CLOCK_REALTIME,&startS5);

    char ** gpuResult = NULL;
    char ** result = NULL;

    // if(host_mp == NULL) {
    result = (char **)malloc(sizeof(char*)*res->totalAttr);
    CHECK_POINTER(result);
    // } else
    //     result = (char **) host_mp->alloc(sizeof(char *) * res->totalAttr);

    // if(dev_mp == NULL)
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuResult, sizeof(char *)* res->totalAttr));
    // else
        // gpuResult = (char **) dev_mp->alloc(sizeof(char *) * res->totalAttr);

    //Stop for 5 - Allocate memory for result
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS5);
    pp->groupby_step5_AllocRes += (endS5.tv_sec - startS5.tv_sec)* BILLION + endS5.tv_nsec - startS5.tv_nsec;

    //Start timer for Step 6 - Copy columns to device
    struct timespec startS6, endS6;
    clock_gettime(CLOCK_REALTIME,&startS6);

    printf("[groupby] res->tupleNum = %ld\n", res->tupleNum);
    printf("[groupby] res->totalAttr = %d\n", res->totalAttr);
    for(int i=0; i<res->totalAttr;i++){
        printf("[groupby] res->attrSize[%d] = %d\n", i, res->attrSize[i]);
    }

    for(int i=0; i<res->totalAttr;i++){
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&result[i], res->tupleNum * res->attrSize[i]));
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(result[i], 0, res->tupleNum * res->attrSize[i]));
        res->content[i] = result[i]; 
        res->dataPos[i] = GPU;
        res->attrTotalSize[i] = res->tupleNum * res->attrSize[i];
        //CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuResult[i], &result[i], sizeof(char *), hipMemcpyHostToDevice));

        if(gb->gbExp[i].func == MIN && res->attrSize[i] == sizeof(int))
            init_int_array<<<grid, block>>>((int *)result[i], res->tupleNum, FLOAT_MAX);
        else if(gb->gbExp[i].func == MAX && res->attrSize[i] == sizeof(int))
            init_int_array<<<grid, block>>>((int *)result[i], res->tupleNum, FLOAT_MIN);
    }
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(gpuResult, result, sizeof(char *) * res->totalAttr, hipMemcpyHostToDevice) );

    // if(dev_mp == NULL)
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbType, sizeof(int)*res->totalAttr));
    // else
    //     gpuGbType = (int *) dev_mp->alloc(sizeof(int) * res->totalAttr);
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbType, res->attrType, sizeof(int)*res->totalAttr, hipMemcpyHostToDevice));

    // if(dev_mp == NULL)
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbSize, sizeof(int)*res->totalAttr));
    // else
    //     gpuGbSize = (int *) dev_mp->alloc(sizeof(int) * res->totalAttr);
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbSize, res->attrSize, sizeof(int)*res->totalAttr, hipMemcpyHostToDevice));

    struct groupByExp *gpuGbExp;
    // if(dev_mp == NULL)
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuGbExp, sizeof(struct groupByExp)*res->totalAttr));
    // else
    //     gpuGbExp = (struct groupByExp *) dev_mp->alloc(sizeof(struct groupByExp) * res->totalAttr);
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbExp, gb->gbExp, sizeof(struct groupByExp)*res->totalAttr, hipMemcpyHostToDevice));

    for(int i=0;i<res->totalAttr;i++){
        struct mathExp * tmpMath;
        if(gb->gbExp[i].exp.opNum == 2){
            if(dev_mp == NULL)
                CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&tmpMath, 2* sizeof(struct mathExp)));
            else
                tmpMath = (struct mathExp *) dev_mp->alloc(2 * sizeof(struct mathExp));
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(tmpMath,(struct mathExp*)gb->gbExp[i].exp.exp,2*sizeof(struct mathExp), hipMemcpyHostToDevice));
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&(gpuGbExp[i].exp.exp), &tmpMath, sizeof(struct mathExp *), hipMemcpyHostToDevice));
        }
    }

    gpuGbColNum = res->totalAttr;

    //Stop for 6 - Copy columns to device
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS6);
    pp->groupby_step6_copyDataCols += (endS6.tv_sec - startS6.tv_sec)* BILLION + endS6.tv_nsec - startS6.tv_nsec;

    
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing

    //Start timer for Step 7 - Calculate aggregate values
    struct timespec startS7, endS7;
    clock_gettime(CLOCK_REALTIME,&startS7);

    if(gbConstant !=1){
        if(video_similarity_exist != 1)
            agg_cal<<<grid,block>>>(gpuContent, gpuGbColNum, gpuGbExp, gpuGbType, gpuGbSize, gpuTupleNum, gpuGbKey, gpu_psum, gpu_groupNum,gpuResult);
        else{
            agg_cal<<<grid,block>>>(gpuContent, gpuGbColNum, gpuGbExp, gpuGbType, gpuGbSize, gpuTupleNum, gpuGbKey, gpu_psum, gpu_groupNum,gpuResult, d_keys, d_values, d_num);
        }
        if(dev_mp == NULL) {
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbKey));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_psum));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_groupNum));
        }
    }
    else{
        if(gb->gbExp[0].func != OUTLIER_LINE){
            printf("[groupby] gb->gbExp[0].func != OUTLIER_LINE\n");
            agg_cal_cons<<<grid,block>>>(gpuContent, gpuGbColNum, gpuGbExp, gpuTupleNum,gpuResult);
        }else{
            printf("[OUTLIER_LINE] OUTLIER_LINE Processing\n");
            int colIndex = gb->gbExp[0].index;
            // dims of vector column
            int size = gb->table->attrSize[colIndex] / sizeof(float);
            printf("colIndex = %d\n", colIndex);
            size_t num  = gb->table->tupleNum;
            // obtain vector column
            // float *dataf = NULL;
            // CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&dataf, sizeof(float) * size * num));
            // CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(dataf, gb->table->content[colIndex], sizeof(float) * size * num, hipMemcpyHostToDevice));
            float *dataf = (float*)(gb->table->content[colIndex]);

            // for(int i = 0; i < 19*10; i++){
            //     printf("%f\n", dataf[i]);
            // }
            CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing

            // init faiss index
            faiss::gpu::StandardGpuResources resources;
            faiss::gpu::GpuIndexFlatConfig config;
            config.device = 0;
            int dim = size;
            
            printf("[outlier_line] tupleNum = %ld\n", num);
            printf("[outlier_line] column dim = %d\n", dim);
    
            CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing

            // for(int i = 0; i < 190; i++){
            //     printf("float[%d] = %f\n", i, dataf[i]);
            // }

            
            faiss::gpu::GpuIndexFlatL2 gpuIndex(&resources, dim, config);

            printf("BreakPoint\n");
            CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                printf("CUDA Error: %s\n", hipGetErrorString(err));
            }


            gpuIndex.add(num, dataf);
            CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
            
            printf("BreakPoint before Faiss search\n");
            // ANNs search for each element in column
            int k = 11;
            float* dist = (float*)malloc( num * k * sizeof(float));
            faiss::idx_t* ind = (faiss::idx_t*)malloc(num * k * sizeof(faiss::idx_t));
            gpuIndex.search(num, dataf, k, dist, ind);
            std::vector<float> sum(num);

            // Compute averages
            for (int i = 0; i < num; ++i) {
                sum[i] = computeSumForSegment(dist, i * k, k);
                // printf("%f\n", sum[i]);
            }

            // Sort the averages
            std::sort(sum.begin(), sum.end());

            // Get the 90% value
            int index = static_cast<int>(0.999 * num);
            float outlier_line = sum[index];

            printf("[outlier_line] 90 percent of device distance = %f\n", outlier_line);
            
            hipMemcpy(res->content[colIndex], &outlier_line, sizeof(float), hipMemcpyHostToDevice);

            // float *distSumHost = (float*)malloc(sizeof(float) * num);
            // CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(distSumHost, distSum, num * sizeof(float), hipMemcpyDeviceToHost));
            // int n = sizeof(distSumHost) / sizeof(distSumHost[0]);
            // printf("distSum size = %d\n", n);
            // std::sort(distSumHost, distSumHost + n);
            // int pos = static_cast<int>(num * 0.9) - 1;
            // printf("[outlier_line] 90% of host distance = %f", distSumHost[pos]);
        }
    }

    if (outlier_score_index == 1){
        // copy column to HOST
        float *scoreCol = (float*)malloc(sizeof(float) * gbCount);
        int *idCol = (int*)malloc(sizeof(int) * gbCount);
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(scoreCol, result[outlier_score_index], sizeof(float) * gbCount, hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(idCol, result[1-outlier_score_index], sizeof(int) * gbCount, hipMemcpyDeviceToHost));
        
        float *finalScoreCol = (float*)malloc(sizeof(float) * gbCount);
        float curScore = 0;
        int curPos = 0;
        
        normalize(scoreCol, gbCount);
        
        // perform sequential processing on outlier_score
        for(int i = 0; i < gbCount; i++){
            if(idCol[i] - curPos > 4){
                curScore = 0;
                finalScoreCol[i] = curScore;
                curPos = idCol[i];
            } else {
                curScore += scoreCol[i];
                finalScoreCol[i] = curScore;
                curPos = idCol[i];
            }
        }
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(result[outlier_score_index], finalScoreCol, sizeof(float) * gbCount, hipMemcpyHostToDevice));
    }

    //Stop for 7 - Calculate aggregate values
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS7);
    pp->groupby_step7_computeAgg += (endS7.tv_sec - startS7.tv_sec)* BILLION + endS7.tv_nsec - startS7.tv_nsec;

    //Start timer for Step 8 - De-allocate memory
    struct timespec startS8, endS8;
    clock_gettime(CLOCK_REALTIME,&startS8);

    for(int i=0; i<gb->table->totalAttr;i++){
        if(gb->table->dataPos[i]==MEM)
            CUDA_SAFE_CALL_NO_SYNC(hipFree(column[i]));
    }
    if(host_mp == NULL)
        free(column);
    if(dev_mp == NULL) {
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuContent));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbType));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbSize));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbExp));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuResult));
    }

    //Stop for 8 - De-allocate memory
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS8);
    pp->groupby_step8_deallocate += (endS8.tv_sec - startS8.tv_sec)* BILLION + endS8.tv_nsec - startS8.tv_nsec;
    
    //Stop total timer
    clock_gettime(CLOCK_REALTIME, &endS0);
    pp->groupby_totalTime += (endS0.tv_sec - startS0.tv_sec)* BILLION + endS0.tv_nsec - startS0.tv_nsec;
    
    //Increase count call
    pp->groupby_callTimes++;


    // hipMemcpy(intArray, res->content[0], sizeof(char) * 23 * res->tupleNum, hipMemcpyDeviceToHost);
    // for(int i = 0; i < res->tupleNum; i++){
    //     printf("video_name[%d] = ", i);
	// 	for(int j = 0; j < 23; j++){
    //         printf("%c", intArray[i*23 + j]);
    //     }
    //     printf("\n");
	// }

    // char floatArray[1024];
    // hipMemcpy(floatArray, res->content[1], sizeof(float) * res->tupleNum, hipMemcpyDeviceToHost);
	// for(int i = 0; i < res->tupleNum; i++){
	// 	printf("score[%d] = %f\n", i, ((float *)floatArray)[i]);
	// }

    // printf("[groupBy] attrType[1] :%d\n", res->attrType[1]);
    hipMemcpy(disArray, res->content[1], sizeof(float) * res->tupleNum, hipMemcpyDeviceToHost);
    for(int j = 0; j < 100; j++){
        printf("%f\t", disArray[j]);
    }

    // printf("[groupBy] attrType[0] :%d\n", res->attrType[0]);
    // hipMemcpy(disArray, res->content[0], res->attrSize[0] * res->tupleNum, hipMemcpyDeviceToHost);
    // for(int i = 0; i < 1; i++){
    //     printf("%f\n", disArray[i]);
    // }

    printf("[groupby] tupleSize = %d\n", res->tupleSize);
    
    clock_gettime(CLOCK_REALTIME,&e);
    tt = (e.tv_sec - s.tv_sec)* BILLION + e.tv_nsec - s.tv_nsec;
    printf("tt = %f\n", tt/(1000*1000));
    printf("<--Groupby Time -->    : %lf\n\n", pp->groupby_totalTime/(1000*1000));
    return res;
}

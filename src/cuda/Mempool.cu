#include "../include/Mempool.h"

Mempool::Mempool(int _type) :
    type(_type)
{
    if(type == MEM) {
        size = CPU_INIT_SIZE;
        base = (char *)malloc(size);
        assert(base != NULL);
    }else if(type == GPU) {
        size = GPU_INIT_SIZE;
        CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **)&base, size) );
    }
    free = base;
}

Mempool::~Mempool()
{
    if(type == MEM) {
        ::free(base);
    } else if(type == GPU) {
        CUDA_SAFE_CALL_NO_SYNC( hipFree(base) );
    }
}

void Mempool::freeto(char *pos)
{
    assert(pos <= free && pos >= base);
    free = pos;
}

void Mempool::resize(size_t newsize)
{
    if(newsize <= size)
        return;

    if(type == MEM) {
        size_t used = usedsize();
        base = (char *) realloc(base, newsize);
        assert(base != NULL);
        free = base + used;
    }else if(type == GPU) {
        char *tmp;
        CUDA_SAFE_CALL_NO_SYNC( hipMalloc((void **)&tmp, newsize) );
        CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(tmp, base, free - base, hipMemcpyDeviceToDevice) );
        free = tmp + (free - base);
        CUDA_SAFE_CALL_NO_SYNC( hipFree(base) );
        base = tmp;
    }
    size = newsize;
}

char *Mempool::alloc(size_t _size)
{
    // while(_size > freesize())
    //     resize(size * 2);

    char *tmp = free;
    free = free + _size;
    return tmp;
}

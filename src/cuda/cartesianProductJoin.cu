#include "hip/hip_runtime.h"
/*
   Copyright (c) 2012-2013 The Ohio State University.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/


#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "../include/common.h"
#include "../include/hashJoin.h"
#include "../include/gpuCudaLib.h"
#include "scanImpl.cu"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#include "../include/Mempool.h"

/*
 * copy left table column.
 */

__global__ void joinLeft(char* result, char* column, int attrSize, int num, int count) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num * count) {
        // cal index of current tid
        int elementIdx = tid / count;

        // initial path
        char* src = column + elementIdx * attrSize;
        char* dst = result + tid * attrSize;

        // copy element
        memcpy(result + tid * attrSize, column + elementIdx * attrSize, attrSize);
    }
}

__global__ void joinRight(char* result, char* column, int attrSize, int num, int count) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < count) {
        memcpy(result + tid * attrSize * num, column, attrSize * num);
    }
}

// __global__ void joinRight(char* result, char* column, int attrSize, int num, int count) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     int totalSize = attrSize * num;

//     // Total number of operations required
//     int operations = count * totalSize;

//     // Each thread copies one element (byte in this case)
//     if (tid < operations) {
//         int destIdx = tid + (tid / totalSize) * totalSize; // calculate destination index by skipping the already copied chunks
//         result[destIdx] = column[tid % totalSize]; // copy data from column to result
//     }
// }

/*
 * hashJoin implements the foreign key join between a fact table and dimension table.
 *
 * Prerequisites:
 *  1. the data to be joined can be fit into GPU device memory.
 *  2. dimension table is not compressed
 *  
 * Input:
 *  jNode: contains information about the two joined tables.
 *  pp: records statistics such as kernel execution time
 *
 * Output:
 *  A new table node
 */

struct tableNode * cartesianProductJoin(struct joinNode *jNode, struct statistic *pp,
                            Mempool *host_mp = NULL, Mempool *dev_mp = NULL, Mempool *res_mp = NULL, int *rightTableHash = NULL){

    // test cartesian input
    // int count = 5;
    // float * test = (float *)malloc(sizeof(float) * 2048 * count);
    // hipMemcpy(test, jNode->rightTable->content[1], sizeof(float) * 2048 * count, hipMemcpyDeviceToHost);
    // for(int i = 0; i < 2048 * count; i++){
    //     printf("(right table) column[%d] = %f\n", i, test[i]);
    // }
    // hipMemcpy(test, jNode->leftTable->content[0], sizeof(float) * 2048 * count, hipMemcpyDeviceToHost);
    // for(int i = 0; i < 2048 * count; i++){
    //     printf("(left table) column[%d] = %f\n", i, test[i]);
    // }

    printf("\n================ [cartesianProductJoin.cu] Start ================\n");

    //Start total timer
    struct timespec startS0, endS0;
    clock_gettime(CLOCK_REALTIME,&startS0);

    //Start timer for Step 1 - Allocate memory for intermediate results
    struct timespec startS1, endS1;
    clock_gettime(CLOCK_REALTIME,&startS1);

    struct tableNode * res = NULL;

    char *gpu_result = NULL, *gpu_fact = NULL;

    int defaultBlock = 4096;
    dim3 grid(defaultBlock);
    dim3 block(256);

    int blockNum;
    int threadNum;

    // blockNum = jNode->leftTable->tupleNum / block.x + 1;
    // if(blockNum < defaultBlock)
    //     grid = blockNum;
    // else
    //     grid = defaultBlock;
    // grid = 1;

    res = (struct tableNode*) malloc(sizeof(struct tableNode));
    CHECK_POINTER(res);

    res->totalAttr = jNode->totalAttr;
    res->tupleSize = jNode->tupleSize;
    res->attrType = (int *) malloc(res->totalAttr * sizeof(int));
    CHECK_POINTER(res->attrType);
    res->attrSize = (int *) malloc(res->totalAttr * sizeof(int));
    CHECK_POINTER(res->attrSize);
    res->attrIndex = (int *) malloc(res->totalAttr * sizeof(int));
    CHECK_POINTER(res->attrIndex);
    res->attrTotalSize = (int *) malloc(res->totalAttr * sizeof(int));
    CHECK_POINTER(res->attrTotalSize);
    res->dataPos = (int *) malloc(res->totalAttr * sizeof(int));
    CHECK_POINTER(res->dataPos);
    res->dataFormat = (int *) malloc(res->totalAttr * sizeof(int));
    CHECK_POINTER(res->dataFormat);
    res->content = (char **) malloc(res->totalAttr * sizeof(char *));
    CHECK_POINTER(res->content);

    for(int i=0;i<jNode->leftOutputAttrNum;i++){
        int pos = jNode->leftPos[i];
        res->attrType[pos] = jNode->leftOutputAttrType[i];
        int index = jNode->leftOutputIndex[i];
        res->attrSize[pos] = jNode->leftTable->attrSize[index];
        res->dataFormat[pos] = UNCOMPRESSED;
    }

    for(int i=0;i<jNode->rightOutputAttrNum;i++){
        int pos = jNode->rightPos[i];
        res->attrType[pos] = jNode->rightOutputAttrType[i];
        int index = jNode->rightOutputIndex[i];
        res->attrSize[pos] = jNode->rightTable->attrSize[index];
        res->dataFormat[pos] = UNCOMPRESSED;
    }

    long primaryKeySize = sizeof(int) * jNode->rightTable->tupleNum;

    //Stop for Step 1 - Allocate memory for intermediate results
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS1);
    pp->joinProf_step1_allocateMem += (endS1.tv_sec - startS1.tv_sec)* BILLION + endS1.tv_nsec - startS1.tv_nsec;

    //Start timer for Step 3 - Join
    struct timespec startS3, endS3;
    clock_gettime(CLOCK_REALTIME,&startS3);

   /*
    *  join on GPU
    */

    int leftTupleNum = jNode->leftTable->tupleNum;
    int rightTupleNum = jNode->rightTable->tupleNum;
    threadNum = grid.x * block.x;
    
    // printf("Join Thread number %d\n", threadNum);
    //sc 20 -> 512, 256
    //sc 15 -> 256, 256

    res->tupleNum = jNode->leftTable->tupleNum * jNode->rightTable->tupleNum;
    
    //Start timer for Step 4 - Materialize result
    struct timespec startS4, endS4;
    clock_gettime(CLOCK_REALTIME,&startS4);

    for(int i=0; i<res->totalAttr; i++){
        int index, pos;
        long colSize = 0, resSize = 0;
        int leftRight = 0;

        int attrSize, attrType;
        char * table = NULL;
        int found = 0 , dataPos, format;

        if (jNode->keepInGpu[i] == 1)
            res->dataPos[i] = GPU;
        else
            res->dataPos[i] = MEM;

        for(int k=0;k<jNode->leftOutputAttrNum;k++){
            if (jNode->leftPos[k] == i){
                found = 1;
                leftRight = 0;
                pos = k;
                break;
            }
        }
        if(!found){
            for(int k=0;k<jNode->rightOutputAttrNum;k++){
                if(jNode->rightPos[k] == i){
                    found = 1;
                    leftRight = 1;
                    pos = k;
                    break;
                }
            }
        }

        if(leftRight == 0){
            index = jNode->leftOutputIndex[pos];
            dataPos = jNode->leftTable->dataPos[index];
            format = jNode->leftTable->dataFormat[index];

            table = jNode->leftTable->content[index];
            attrSize  = jNode->leftTable->attrSize[index];
            attrType  = jNode->leftTable->attrType[index];
            colSize = jNode->leftTable->attrTotalSize[index];

            resSize = res->tupleNum * attrSize;
        }else{
            index = jNode->rightOutputIndex[pos];
            dataPos = jNode->rightTable->dataPos[index];
            format = jNode->rightTable->dataFormat[index];

            table = jNode->rightTable->content[index];
            attrSize = jNode->rightTable->attrSize[index];
            attrType = jNode->rightTable->attrType[index];
            colSize = jNode->rightTable->attrTotalSize[index];

            resSize = attrSize * res->tupleNum;
            leftRight = 1;
        }

        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_result,resSize));


        if(leftRight == 0){

            //Start timer for Step 41 - Materialize left table result
            struct timespec startS41, endS41;
            clock_gettime(CLOCK_REALTIME,&startS41);

            if(format == UNCOMPRESSED){

                if(dataPos == MEM || dataPos == MMAP || dataPos == PINNED){
                    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpu_fact, colSize));
                    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_fact, table, colSize,hipMemcpyHostToDevice));
                }else{
                    gpu_fact = table;
                }
                
                int threadsPerBlock = 256;
                int blocks = (leftTupleNum * rightTupleNum + threadsPerBlock - 1) / threadsPerBlock;
                joinLeft<<<blocks,threadsPerBlock>>>(gpu_result, gpu_fact, attrSize, leftTupleNum, rightTupleNum);
                CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());
                // if(attrSize == sizeof(int))
                //     joinFact_int<<<grid,block>>>(filterPsum,gpu_fact, attrSize, jNode->leftTable->tupleNum,filterNum,gpu_result);
                // else
                //     joinFact_other<<<grid,block>>>(gpu_resPsum,gpu_fact, attrSize, jNode->leftTable->tupleNum,gpuFactFilter,gpu_result);

            }

            //Stop for Step 41 - Materialize left table result
            CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
            clock_gettime(CLOCK_REALTIME, &endS41);
            pp->joinProf_step41_materialize_res_left += (endS41.tv_sec - startS41.tv_sec)* BILLION + endS41.tv_nsec - startS41.tv_nsec;

        }else{

            //Start timer for Step 42 - Materialize right table result
            struct timespec startS42, endS42;
            clock_gettime(CLOCK_REALTIME,&startS42);

            if(format == UNCOMPRESSED){

                if(dataPos == MEM || dataPos == MMAP || dataPos == PINNED){
                    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpu_fact, colSize));
                    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_fact, table, colSize,hipMemcpyHostToDevice));
                }else{
                    gpu_fact = table;
                }

                int block = 256;
                int grid = (leftTupleNum * rightTupleNum * attrSize + block - 1) / block;
                printf("[cartesianJoin] attrSize = %d, rightTupleNum = %d, leftTupleNum = %d\n", attrSize, rightTupleNum, leftTupleNum);
                joinRight<<<grid, block>>>(gpu_result, gpu_fact, attrSize, rightTupleNum, leftTupleNum);
                CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());
                // if(attrType == sizeof(int))
                //     joinDim_int_new<<<grid,block>>>(filterPsum,gpu_fact, attrSize, jNode->leftTable->tupleNum, filterNum, JRes, gpu_result);
                // else
                //     joinDim_other<<<grid,block>>>(gpu_resPsum,gpu_fact, attrSize, jNode->leftTable->tupleNum, gpuFactFilter,gpu_result);

            }

            //Stop for Step 42 - Materialize right table result
            CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
            clock_gettime(CLOCK_REALTIME, &endS42);
            pp->joinProf_step42_materialize_res_right += (endS42.tv_sec - startS42.tv_sec)* BILLION + endS42.tv_nsec - startS42.tv_nsec;
        }
        
        res->attrTotalSize[i] = resSize;
        res->dataFormat[i] = UNCOMPRESSED;
        if(res->dataPos[i] == MEM){
            res->content[i] = (char *) malloc(resSize);
            memset(res->content[i],0,resSize);
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(res->content[i],gpu_result,resSize,hipMemcpyDeviceToHost));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_result));

        }else if(res->dataPos[i] == GPU){
            res->content[i] = gpu_result;
        }
        if(dataPos == MEM || dataPos == MMAP || dataPos == PINNED)
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_fact));

    }

    //Stop for Step 4 - Materialize result
    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
    clock_gettime(CLOCK_REALTIME, &endS4);
    pp->joinProf_step4_materialize_res += (endS4.tv_sec - startS4.tv_sec)* BILLION + endS4.tv_nsec - startS4.tv_nsec;
    
    //Stop total timer
    clock_gettime(CLOCK_REALTIME, &endS0);
    pp->cartesianJoin_totalTime += (endS0.tv_sec - startS0.tv_sec)* BILLION + endS0.tv_nsec - startS0.tv_nsec;
    
    //Increase count call
    pp->join_callTimes++;

    // test cartesian output
    // hipMemcpy(test, res->content[1], sizeof(float) * 2048 * count, hipMemcpyDeviceToHost);
    // for(int i = 0; i < 2048 * count; i++){
    //     printf("(cartesianProductJoin) column[%d] = %f\n", i, test[i]);
    // }

    return res;

}
